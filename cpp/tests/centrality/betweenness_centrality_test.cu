#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gtest/gtest.h"
#include "gmock/gmock.h"

#include <thrust/device_vector.h>
#include "test_utils.h"
#include <utility>

#include <graph.hpp>
#include <algorithms.hpp>

#include <queue> // C++ Reference Algorithm
#include <stack> // C++ Reference Algorithm

#include <converters/COOtoCSR.cuh> // Loads GraphCSR from .mtx
#include <fstream>


// =============================================================================
// C++ Reference Implementation
// =============================================================================
template<typename VT, typename ET>
void populate_neighbors(VT *indices, ET *offsets,
                        VT w, std::vector<VT> &neighbors) {
  ET edge_start = offsets[w];
  ET edge_end = offsets[w + 1];
  ET edge_count = edge_end - edge_start;

  neighbors.clear(); // Reset neighbors vector's size
  for (ET edge_idx = 0; edge_idx < edge_count; ++edge_idx) {
    VT dst = indices[edge_start + edge_idx];
    neighbors.push_back(dst);
  }
}

// TODO: This should be moved to BFS testing on the c++ side
// This implements the BFS from (Brandes, 2001)
template<typename VT, typename ET, typename WT, typename result_t>
void ref_bfs(VT *indices, ET *offsets, VT const number_of_vertices,
             std::queue<VT> &Q,
             std::stack<VT> &S,
             std::vector<VT> &dist,
             std::vector<std::vector<VT>> &pred,
             std::vector<result_t> &sigmas,
             VT s) { // TODO(xcadet) Should rename to source
  std::vector<VT> neighbors;
  for (VT w = 0 ; w < number_of_vertices; ++w) {
    pred[w].clear();
    dist[w] = std::numeric_limits<VT>::max();
    sigmas[w] = 0;
  }
  dist[s] = 0;
  sigmas[s] = 1;
  Q.push(s);
  //   b. Traversal
  while (!Q.empty()) {
    VT v = Q.front();
    Q.pop();
    S.push(v);
    populate_neighbors<VT, ET>(indices, offsets, v, neighbors);
    for (VT w : neighbors) {
      // Path Discovery:
      // Found for the first time?
      if (dist[w] == std::numeric_limits<VT>::max()) {
        dist[w] = dist[v] + 1;
        Q.push(w);
      }
      // Path counting
      // Edge(v, w) on  a shortest path?
      if (dist[w] == dist[v] + 1) {
        sigmas[w] +=  sigmas[v];
        pred[w].push_back(v);
        // TODO(xcadet) This is for debugging purpose (78 is a problem in email-EU-core)
        if (w == 718) {
          printf("[DBG][REF][BFS] %d(%d)[%d] -> %d(%d)[%d]\n", v, dist[v], (int)sigmas[v], w, dist[w], (int)sigmas[w]);
        }
      }
    }
  }
}

// Algorithm 1: Shortest-path vertex betweenness, (Brandes, 2001)
template <typename VT, typename ET, typename WT, typename result_t>
void reference_betweenness_centrality_impl(VT *indices, ET *offsets,
                                           VT const number_of_vertices,
                                           result_t *result) {
  std::queue<VT> Q;
  std::stack<VT> S;
  // NOTE: dist is of type VT not WT
  std::vector<VT> dist(number_of_vertices);
  std::vector<std::vector<VT>> pred(number_of_vertices);
  std::vector<result_t> sigmas(number_of_vertices);
  std::vector<result_t> deltas(number_of_vertices);

  std::vector<VT> neighbors;

  for (VT s = 0; s < number_of_vertices; ++s) { 
    // Step 1: Single-source shortest-paths problem
    //   a. Initialization
    ref_bfs<VT, ET, WT, result_t>(indices, offsets, number_of_vertices,
                                  Q, S,
                                  dist, pred, sigmas, s);
    //  Step 2: Accumulation
    //          Back propagation of dependencies
    for (VT v = 0; v < number_of_vertices; ++v) {
      deltas[v] = 0;
    }
    while (!S.empty()) {
      VT w = S.top();
      S.pop();
      for (VT v : pred[w]) {
        deltas[v] += (sigmas[v] / sigmas[w]) * (1.0 + deltas[w]);
      }
      if (w != s) {
        result[w] += deltas[w];
      }
    }
  }
}

template <typename VT, typename ET, typename WT, typename result_t>
void reference_betweenness_centrality(cugraph::experimental::GraphCSR<VT, ET, WT> &graph,
                                      result_t *result, bool normalize) {

  VT number_of_vertices = graph.number_of_vertices;
  ET number_of_edges = graph.number_of_edges;
  std::vector<VT> indices(number_of_edges);
  std::vector<ET> offsets(number_of_vertices + 1);

  hipMemcpy(indices.data(), graph.indices,
             sizeof(VT) * indices.size(), hipMemcpyDeviceToHost);
  hipMemcpy(offsets.data(), graph.offsets,
             sizeof(ET) * offsets.size(), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  reference_betweenness_centrality_impl<VT, ET, WT, result_t>(indices.data(), offsets.data(),
                                        number_of_vertices, result);
  if (normalize && number_of_vertices > 2) {
    result_t factor = static_cast<result_t>(number_of_vertices - 1) * static_cast<result_t>(number_of_vertices - 2);
    for (VT v = 0; v < number_of_vertices; ++v) {
      result[v] /= factor;
    }
  }
}
// Explicit declaration
template void reference_betweenness_centrality<int, int, float, float>(cugraph::experimental::GraphCSR<int, int, float> &,
                                                                  float *, bool);
// =============================================================================
// Utility functions
// =============================================================================
/**
 * @brief     Extract betweenness centality values from file
 *
 * This function reads the content of a file containing betweenness values
 * The expected format per line is '<vertex_idx> <betweenness_centrality>'
 *
 * @tparam VT           Type of vertex identifiers. Supported value : int (signed, 32-bit)
 * @tparam ET           Type of edge identifiers. Supported value : int (signed, 32-bit)
 * @tparam result_t     Type of betweenness centrality value: float
 *
 * @param[out] result   Reference to a vector that is resized and filled with betweenness value
 * @param[in] bc_file   Path to the file to extract betweenness from
 *
 */
// FIXME: This is not BC specific, it simply reads '<VT> <result_t>\n' files
template <typename VT, typename result_t>
void extract_bc(std::vector<result_t> &result, std::string bc_file) {
  VT vid = 0; // Not really usefull, nx_bc_file is expected to be sorted
  result_t bc = 0; // Not really usefull, nx_bc_file is expected to be sorted

  result.clear();
  std::ifstream ifs(bc_file);
  ASSERT_TRUE(ifs.is_open());

  while (ifs >> vid >> bc) {
    result.push_back(bc);
  }
  ifs.close();
}

// TODO(xcadet): This could be useful in other testsuite (SSSP, BFS, ...)
template<typename VT, typename ET, typename WT>
void generate_graph_csr(CSR_Result_Weighted<VT, WT> &csr_result, VT &m, VT &nnz, std::string matrix_file) {
  FILE* fpin = fopen(matrix_file.c_str(),"r");
  ASSERT_NE(fpin, nullptr) << "fopen (" << matrix_file << ") failure.";

  int k;
  MM_typecode mc;
  ASSERT_EQ(mm_properties<int>(fpin, 1, &mc, &m, &k, &nnz),0) << "could not read Matrix Market file properties"<< "\n";
  ASSERT_TRUE(mm_is_matrix(mc));
  ASSERT_TRUE(mm_is_coordinate(mc));
  ASSERT_FALSE(mm_is_complex(mc));
  ASSERT_FALSE(mm_is_skew(mc));

  // Allocate memory on host
  std::vector<int> cooRowInd(nnz), cooColInd(nnz);
  std::vector<float> cooVal(nnz);

  // Read
  ASSERT_EQ( (mm_to_coo<int, float>(fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], &cooVal[0], NULL)) , 0)<< "could not read matrix data"<< "\n";
  ASSERT_EQ(fclose(fpin),0);

  ConvertCOOtoCSR_weighted(&cooRowInd[0], &cooColInd[0], &cooVal[0], nnz, csr_result);
}

// TODO(xcadet): This may actually operate an exact comparison when b == 0
template <typename T>
bool compare_close(const T &a, const T&b, const double epsilon) {
  return (a >= b * (1.0 - epsilon)) and (a <= b * (1.0 + epsilon));
}


// =============================================================================
// Test Suite
// =============================================================================
struct BetweennessCentralityTest : public ::testing::Test
{
};

struct BetweennessCentralityBFSTest : public ::testing::Test
{
};


// BFS: Checking for shortest_path counting correctness
// -----------------------------------------------------------------------------
// TODO(xcadet) Parametrize this part for VT, ET, WT, result_t
TEST_F(BetweennessCentralityBFSTest, CheckReference) {
  // TODO(xcadet) This dataset was manually generated and is not provided
  //std::string matrix_file(get_rapids_dataset_root_dir() + "/" + "email-Eu-core-gen.mtx");
  std::string matrix_file("../../datasets/email-Eu-core-gen.mtx");
  int m, nnz;
  CSR_Result_Weighted<int, float> csr_result;
  generate_graph_csr<int, int, float>(csr_result, m, nnz, matrix_file);
  cugraph::experimental::GraphCSR<int, int, float> graph(csr_result.rowOffsets,
                                                     csr_result.colIndices,
                                                     csr_result.edgeWeights,
                                                     m, nnz);
  // FIXME: THIS IS CRITICAL:
  graph.prop.directed = true;
  std::vector<float> result(graph.number_of_vertices);

  int source = 2;
  // Ref BC_BFS requires many working values
  int number_of_vertices = graph.number_of_vertices;
  int number_of_edges = graph.number_of_edges;
  //
  std::vector<int> indices(number_of_edges);
  std::vector<int> offsets(number_of_vertices + 1);

  hipMemcpy(indices.data(), graph.indices,
             sizeof(int) * indices.size(), hipMemcpyDeviceToHost);
  hipMemcpy(offsets.data(), graph.offsets,
             sizeof(int) * offsets.size(), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  std::queue<int> Q;
  std::stack<int> S;
  std::vector<int> ref_bfs_dist(number_of_vertices);
  std::vector<std::vector<int>> ref_bfs_pred(number_of_vertices);
  std::vector<float> ref_bfs_sigmas(number_of_vertices);
  ref_bfs<int, int, float, float>(indices.data(), offsets.data(),
                                  number_of_vertices, Q, S,
                                  ref_bfs_dist, ref_bfs_pred,
                                  ref_bfs_sigmas, source);



  // Device data for cugraph_bfs
  thrust::device_vector<int> d_cugraph_dist(number_of_vertices);
  thrust::device_vector<int> d_cugraph_pred(number_of_vertices);
  thrust::device_vector<int> d_cugraph_sigmas(number_of_vertices);

  // This test only checks for sigmas equality
  std::vector<int> cugraph_sigmas(number_of_vertices);

  printf("Is graph directed ? %d\n", graph.prop.directed);
  cugraph::bfs<int, int, float>(graph, d_cugraph_dist.data().get(),
                                d_cugraph_pred.data().get(),
                                d_cugraph_sigmas.data().get(),
                                source, graph.prop.directed);
  hipMemcpy(cugraph_sigmas.data(), d_cugraph_sigmas.data().get(),
             sizeof(int) * d_cugraph_sigmas.size(), hipMemcpyDeviceToHost);
  // TODO(xcadet): The implicit cast comes from BFS shortest_path counter being
  // of type VT, while the ref_bfs uses float values
  for (int i = 0 ; i < number_of_vertices ; ++i) {
    EXPECT_TRUE(compare_close((float)cugraph_sigmas[i], ref_bfs_sigmas[i], 0.0001)) <<
                "[MISMATCH] vaid = " << i << ", cugraph = " <<
                cugraph_sigmas[i] << " c++ ref = " << ref_bfs_sigmas[i];
    //std::cout << "Sigmas[" << i << "] = " << cugraph_sigmas[i] << std::endl;
  }
  std::cout << "Graph number_of_vertices " << number_of_vertices << ", number_of_edges " << number_of_edges << std::endl;
  int sum_sigmas_cugraph = thrust::reduce(thrust::host, cugraph_sigmas.begin(), cugraph_sigmas.end(), 0);
  int sum_sigmas_ref = thrust::reduce(thrust::host, ref_bfs_sigmas.begin(), ref_bfs_sigmas.end(), 0);
  std::cout << "Source " << source << ", cugraph: " << sum_sigmas_cugraph << ", ref " << sum_sigmas_ref << std::endl;;
}


// BC
// -----------------------------------------------------------------------------
/*
TEST_F(BetweennessCentralityTest, CheckReference)
{
  // FIXME: This could be standardized for tests?
  //        Could simplify usage of external storage
  //std::string matrix_file(get_rapids_dataset_root_dir() + "/" + "netscience.mtx");
  //std::string matrix_file(get_rapids_dataset_root_dir() + "/" + "karate.mtx");
  std::string matrix_file(get_rapids_dataset_root_dir() + "/" + "polbooks.mtx");
  int m, nnz;
  CSR_Result_Weighted<int, float> csr_result;
  generate_graph_csr<int, int, float>(csr_result, m, nnz, matrix_file);
  cugraph::experimental::GraphCSR<int, int, float> G(csr_result.rowOffsets,
                                                     csr_result.colIndices,
                                                     csr_result.edgeWeights,
                                                     m, nnz);

  std::vector<float>            result(G.number_of_vertices);
  std::vector<float> expected;

  //extract_bc<int, float>(expected, std::string("../../nxcheck/nx_netscience.txt"));
  //extract_bc<int, float>(expected, std::string("../../nxcheck/nx_karate.txt"));
  //extract_bc<int, float>(expected, std::string("../../nxcheck/nx_dolphins.txt"));
  extract_bc<int, float>(expected, std::string("../../nxcheck/nx_polbooks_unormalized.txt"));

  //cugraph::betweenness_centrality(G, d_result.data().get());
  //hipMemcpy(result.data(), d_result.data().get(), sizeof(float) * num_verts, hipMemcpyDeviceToHost);

  std::vector<float> ref_result(G.number_of_vertices);
  reference_betweenness_centrality(G, ref_result.data(), false);
  for (int i = 0 ; i < G.number_of_vertices ; ++i)
    EXPECT_TRUE(compare_close(ref_result[i], expected[i], 0.0001)) <<
                "[MISMATCH] vaid = " << i << ", c++ implem = " <<
                ref_result[i] << " expected = " << expected[i];
}

TEST_F(BetweennessCentralityTest, SimpleGraph)
{
  std::vector<int>  graph_offsets{ { 0, 1, 2, 5, 7, 10, 12, 14 } };
  std::vector<int>  graph_indices{ { 2, 2, 0, 1, 3, 2, 4, 3, 5, 6, 4, 6, 4, 5 } };

  std::vector<float> expected{ {0.0, 0.0, 0.6, 0.6, 0.5333333, 0.0, 0.0 } };

  int num_verts = graph_offsets.size() - 1;
  int num_edges = graph_indices.size();

  thrust::device_vector<int>    d_graph_offsets(graph_offsets);
  thrust::device_vector<int>    d_graph_indices(graph_indices);
  thrust::device_vector<float>  d_result(num_verts);

  std::vector<float>            result(num_verts);

  cugraph::experimental::GraphCSR<int,int,float> G(d_graph_offsets.data().get(),
                                                   d_graph_indices.data().get(),
                                                   nullptr,
                                                   num_verts,
                                                   num_edges);

  cugraph::betweenness_centrality(G, d_result.data().get());

  hipMemcpy(result.data(), d_result.data().get(), sizeof(float) * num_verts, hipMemcpyDeviceToHost);

  for (int i = 0 ; i < num_verts ; ++i)
    EXPECT_FLOAT_EQ(result[i], expected[i]);

  // TODO(xcadet) Remove this part, it is for testing the reference
  std::vector<float> ref_result(num_verts);
  reference_betweenness_centrality(G, ref_result.data(), true);
  for (int i = 0 ; i < num_verts ; ++i)
    EXPECT_FLOAT_EQ(ref_result[i], expected[i]);
}
*/

int main( int argc, char** argv )
{
    rmmInitialize(nullptr);
    testing::InitGoogleTest(&argc,argv);
    int rc = RUN_ALL_TESTS();
    rmmFinalize();
    return rc;
}