#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gtest/gtest.h"
#include "gmock/gmock.h"

#include <thrust/device_vector.h>
#include "test_utils.h"
#include <utility>

#include <graph.hpp>
#include <algorithms.hpp>

#include <queue> // C++ Reference Algorithm
#include <stack> // C++ Reference Algorithm

#include <converters/COOtoCSR.cuh> // Loads GraphCSR from .mtx
#include <fstream>

#ifndef TEST_EPSILON
 #define TEST_EPSILON 0.0001
#endif


// =============================================================================
// C++ Reference Implementation
// =============================================================================
template<typename VT, typename ET>
void populate_neighbors(VT *indices, ET *offsets,
                        VT w, std::vector<VT> &neighbors) {
  ET edge_start = offsets[w];
  ET edge_end = offsets[w + 1];
  ET edge_count = edge_end - edge_start;

  neighbors.clear(); // Reset neighbors vector's size
  for (ET edge_idx = 0; edge_idx < edge_count; ++edge_idx) {
    VT dst = indices[edge_start + edge_idx];
    neighbors.push_back(dst);
  }
}

// TODO: This colud be moved to BFS testing on the c++ side
// This implements the BFS from (Brandes, 2001) with shortest path counting
template<typename VT, typename ET, typename WT, typename result_t>
void ref_bfs(VT *indices, ET *offsets, VT const number_of_vertices,
             std::queue<VT> &Q,
             std::stack<VT> &S,
             std::vector<VT> &dist,
             std::vector<std::vector<VT>> &pred,
             std::vector<result_t> &sigmas,
             VT source) {
  std::vector<VT> neighbors;
  for (VT w = 0 ; w < number_of_vertices; ++w) {
    pred[w].clear();
    dist[w] = std::numeric_limits<VT>::max();
    sigmas[w] = 0;
  }
  dist[source] = 0;
  sigmas[source] = 1;
  Q.push(source);
  //   b. Traversal
  while (!Q.empty()) {
    VT v = Q.front();
    Q.pop();
    S.push(v);
    populate_neighbors<VT, ET>(indices, offsets, v, neighbors);
    for (VT w : neighbors) {
      // Path Discovery:
      // Found for the first time?
      if (dist[w] == std::numeric_limits<VT>::max()) {
        dist[w] = dist[v] + 1;
        Q.push(w);
      }
      // Path counting
      // Edge(v, w) on  a shortest path?
      if (dist[w] == dist[v] + 1) {
        sigmas[w] +=  sigmas[v];
        pred[w].push_back(v);
      }
    }
  }
}

template<typename VT, typename ET, typename WT, typename result_t>
void ref_accumulation(result_t *result,
                      VT const number_of_vertices,
                      std::stack<VT> &S,
                      std::vector<std::vector<VT>> &pred,
                      std::vector<result_t> &sigmas,
                      std::vector<result_t> &deltas,
                      VT source) {
  for (VT v = 0; v < number_of_vertices; ++v) {
    deltas[v] = 0;
  }
  while (!S.empty()) {
    VT w = S.top();
    S.pop();
    for (VT v : pred[w]) {
      deltas[v] += (sigmas[v] / sigmas[w]) * (1.0 + deltas[w]);
    }
    if (w != source) {
      result[w] += deltas[w];
    }
  }
}

// Algorithm 1: Shortest-path vertex betweenness, (Brandes, 2001)
template <typename VT, typename ET, typename WT, typename result_t>
void reference_betweenness_centrality_impl(VT *indices, ET *offsets,
                                           VT const number_of_vertices,
                                           result_t *result,
                                           VT const *sources,
                                           VT const number_of_sources) {
  std::queue<VT> Q;
  std::stack<VT> S;
  // NOTE: dist is of type VT not WT
  std::vector<VT> dist(number_of_vertices);
  std::vector<std::vector<VT>> pred(number_of_vertices);
  std::vector<result_t> sigmas(number_of_vertices);
  std::vector<result_t> deltas(number_of_vertices);

  std::vector<VT> neighbors;

  if (sources) {
    for (VT source_idx = 0; source_idx < number_of_sources; ++source_idx) {
      VT s = sources[source_idx];
      // Step 1: Single-source shortest-paths problem
      //   a. Initialization
      ref_bfs<VT, ET, WT, result_t>(indices, offsets, number_of_vertices,
                                    Q, S,
                                    dist, pred, sigmas, s);
      //  Step 2: Accumulation
      //          Back propagation of dependencies
      ref_accumulation<VT, ET, WT, result_t>(result,
                                             number_of_vertices,
                                             S,
                                             pred,
                                             sigmas,
                                             deltas,
                                             s);
    }
  } else {
    for (VT s = 0; s < number_of_vertices; ++s) {
      // Step 1: Single-source shortest-paths problem
      //   a. Initialization
      ref_bfs<VT, ET, WT, result_t>(indices, offsets, number_of_vertices,
                                    Q, S,
                                    dist, pred, sigmas, s);
      //  Step 2: Accumulation
      //          Back propagation of dependencies
      ref_accumulation<VT, ET, WT, result_t>(result,
                                             number_of_vertices,
                                             S,
                                             pred,
                                             sigmas,
                                             deltas,
                                             s);
    }
  }
}

template <typename VT, typename ET, typename WT, typename result_t>
void reference_betweenness_centrality(cugraph::experimental::GraphCSR<VT, ET, WT> const &graph,
                                      result_t *result,
                                      bool normalize,
                                      bool endpoints, // This is not yet implemented
                                      VT const number_of_sources,
                                      VT const *sources) {

  VT number_of_vertices = graph.number_of_vertices;
  ET number_of_edges = graph.number_of_edges;
  thrust::host_vector<VT> h_indices(number_of_edges);
  thrust::host_vector<ET> h_offsets(number_of_vertices + 1);

  thrust::device_ptr<VT>  d_indices((VT *)&graph.indices[0]);
  thrust::device_ptr<ET>  d_offsets((ET *)&graph.offsets[0]);

  thrust::copy(d_indices, d_indices + number_of_edges, h_indices.begin());
  thrust::copy(d_offsets, d_offsets + (number_of_vertices + 1), h_offsets.begin());


  /*
  hipMemcpyAsync(&h_indices[0], &graph.indices[0],
             sizeof(VT) * h_indices.size(), hipMemcpyDeviceToHost, nullptr);
  hipMemcpyAsync(&h_offsets[0], &graph.offsets[0],
             h_offsets.size() * sizeof(ET), hipMemcpyDeviceToHost, nullptr);
  CUDA_CHECK_LAST();
  */
  hipDeviceSynchronize();
  reference_betweenness_centrality_impl<VT, ET, WT, result_t>(&h_indices[0],
                                                              &h_offsets[0],
                                                              number_of_vertices,
                                                              result, sources,
                                                              number_of_sources);
  if (normalize && number_of_vertices > 2) {
    result_t factor = static_cast<result_t>(number_of_vertices - 1) * static_cast<result_t>(number_of_vertices - 2);
    for (VT v = 0; v < number_of_vertices; ++v) {
      result[v] /= factor;
    }
  }
}
// Explicit declaration
template void reference_betweenness_centrality<int, int, float, float>(cugraph::experimental::GraphCSR<int, int, float> const&,
                                                                  float *, bool, bool, const int, int const *);
// =============================================================================
// Utility functions
// =============================================================================
/**
 * @brief     Extract betweenness centality values from file
 *
 * This function reads the content of a file containing betweenness values
 * The expected format per line is '<vertex_idx> <betweenness_centrality>'
 *
 * @tparam VT           Type of vertex identifiers. Supported value : int (signed, 32-bit)
 * @tparam ET           Type of edge identifiers. Supported value : int (signed, 32-bit)
 * @tparam result_t     Type of betweenness centrality value: float
 *
 * @param[out] result   Reference to a vector that is resized and filled with betweenness value
 * @param[in] bc_file   Path to the file to extract betweenness from
 *
 */
// FIXME: This is not BC specific, it simply reads '<VT> <result_t>\n' files
template <typename VT, typename result_t>
void extract_bc(std::vector<result_t> &result, std::string bc_file) {
  VT vid = 0; // Not really usefull, nx_bc_file is expected to be sorted
  result_t bc = 0; // Not really usefull, nx_bc_file is expected to be sorted

  result.clear();
  std::ifstream ifs(bc_file);
  ASSERT_TRUE(ifs.is_open());

  while (ifs >> vid >> bc) {
    result.push_back(bc);
  }
  ifs.close();
}

// TODO(xcadet): This could be useful in other testsuite (SSSP, BFS, ...)
template<typename VT, typename ET, typename WT>
void generate_graph_csr(CSR_Result_Weighted<VT, WT> &csr_result, VT &m, VT &nnz, bool &is_directed, std::string matrix_file) {
  FILE* fpin = fopen(matrix_file.c_str(),"r");
  ASSERT_NE(fpin, nullptr) << "fopen (" << matrix_file << ") failure.";

  int k;
  MM_typecode mc;
  ASSERT_EQ(mm_properties<int>(fpin, 1, &mc, &m, &k, &nnz),0) << "could not read Matrix Market file properties"<< "\n";
  ASSERT_TRUE(mm_is_matrix(mc));
  ASSERT_TRUE(mm_is_coordinate(mc));
  ASSERT_FALSE(mm_is_complex(mc));
  ASSERT_FALSE(mm_is_skew(mc));
  is_directed = !mm_is_symmetric(mc);

  // Allocate memory on host
  std::vector<int> cooRowInd(nnz), cooColInd(nnz);
  std::vector<float> cooVal(nnz);

  // Read
  ASSERT_EQ( (mm_to_coo<int, float>(fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], &cooVal[0], NULL)) , 0)<< "could not read matrix data"<< "\n";
  ASSERT_EQ(fclose(fpin),0);

  ConvertCOOtoCSR_weighted(&cooRowInd[0], &cooColInd[0], &cooVal[0], nnz, csr_result);
  CUDA_CHECK_LAST();
}

// TODO(xcadet): This may actually operate an exact comparison when b == 0
template <typename T>
bool compare_close(const T &a, const T&b, const double epsilon) {
  return (a >= b * (1.0 - epsilon)) and (a <= b * (1.0 + epsilon));
}


// =============================================================================
// Test Suite
// =============================================================================

// Defines Betweenness Centrality UseCase
// SSSP codes uses type of Graph parameter that could be used
//TODO(xcadet) Use VT for srcs
typedef struct BC_Usecase_t {
  std::string config_;
  std::string file_path_;
  int number_of_sources_;
  BC_Usecase_t(const std::string& config, int number_of_sources)
               : config_(config), number_of_sources_(number_of_sources) {
    // assume relative paths are relative to RAPIDS_DATASET_ROOT_DIR
    // FIXME: Use platform independent stuff from c++14/17 on compiler update
    const std::string& rapidsDatasetRootDir = get_rapids_dataset_root_dir();
    if ((config_ != "") && (config_[0] != '/')) {
      file_path_ = rapidsDatasetRootDir + "/" + config_;
    } else {
      file_path_ = config_;
    }
  };
} BC_Usecase;

/*
struct BetweennessCentralityTest : public ::testing::Test
{
};
*/
class Tests_BC : public ::testing::TestWithParam<BC_Usecase> {
  public:
  Tests_BC() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}
  // TODO(xcadet) Should normalize be part of the configuration?
  template <typename VT, typename ET, typename WT, typename result_t,
            bool normalize, bool endpoints>
  void run_current_test(const BC_Usecase &configuration) {
    // Step 1: Construction of the graph based on configuration
    VT m;
    ET nnz;
    CSR_Result_Weighted<VT, WT> csr_result;
    bool is_directed = false;
    generate_graph_csr<VT, ET, WT>(csr_result, m, nnz, is_directed,
                                   configuration.file_path_);
    hipDeviceSynchronize();
    cugraph::experimental::GraphCSR<VT, ET, WT> G(csr_result.rowOffsets,
                                                  csr_result.colIndices,
                                                  csr_result.edgeWeights,
                                                  m, nnz);
    G.prop.directed = is_directed;

    CUDA_CHECK_LAST();
    std::vector<result_t> result(G.number_of_vertices, 0);
    std::vector<result_t> expected(G. number_of_vertices, 0);

    // Step 2: Generation of sources based on configuration
    //         if number_of_sources_ is 0 then sources must be nullptr
    //         Otherwise we only  use the first k values
    ASSERT_TRUE(configuration.number_of_sources_ >= 0
           && configuration.number_of_sources_ <= G.number_of_vertices)
           << "Number number of sources should be >= 0 and"
           << " less than the number of vertices in the graph";
    std::vector<VT> sources(configuration.number_of_sources_);
    std::iota(sources.begin(), sources.end(), 0);

    VT *sources_ptr = nullptr;
    if (configuration.number_of_sources_ > 0) {
      sources_ptr = sources.data();
    }

    // TODO(xcadet) reference should also include normalize, endpooint, number_of_sources and sources
    reference_betweenness_centrality(G, expected.data(),
                                     normalize, endpoints,
                                     //weights
                                     configuration.number_of_sources_,
                                     sources_ptr);

    sources_ptr = nullptr;
    if (configuration.number_of_sources_ > 0) {
      sources_ptr = sources.data();
    }

    printf("[DBG] Number of vertices %d\n", G.number_of_vertices);
    thrust::device_vector<result_t>  d_result(G.number_of_vertices);
    cugraph::betweenness_centrality(G, d_result.data().get(),
                                    normalize, endpoints,
                                    static_cast<WT*>(nullptr),
                                    configuration.number_of_sources_,
                                    sources_ptr,
                                    cugraph::cugraph_bc_implem_t::CUGRAPH_DEFAULT);
    hipDeviceSynchronize();
    std::cout << "[DBG][BC] CUGRAPH IS DONE COMPUTING" << std::endl;
    hipMemcpy(result.data(), d_result.data().get(),
               sizeof(result_t) * G.number_of_vertices,
               hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for (int i = 0 ; i < G.number_of_vertices ; ++i)
      EXPECT_NEAR(result[i], expected[i], TEST_EPSILON) <<
                  "[MISMATCH] vaid = " << i << ", cugraph = " <<
                  result[i] << " expected = " << expected[i];
    std::cout << "[DBG][BC] Perfect math over " << G.number_of_vertices << std::endl;
  }



};

struct BetweennessCentralityBFSTest : public ::testing::Test
{
};


/*
// BFS: Checking for shortest_path counting correctness
// -----------------------------------------------------------------------------
// TODO(xcadet) Parametrize this part for VT, ET, WT, result_t

TEST_F(BetweennessCentralityBFSTest, CheckReference) {
  // TODO(xcadet) This dataset was manually generated and is not provided
  //std::string matrix_file(get_rapids_dataset_root_dir() + "/" + "email-Eu-core-gen.mtx");
  std::string matrix_file("../../datasets/email-Eu-core-gen.mtx");
  //std::string matrix_file("../../datasets/karate-directed.mtx");
  int m, nnz;
  CSR_Result_Weighted<int, float> csr_result;
  generate_graph_csr<int, int, float>(csr_result, m, nnz, matrix_file);
  cugraph::experimental::GraphCSR<int, int, float> graph(csr_result.rowOffsets,
                                                     csr_result.colIndices,
                                                     csr_result.edgeWeights,
                                                     m, nnz);
  // FIXME: THIS IS CRITICAL:
  graph.prop.directed = true;
  std::vector<float> result(graph.number_of_vertices);

  //int source = 2;
  int source = 12;
  // Ref BC_BFS requires many working values
  int number_of_vertices = graph.number_of_vertices;
  int number_of_edges = graph.number_of_edges;
  //
  std::vector<int> indices(number_of_edges);
  std::vector<int> offsets(number_of_vertices + 1);

  hipMemcpy(indices.data(), graph.indices,
             sizeof(int) * indices.size(), hipMemcpyDeviceToHost);
  hipMemcpy(offsets.data(), graph.offsets,
             sizeof(int) * offsets.size(), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  std::queue<int> Q;
  std::stack<int> S;
  std::vector<int> ref_bfs_dist(number_of_vertices);
  std::vector<std::vector<int>> ref_bfs_pred(number_of_vertices);
  std::vector<float> ref_bfs_sigmas(number_of_vertices);

  ref_bfs<int, int, float, float>(indices.data(), offsets.data(),
                                  number_of_vertices, Q, S,
                                  ref_bfs_dist, ref_bfs_pred,
                                  ref_bfs_sigmas, source);



  // Device data for cugraph_bfs
  thrust::device_vector<int> d_cugraph_dist(number_of_vertices);
  thrust::device_vector<int> d_cugraph_pred(number_of_vertices);
  thrust::device_vector<int> d_cugraph_sigmas(number_of_vertices);

  // This test only checks for sigmas equality
  std::vector<int> cugraph_sigmas(number_of_vertices);

  printf("Is graph directed ? %d\n", graph.prop.directed);
  cugraph::bfs<int, int, float>(graph, d_cugraph_dist.data().get(),
                                d_cugraph_pred.data().get(),
                                d_cugraph_sigmas.data().get(),
                                source, graph.prop.directed);
  hipMemcpy(cugraph_sigmas.data(), d_cugraph_sigmas.data().get(),
             sizeof(int) * d_cugraph_sigmas.size(), hipMemcpyDeviceToHost);
  // TODO(xcadet): The implicit cast comes from BFS shortest_path counter being
  // of type VT, while the ref_bfs uses float values
  for (int i = 0 ; i < number_of_vertices ; ++i) {
    EXPECT_TRUE(compare_close((float)cugraph_sigmas[i], ref_bfs_sigmas[i], 0.0001)) <<
                "[MISMATCH] vaid = " << i << ", cugraph = " <<
                cugraph_sigmas[i] << " c++ ref = " << ref_bfs_sigmas[i];
    //std::cout << "Sigmas[" << i << "] = " << cugraph_sigmas[i] << std::endl;
  }
  std::cout << "Graph number_of_vertices " << number_of_vertices << ", number_of_edges " << number_of_edges << std::endl;
  int sum_sigmas_cugraph = thrust::reduce(thrust::host, cugraph_sigmas.begin(), cugraph_sigmas.end(), 0);
  int sum_sigmas_ref = thrust::reduce(thrust::host, ref_bfs_sigmas.begin(), ref_bfs_sigmas.end(), 0);
  std::cout << "Source " << source << ", cugraph: " << sum_sigmas_cugraph << ", ref " << sum_sigmas_ref << std::endl;;
}
*/


// BC
// -----------------------------------------------------------------------------
/*
TEST_F(BetweennessCentralityTest, SimpleGraph)
{
  std::vector<int>  graph_offsets{ { 0, 1, 2, 5, 7, 10, 12, 14 } };
  std::vector<int>  graph_indices{ { 2, 2, 0, 1, 3, 2, 4, 3, 5, 6, 4, 6, 4, 5 } };

  std::vector<float> expected{ {0.0, 0.0, 0.6, 0.6, 0.5333333, 0.0, 0.0 } };

  int num_verts = graph_offsets.size() - 1;
  int num_edges = graph_indices.size();

  thrust::device_vector<int>    d_graph_offsets(graph_offsets);
  thrust::device_vector<int>    d_graph_indices(graph_indices);
  thrust::device_vector<float>  d_result(num_verts);

  std::vector<float>            result(num_verts);

  cugraph::experimental::GraphCSR<int,int,float> G(d_graph_offsets.data().get(),
                                                   d_graph_indices.data().get(),
                                                   nullptr,
                                                   num_verts,
                                                   num_edges);

  cugraph::betweenness_centrality(G, d_result.data().get());

  hipMemcpy(result.data(), d_result.data().get(), sizeof(float) * num_verts, hipMemcpyDeviceToHost);

  for (int i = 0 ; i < num_verts ; ++i)
    EXPECT_FLOAT_EQ(result[i], expected[i]);

  // TODO(xcadet) Remove this part, it is for testing the reference
  std::vector<float> ref_result(num_verts);
  reference_betweenness_centrality(G, ref_result.data(), true);
  for (int i = 0 ; i < num_verts ; ++i)
    EXPECT_FLOAT_EQ(ref_result[i], expected[i]);
}
*/
// Verifiy Un-Normalized results
TEST_P(Tests_BC, CheckFP32_NO_NORMALIZE_NO_ENDPOINTS) {
  run_current_test<int, int, float, float, false, false>(GetParam());
}

TEST_P(Tests_BC, CheckFP64_NO_NORMALIZE_NO_ENDPOINTS) {
  run_current_test<int, int, float, float, false, false>(GetParam());
}

// Verifiy Normalized results
TEST_P(Tests_BC, CheckFP32_NORMALIZE_NO_ENPOINTS) {
  run_current_test<int, int, float, float, true, false>(GetParam());
}

TEST_P(Tests_BC, CheckFP64_NORMALIZE_NO_ENPOINTS) {
  run_current_test<int, int, float, float, true, false>(GetParam());
}

// FIXME: There is an InvalidValue on a Memcopy only on tests/datasets/dblp.mtx
INSTANTIATE_TEST_CASE_P(
  simple_test,
  Tests_BC,
  ::testing::Values(
      BC_Usecase("test/datasets/karate.mtx", 0),
      BC_Usecase("test/datasets/karate.mtx", 4),
      BC_Usecase("test/datasets/karate.mtx", 10),
      BC_Usecase("test/datasets/polbooks.mtx", 0),
      BC_Usecase("test/datasets/polbooks.mtx", 4),
      BC_Usecase("test/datasets/polbooks.mtx", 10),
      BC_Usecase("test/datasets/netscience.mtx", 0),
      BC_Usecase("test/datasets/netscience.mtx", 4),
      BC_Usecase("test/datasets/netscience.mtx", 100),
      BC_Usecase("test/datasets/wiki2003.mtx", 100),
      BC_Usecase("test/datasets/wiki2003.mtx", 1000)
    )
);


int main( int argc, char** argv )
{
  rmmInitialize(nullptr);
  testing::InitGoogleTest(&argc,argv);
  int rc = RUN_ALL_TESTS();
  rmmFinalize();
  return rc;
}