#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gtest/gtest.h"
#include "gmock/gmock.h"

#include <thrust/device_vector.h>
#include "test_utils.h"
#include <utility>

#include <graph.hpp>
#include <algorithms.hpp>

#include <queue> // C++ Reference Algorithm
#include <stack> // C++ Reference Algorithm

#include <converters/COOtoCSR.cuh> // Loads GraphCSR from .mtx
#include <fstream>

#ifndef TEST_EPSILON
 #define TEST_EPSILON 0.0001
#endif

// NOTE: Defines under which values the difference should  be discarded when
// considering values are close to zero
//  i.e: Do we consider that the difference between 1.3e-9 and 8.e-12 is
// significant
# ifndef TEST_ZERO_THRESHOLD
 #define TEST_ZERO_THRESHOLD 1e-10
#endif


// =============================================================================
// C++ Reference Implementation
// =============================================================================
template<typename VT, typename ET>
void populate_neighbors(VT *indices, ET *offsets,
                        VT w, std::vector<VT> &neighbors) {
  ET edge_start = offsets[w];
  ET edge_end = offsets[w + 1];
  ET edge_count = edge_end - edge_start;

  neighbors.clear(); // Reset neighbors vector's size
  for (ET edge_idx = 0; edge_idx < edge_count; ++edge_idx) {
    VT dst = indices[edge_start + edge_idx];
    neighbors.push_back(dst);
  }
}

// TODO: This colud be moved to BFS testing on the c++ side
// This implements the BFS from (Brandes, 2001) with shortest path counting
template<typename VT, typename ET, typename WT, typename result_t>
void ref_bfs(VT *indices, ET *offsets, VT const number_of_vertices,
             std::queue<VT> &Q,
             std::stack<VT> &S,
             std::vector<VT> &dist,
             std::vector<std::vector<VT>> &pred,
             std::vector<double> &sigmas,
             VT source) {
  std::vector<VT> neighbors;
  for (VT w = 0 ; w < number_of_vertices; ++w) {
    pred[w].clear();
    dist[w] = std::numeric_limits<VT>::max();
    sigmas[w] = 0;
  }
  dist[source] = 0;
  sigmas[source] = 1;
  Q.push(source);
  //   b. Traversal
  while (!Q.empty()) {
    VT v = Q.front();
    Q.pop();
    S.push(v);
    populate_neighbors<VT, ET>(indices, offsets, v, neighbors);
    for (VT w : neighbors) {
      // Path Discovery:
      // Found for the first time?
      if (dist[w] == std::numeric_limits<VT>::max()) {
        dist[w] = dist[v] + 1;
        Q.push(w);
      }
      // Path counting
      // Edge(v, w) on  a shortest path?
      if (dist[w] == dist[v] + 1) {
        sigmas[w] +=  sigmas[v];
        pred[w].push_back(v);
      }
    }
  }
}

template<typename VT, typename ET, typename WT, typename result_t>
void ref_accumulation(result_t *result,
                      VT const number_of_vertices,
                      std::stack<VT> &S,
                      std::vector<std::vector<VT>> &pred,
                      std::vector<double> &sigmas,
                      std::vector<result_t> &deltas,
                      VT source) {
  for (VT v = 0; v < number_of_vertices; ++v) {
    deltas[v] = 0;
  }
  while (!S.empty()) {
    VT w = S.top();
    S.pop();
    for (VT v : pred[w]) {
      deltas[v] += (sigmas[v] / sigmas[w]) * (1.0 + deltas[w]);
    }
    if (w != source) {
      result[w] += deltas[w];
    }
  }
}

// Algorithm 1: Shortest-path vertex betweenness, (Brandes, 2001)
template <typename VT, typename ET, typename WT, typename result_t>
void reference_betweenness_centrality_impl(VT *indices, ET *offsets,
                                           VT const number_of_vertices,
                                           result_t *result,
                                           VT const *sources,
                                           VT const number_of_sources) {
  std::queue<VT> Q;
  std::stack<VT> S;
  // NOTE: dist is of type VT not WT
  std::vector<VT> dist(number_of_vertices);
  std::vector<std::vector<VT>> pred(number_of_vertices);
  std::vector<double> sigmas(number_of_vertices);
  std::vector<result_t> deltas(number_of_vertices);

  std::vector<VT> neighbors;

  if (sources) {
    for (VT source_idx = 0; source_idx < number_of_sources; ++source_idx) {
      VT s = sources[source_idx];
      // Step 1: Single-source shortest-paths problem
      //   a. Initialization
      ref_bfs<VT, ET, WT, result_t>(indices, offsets, number_of_vertices,
                                    Q, S,
                                    dist, pred, sigmas, s);
      //  Step 2: Accumulation
      //          Back propagation of dependencies
      ref_accumulation<VT, ET, WT, result_t>(result,
                                             number_of_vertices,
                                             S,
                                             pred,
                                             sigmas,
                                             deltas,
                                             s);
    }
  } else {
    for (VT s = 0; s < number_of_vertices; ++s) {
      // Step 1: Single-source shortest-paths problem
      //   a. Initialization
      ref_bfs<VT, ET, WT, result_t>(indices, offsets, number_of_vertices,
                                    Q, S,
                                    dist, pred, sigmas, s);
      //  Step 2: Accumulation
      //          Back propagation of dependencies
      ref_accumulation<VT, ET, WT, result_t>(result,
                                             number_of_vertices,
                                             S,
                                             pred,
                                             sigmas,
                                             deltas,
                                             s);
    }
  }
}

template <typename VT, typename ET, typename WT, typename result_t>
void reference_rescale(result_t *result, bool normalize, bool directed, VT const number_of_vertices, VT const number_of_sources) {
  bool modified = false;
  result_t rescale_factor = static_cast<result_t>(1);
  result_t casted_number_of_sources = static_cast<result_t>(number_of_sources);
  result_t casted_number_of_vertices = static_cast<result_t>(number_of_vertices);
  if (normalize) {
    if (number_of_vertices > 2) {
      rescale_factor /= ((casted_number_of_vertices - 1) * (casted_number_of_vertices - 2));
      modified = true;
    }
  } else {
    if (!directed) {
      rescale_factor /= static_cast<result_t>(2);
      modified = true;
    }
  }
  if (modified) {
    if (number_of_sources > 0) {
      rescale_factor *= (casted_number_of_vertices / casted_number_of_sources);
    }
  }
  for (auto idx = 0; idx < number_of_vertices; ++idx) {
    result[idx] *=  rescale_factor;
  }
}

template <typename VT, typename ET, typename WT, typename result_t>
void reference_betweenness_centrality(cugraph::experimental::GraphCSR<VT, ET, WT> const &graph,
                                      result_t *result,
                                      bool normalize,
                                      bool endpoints, // This is not yet implemented
                                      VT const number_of_sources,
                                      VT const *sources) {

  VT number_of_vertices = graph.number_of_vertices;
  ET number_of_edges = graph.number_of_edges;
  thrust::host_vector<VT> h_indices(number_of_edges);
  thrust::host_vector<ET> h_offsets(number_of_vertices + 1);

  thrust::device_ptr<VT>  d_indices((VT *)&graph.indices[0]);
  thrust::device_ptr<ET>  d_offsets((ET *)&graph.offsets[0]);

  thrust::copy(d_indices, d_indices + number_of_edges, h_indices.begin());
  thrust::copy(d_offsets, d_offsets + (number_of_vertices + 1), h_offsets.begin());

  hipDeviceSynchronize();

  reference_betweenness_centrality_impl<VT, ET, WT, result_t>(&h_indices[0],
                                                              &h_offsets[0],
                                                              number_of_vertices,
                                                              result, sources,
                                                              number_of_sources);
  reference_rescale<VT, ET, WT, result_t>(result, normalize, graph.prop.directed, number_of_vertices, number_of_sources);
}
// Explicit declaration
template void reference_betweenness_centrality<int, int, float, float>(cugraph::experimental::GraphCSR<int, int, float> const&,
                                                                  float *, bool, bool, const int, int const *);
template void reference_betweenness_centrality<int, int, double, double>(cugraph::experimental::GraphCSR<int, int, double> const&,
                                                                  double *, bool, bool, const int, int const *);

// =============================================================================
// Utility functions
// =============================================================================
// TODO: This could be useful in other testsuite (SSSP, BFS, ...)
template<typename VT, typename ET, typename WT>
void generate_graph_csr(CSR_Result_Weighted<VT, WT> &csr_result, VT &m, VT &nnz, bool &is_directed, std::string matrix_file) {
  FILE* fpin = fopen(matrix_file.c_str(),"r");
  ASSERT_NE(fpin, nullptr) << "fopen (" << matrix_file << ") failure.";

  VT k;
  MM_typecode mc;
  ASSERT_EQ(mm_properties<VT>(fpin, 1, &mc, &m, &k, &nnz),0) << "could not read Matrix Market file properties"<< "\n";
  ASSERT_TRUE(mm_is_matrix(mc));
  ASSERT_TRUE(mm_is_coordinate(mc));
  ASSERT_FALSE(mm_is_complex(mc));
  ASSERT_FALSE(mm_is_skew(mc));
  is_directed = !mm_is_symmetric(mc);

  // Allocate memory on host
  std::vector<VT> cooRowInd(nnz), cooColInd(nnz);
  std::vector<WT> cooVal(nnz);

  // Read
  ASSERT_EQ( (mm_to_coo<VT, WT>(fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], &cooVal[0], NULL)) , 0)<< "could not read matrix data"<< "\n";
  ASSERT_EQ(fclose(fpin),0);

  ConvertCOOtoCSR_weighted(&cooRowInd[0], &cooColInd[0], &cooVal[0], nnz, csr_result);
  CUDA_CHECK_LAST();
}

// Compare while allowing relatie error of epsilon
// zero_threshold indicates when  we should drop comparison for small numbers
template <typename T, typename precision_t>
bool compare_close(const T &a, const T&b, const precision_t epsilon,
                   precision_t zero_threshold) {
  return ((zero_threshold > a && zero_threshold > b))
          || (a >= b * (1.0 - epsilon)) && (a <= b * (1.0 + epsilon));
}

// =============================================================================
// Test Suite
// =============================================================================
// Defines Betweenness Centrality UseCase
// SSSP's test suite codes uses type of Graph parameter that could be used
// (MTX / RMAT)
//TODO: Use VT for number_of_sources
typedef struct BC_Usecase_t {
  std::string config_;
  std::string file_path_;
  int number_of_sources_;
  BC_Usecase_t(const std::string& config, int number_of_sources)
               : config_(config), number_of_sources_(number_of_sources) {
    // assume relative paths are relative to RAPIDS_DATASET_ROOT_DIR
    // FIXME: Use platform independent stuff from c++14/17 on compiler update
    const std::string& rapidsDatasetRootDir = get_rapids_dataset_root_dir();
    if ((config_ != "") && (config_[0] != '/')) {
      file_path_ = rapidsDatasetRootDir + "/" + config_;
    } else {
      file_path_ = config_;
    }
  };
} BC_Usecase;

class Tests_BC : public ::testing::TestWithParam<BC_Usecase> {
  public:
  Tests_BC() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}
  // TODO(xcadet) Should normalize be part of the configuration?
  template <typename VT, typename ET, typename WT, typename result_t,
            bool normalize, bool endpoints>
  void run_current_test(const BC_Usecase &configuration) {
    // Step 1: Construction of the graph based on configuration
    VT m;
    ET nnz;
    CSR_Result_Weighted<VT, WT> csr_result;
    bool is_directed = false;
    generate_graph_csr<VT, ET, WT>(csr_result, m, nnz, is_directed,
                                   configuration.file_path_);
    hipDeviceSynchronize();
    cugraph::experimental::GraphCSR<VT, ET, WT> G(csr_result.rowOffsets,
                                                  csr_result.colIndices,
                                                  csr_result.edgeWeights,
                                                  m, nnz);
    G.prop.directed = is_directed;

    CUDA_CHECK_LAST();
    std::vector<result_t> result(G.number_of_vertices, 0);
    std::vector<result_t> expected(G. number_of_vertices, 0);

    // Step 2: Generation of sources based on configuration
    //         if number_of_sources_ is 0 then sources must be nullptr
    //         Otherwise we only  use the first k values
    ASSERT_TRUE(configuration.number_of_sources_ >= 0
           && configuration.number_of_sources_ <= G.number_of_vertices)
           << "Number number of sources should be >= 0 and"
           << " less than the number of vertices in the graph";
    std::vector<VT> sources(configuration.number_of_sources_);
    std::iota(sources.begin(), sources.end(), 0);

    VT *sources_ptr = nullptr;
    if (configuration.number_of_sources_ > 0) {
      sources_ptr = sources.data();
    }

    reference_betweenness_centrality(G, expected.data(),
                                     normalize, endpoints,
                                     // TODO: weights
                                     configuration.number_of_sources_,
                                     sources_ptr);

    sources_ptr = nullptr;
    if (configuration.number_of_sources_ > 0) {
      sources_ptr = sources.data();
    }

    thrust::device_vector<result_t>  d_result(G.number_of_vertices);
    cugraph::betweenness_centrality(G, d_result.data().get(),
                                    normalize, endpoints,
                                    static_cast<WT*>(nullptr),
                                    configuration.number_of_sources_,
                                    sources_ptr,
                                    cugraph::cugraph_bc_implem_t::CUGRAPH_DEFAULT);
    hipDeviceSynchronize();
    CUDA_TRY(hipMemcpy(result.data(), d_result.data().get(),
               sizeof(result_t) * G.number_of_vertices,
               hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    for (int i = 0 ; i < G.number_of_vertices ; ++i)
      EXPECT_TRUE(compare_close(result[i], expected[i], TEST_EPSILON, TEST_ZERO_THRESHOLD)) <<
                  "[MISMATCH] vaid = " << i << ", cugraph = " <<
                  result[i] << " expected = " << expected[i];
  }
};

// BFS: Checking for shortest_path counting correctness
// -----------------------------------------------------------------------------
// TODO: For now this BFS testing is done here, as the tests mostly focused
// around shortest path counting. It should probably used as a part of a
// C++ test suite
class Tests_BFS : public ::testing::TestWithParam<BC_Usecase> {
  public:
  Tests_BFS() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}
  template <typename VT, typename ET, typename WT, typename result_t>
  void run_current_test(const BC_Usecase &configuration) {
    // Step 1: Construction of the graph based on configuration
    VT m;
    ET nnz;
    CSR_Result_Weighted<VT, WT> csr_result;
    bool is_directed = false;
    generate_graph_csr<VT, ET, WT>(csr_result, m, nnz, is_directed,
                                   configuration.file_path_);
    hipDeviceSynchronize();
    cugraph::experimental::GraphCSR<VT, ET, WT> G(csr_result.rowOffsets,
                                                  csr_result.colIndices,
                                                  csr_result.edgeWeights,
                                                  m, nnz);
    G.prop.directed = is_directed;

    CUDA_CHECK_LAST();
    std::vector<result_t> result(G.number_of_vertices, 0);
    std::vector<result_t> expected(G. number_of_vertices, 0);

    // Step 2: Generation of sources based on configuration
    //         if number_of_sources_ is 0 then sources must be nullptr
    //         Otherwise we only  use the first k values
    ASSERT_TRUE(configuration.number_of_sources_ >= 0
           && configuration.number_of_sources_ <= G.number_of_vertices)
           << "Number number of sources should be >= 0 and"
           << " less than the number of vertices in the graph";

    //TODO(xcadet) Make it generic again (it made it easier to check)
    VT source = configuration.number_of_sources_;

    VT number_of_vertices = G.number_of_vertices;
    ET number_of_edges = G.number_of_edges;
    std::vector<VT> indices(number_of_edges);
    std::vector<ET> offsets(number_of_vertices + 1);

    CUDA_TRY(hipMemcpy(indices.data(), G.indices,
              sizeof(VT) * indices.size(), hipMemcpyDeviceToHost));
    CUDA_TRY(hipMemcpy(offsets.data(), G.offsets,
              sizeof(ET) * offsets.size(), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    std::queue<VT> Q;
    std::stack<VT> S;
    std::vector<VT> ref_bfs_dist(number_of_vertices);
    std::vector<std::vector<VT>> ref_bfs_pred(number_of_vertices);
    std::vector<double> ref_bfs_sigmas(number_of_vertices);

    ref_bfs<VT, ET, WT, result_t>(indices.data(), offsets.data(),
                                    number_of_vertices, Q, S,
                                    ref_bfs_dist, ref_bfs_pred,
                                    ref_bfs_sigmas, source);

    // Device data for cugraph_bfs
    thrust::device_vector<VT> d_cugraph_dist(number_of_vertices);
    thrust::device_vector<VT> d_cugraph_pred(number_of_vertices);
    thrust::device_vector<double> d_cugraph_sigmas(number_of_vertices);

    // This test only checks for sigmas equality
    std::vector<double> cugraph_sigmas(number_of_vertices);

    printf("Is graph directed ? %d\n", G.prop.directed);
    cugraph::bfs<VT, ET, WT>(G, d_cugraph_dist.data().get(),
                                  d_cugraph_pred.data().get(),
                                  d_cugraph_sigmas.data().get(),
                                  source, G.prop.directed);
    CUDA_TRY(hipMemcpy(cugraph_sigmas.data(), d_cugraph_sigmas.data().get(),
              sizeof(double) * d_cugraph_sigmas.size(), hipMemcpyDeviceToHost));
    // TODO(xcadet): The implicit cast comes from BFS shortest_path counter being
    // of type VT, while the ref_bfs uses float values
    for (int i = 0 ; i < number_of_vertices ; ++i) {
      EXPECT_TRUE(compare_close(cugraph_sigmas[i], ref_bfs_sigmas[i], TEST_EPSILON, TEST_ZERO_THRESHOLD)) <<
                  "[MISMATCH] vaid = " << i << ", cugraph = " <<
                  cugraph_sigmas[i] << " c++ ref = " << ref_bfs_sigmas[i];
    }
  }
};
//==============================================================================
// Tests
//==============================================================================
// BC
// -----------------------------------------------------------------------------
// Verifiy Un-Normalized results
// Endpoint parameter is currently not usefull, is for later use
TEST_P(Tests_BC, CheckFP32_NO_NORMALIZE_NO_ENDPOINTS) {
  run_current_test<int, int, float, float, false, false>(GetParam());
}

TEST_P(Tests_BC, CheckFP64_NO_NORMALIZE_NO_ENDPOINTS) {
  run_current_test<int, int, double, double, false, false>(GetParam());
}

// Verifiy Normalized results
TEST_P(Tests_BC, CheckFP32_NORMALIZE_NO_ENPOINTS) {
  run_current_test<int, int, float, float, true, false>(GetParam());
}

TEST_P(Tests_BC, CheckFP64_NORMALIZE_NO_ENPOINTS) {
  run_current_test<int, int, double, double, true, false>(GetParam());
}

// FIXME: There is an InvalidValue on a Memcopy only on tests/datasets/dblp.mtx
INSTANTIATE_TEST_CASE_P(
  simple_test,
  Tests_BC,
  ::testing::Values(
      BC_Usecase("test/datasets/karate.mtx", 0),
      BC_Usecase("test/datasets/polbooks.mtx", 0),
      BC_Usecase("test/datasets/netscience.mtx", 0),
      BC_Usecase("test/datasets/netscience.mtx", 100),
      BC_Usecase("test/datasets/wiki2003.mtx", 4),
      BC_Usecase("test/datasets/wiki-Talk.mtx", 4)
    )
);

// BFS
// -----------------------------------------------------------------------------
// TODO(xcadet): This should be specialized for BFS
TEST_P(Tests_BFS, CheckFP32_NO_NORMALIZE_NO_ENDPOINTS) {
  run_current_test<int, int, float, float>(GetParam());
}

TEST_P(Tests_BFS, CheckFP64_NO_NORMALIZE_NO_ENDPOINTS) {
  run_current_test<int, int, double, double>(GetParam());
}

INSTANTIATE_TEST_CASE_P(
  simple_test,
  Tests_BFS,
  ::testing::Values(
    BC_Usecase("test/datasets/karate.mtx", 0),
    BC_Usecase("test/datasets/polbooks.mtx", 0),
    BC_Usecase("test/datasets/netscience.mtx", 0),
    BC_Usecase("test/datasets/netscience.mtx", 100),
    BC_Usecase("test/datasets/wiki2003.mtx", 1000),
    BC_Usecase("test/datasets/wiki-Talk.mtx", 1000)
  )
);

int main( int argc, char** argv )
{
  rmmInitialize(nullptr);
  testing::InitGoogleTest(&argc,argv);
  int rc = RUN_ALL_TESTS();
  rmmFinalize();
  return rc;
}
