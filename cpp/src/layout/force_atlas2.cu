/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <string>
#include <sstream>
#include <iostream>
#include <iomanip>
#include "hipcub/hipcub.hpp"
#include <algorithm>
#include <iomanip>

#include <rmm_utils.h>

#include "utilities/graph_utils.cuh"
#include "utilities/error_utils.h"
#include <cugraph.h>
#include <graph.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_buffer.hpp>

#include "barnes_hut.h" 
#include "exact_fa2.h"

namespace cugraph {

template <typename VT, typename ET, typename WT>
void force_atlas2(experimental::GraphCSR<VT, ET, WT> const &graph,
                  float *x_pos, float *y_pos, const int max_iter,
                  float *x_start,
                  float *y_start, bool outbound_attraction_distribution,
                  bool lin_log_mode, bool prevent_overlapping,
                  const float edge_weight_influence,
                  const float jitter_tolerance, bool barnes_hut_optimize,
                  const float barnes_hut_theta, const float scaling_ratio,
                  bool strong_gravity_mode, const float gravity) {

    CUGRAPH_EXPECTS( x_pos != nullptr,
            "Invid API parameter: X_pos array should be of size V" );
    CUGRAPH_EXPECTS( y_pos != nullptr ,
            "Invid API parameter: Y_pos array should be of size V" );

    const VT *csrPtr = graph.offsets;
    const ET *csrInd = graph.indices;
    const WT *v = graph.edge_data;
    const VT n = graph.number_of_vertices;

    if (barnes_hut_optimize) {
        cugraph::detail::barnes_hut<VT, ET, WT>(csrPtr, csrInd, v, n,
                x_pos, y_pos, max_iter, x_start,
                y_start, outbound_attraction_distribution,
                lin_log_mode, prevent_overlapping, edge_weight_influence,
                jitter_tolerance,barnes_hut_theta, scaling_ratio,
                strong_gravity_mode, gravity);
    } else {
        cugraph::detail::exact_fa2<VT, ET, WT>(csrPtr, csrInd, v, n,
                x_pos, y_pos, max_iter, x_start,
                y_start, outbound_attraction_distribution,
                lin_log_mode, prevent_overlapping, edge_weight_influence,
                jitter_tolerance, scaling_ratio,
                strong_gravity_mode, gravity);
    }
}

template void force_atlas2<int, int, float>(
        experimental::GraphCSR<int, int, float> const &graph,
        float *x_pos, float *y_pos, const int max_iter,
        float *x_start, float *y_start,
        bool outbound_attraction_distribution,
        bool lin_log_mode, bool prevent_overlapping,
        const float edge_weight_influence, const float jitter_tolerance,
        bool barnes_hut_optimize, const float barnes_hut_theta,
        const float scaling_ratio, bool strong_gravity_mode,
        const float gravity);

template void cugraph::detail::barnes_hut<int, int, float>(
        const int *csrPtr, const int *csrInd, const float *v, const int n,
        float *x_pos, float *y_pos, int max_iter,
        float *x_start, float * y_start,
        bool outbount_attraction_distribution,
        bool lin_log_mode, bool prevent_overlapping,
        const float edge_weight_influence, const float jitter_tolerance,
        const float barnes_hut_theta, const float scaling_ratio,
        bool strong_gravity_mode,
		const float gravity);

template void cugraph::detail::exact_fa2<int, int, float>(
        const int *csrPtr, const int *csrInd, const float *v, const int n,
        float *x_pos, float *y_pos, const int max_iter,
        float *x_start, float *y_start,
        bool outbound_attraction_distribution,
        bool lin_log_mode, bool prevent_overlapping,
        const float edge_weight_influence, const float jitter_tolerance,
        const float scaling_ratio, bool strong_gravity_mode,
        const float gravity);

} // namespace cugraph
