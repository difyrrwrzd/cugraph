#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <raft/handle.hpp>
#include "load_balance.cuh"
#include "bfs_comms.cuh"

namespace cugraph {

namespace detail {

namespace opg {

struct bitwise_or {
  __device__ unsigned operator()(unsigned& a, unsigned & b) { return a | b; }
};

struct remove_visited {
  __device__ unsigned operator()(unsigned& visited, unsigned& output) {
    //OUTPUT AND VISITED - common bits between output and visited
    //OUTPUT AND (NOT (OUTPUT AND VISITED))
    // - remove common bits between output and visited from output
    return (output & (~( output & visited )));
  }
};

template <typename VT>
struct bfs_frontier_pred {
  unsigned * output_frontier_;
  VT * predecessors_;

  bfs_frontier_pred(
      unsigned * output_frontier,
      VT * predecessors) :
    output_frontier_(output_frontier),
    predecessors_(predecessors) {}

  __device__ void operator()(VT src, VT dst) {
    unsigned active_bit = static_cast<unsigned>(1)<<(dst % BitsPWrd<unsigned>);
    unsigned prev_word =
      atomicOr(output_frontier_ + (dst/BitsPWrd<unsigned>), active_bit);
    //If this thread activates the frontier bitmap for a destination
    //then the source is the predecessor of that destination
    if (prev_word & active_bit == 0) {
      predecessors_[dst] = src;
    }
  }
};

template <typename VT>
struct bfs_frontier_pred_dist {
  unsigned * output_frontier_;
  VT * predecessors_;
  VT * distances_;
  VT level_;

  bfs_frontier_pred_dist(
      unsigned * output_frontier,
      VT * predecessors,
      VT * distances, VT level) :
    output_frontier_(output_frontier),
    predecessors_(predecessors),
    distances_(distances),
    level_(level) {}

  __device__ void operator()(VT src, VT dst) {
    unsigned active_bit = static_cast<unsigned>(1)<<(dst % BitsPWrd<unsigned>);
    unsigned prev_word =
      atomicOr(output_frontier_ + (dst/BitsPWrd<unsigned>), active_bit);
    //If this thread activates the frontier bitmap for a destination
    //then the source is the predecessor of that destination
    if (prev_word & active_bit == 0) {
      distances_[dst] = level_;
      predecessors_[dst] = src;
    }
  }
};

struct is_not_equal {
  unsigned cmp_;
  unsigned * flag_;
  is_not_equal(unsigned cmp, unsigned * flag) : cmp_(cmp), flag_(flag) {}
  __device__ void operator()(unsigned& val) {
    if (val != cmp_) {
      *flag_ = 1;
    }
  }
};

template <typename VT, typename ET, typename WT>
void bfs(raft::handle_t const &handle,
    cugraph::experimental::GraphCSRView<VT, ET, WT>& graph,
    VT *distances,
    VT *predecessors,
    const VT start_vertex) {

  //We need to keep track if a vertex is visited or its status
  //This needs to be done for all the vertices in the global graph
  size_t word_count = number_of_words(graph.number_of_vertices);
  rmm::device_vector<unsigned> input_frontier(word_count);
  rmm::device_vector<unsigned> output_frontier(word_count);
  rmm::device_vector<unsigned> visited(word_count);

  rmm::device_vector<unsigned> frontier_not_empty(1);

  //Load balancer for calls to bfs functors
  LoadBalanceExecution<VT, ET, WT> lb(handle, graph);

  //Functor to check if frontier is empty
  is_not_equal neq(static_cast<unsigned>(0), frontier_not_empty.data().get());

  hipStream_t stream = handle.get_stream();

  //Fill predecessors with an invalid vertex id
  thrust::fill(rmm::exec_policy(stream)->on(stream),
      predecessors, predecessors + graph.number_of_vertices,
      graph.number_of_vertices);

  VT level = 0;
  if (distances != nullptr) {
    thrust::fill(rmm::exec_policy(stream)->on(stream),
        distances, distances + graph.number_of_vertices,
        std::numeric_limits<VT>::max());
  }

  //BFS communications wrapper
  BFSCommunicator<VT, ET, WT> bfs_comm(handle, word_count);

  //0. 'Insert' starting vertex in the input frontier
  input_frontier[start_vertex/BitsPWrd<unsigned>] =
    static_cast<unsigned>(1)<<(start_vertex%BitsPWrd<unsigned>);

  do {
    //1. Mark all input frontier vertices as visited
    thrust::transform(rmm::exec_policy(stream)->on(stream),
        input_frontier.begin(), input_frontier.end(),
        visited.begin(),
        visited.begin(),
        bitwise_or());

    //2. Clear out output frontier
    thrust::fill(
        output_frontier.begin(),
        output_frontier.end(),
        static_cast<unsigned>(0));

    //3. Create output frontier from input frontier
    if (distances != nullptr) {
      //BFS Functor for frontier calculation
      bfs_frontier_pred_dist<VT> bfs_op(
          output_frontier.data().get(), predecessors, distances, level++);
      lb.run(bfs_op, input_frontier.data().get());
    } else {
      //BFS Functor for frontier calculation
      bfs_frontier_pred<VT> bfs_op(
          output_frontier.data().get(), predecessors);
      lb.run(bfs_op, input_frontier.data().get());
    }

    //3a. Combine output frontier from all GPUs
    bfs_comm.allreduce(output_frontier);

    //4. 'Remove' all vertices in output frontier
    //that are already visited
    thrust::transform(rmm::exec_policy(stream)->on(stream),
        visited.begin(), visited.end(),
        output_frontier.begin(),
        output_frontier.begin(),
        remove_visited());

    //5. Use the output frontier as input for the next step
    input_frontier.swap(output_frontier);

    //6. If all bits in input frontier are inactive then bfs is done
    frontier_not_empty[0] = 0;
    thrust::for_each(rmm::exec_policy(stream)->on(stream),
        input_frontier.begin(), input_frontier.end(),
        neq);
  } while (frontier_not_empty[0] == 1);

  //In place reduce to collect predecessors
  handle.get_comms().allreduce(
      predecessors, predecessors,
      graph.number_of_vertices,
      raft::comms::op_t::MIN,
      handle.get_stream());

  //If the bfs loop does not assign a predecessor for a vertex
  //then its value will be graph.number_of_vertices. This needs to be
  //replaced by invalid vertex id to denote that a vertex does have
  //a predecessor
  thrust::replace(rmm::exec_policy(stream)->on(stream),
      predecessors, predecessors + graph.number_of_vertices,
      graph.number_of_vertices,
      cugraph::experimental::invalid_vertex_id<VT>::value);

  if (distances != nullptr) {
    //In place reduce to collect predecessors
    handle.get_comms().allreduce(
        distances, distances,
        graph.number_of_vertices,
        raft::comms::op_t::MIN,
        handle.get_stream());
  }

}

template void bfs(raft::handle_t const &handle,
    cugraph::experimental::GraphCSRView<int, int, float> &graph,
    int *distances,
    int *predecessors,
    const int start_vertex);

template void bfs(raft::handle_t const &handle,
    cugraph::experimental::GraphCSRView<int, int, double> &graph,
    int *distances,
    int *predecessors,
    const int start_vertex);


}//namespace opg

}//namespace detail

}//namespace cugraph
