 /*
 * Copyright (c) 2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

// Graph analytics features 
// Author: Alex Fender afender@nvidia.com

#include <cugraph.h>
#include "graph_utils.cuh"
#include "pagerank.cuh"
#include "COOtoCSR.cuh"
#include "utilities/error_utils.h"
#include "bfs.cuh"

#include <hip/library_types.h>
#include <nvgraph/nvgraph.h>

#include <rmm_utils.h>

void gdf_col_delete(gdf_column* col) {
  if (col) {
    col->size = 0; 
    if(col->data) {
      ALLOC_FREE_TRY(col->data, nullptr);
    }
#if 1
// If delete col is executed, the memory pointed by col is no longer valid and
// can be used in another memory allocation, so executing col->data = nullptr
// after delete col is dangerous, also, col = nullptr has no effect here (the
// address is passed by value, for col = nullptr should work, the input
// parameter should be gdf_column*& col (or alternatively, gdf_column** col and
// *col = nullptr also work)
    col->data = nullptr;
    delete col;
#else
    delete col;
    col->data = nullptr;
    col = nullptr;
#endif
  }
}

void gdf_col_release(gdf_column* col) {
  delete col;
}

void cpy_column_view(const gdf_column *in, gdf_column *out) {
  if (in != nullptr && out !=nullptr) {
    gdf_column_view(out, in->data, in->valid, in->size, in->dtype);
  }
}

gdf_error gdf_adj_list_view(gdf_graph *graph, const gdf_column *offsets, 
                                 const gdf_column *indices, const gdf_column *edge_data) {
  GDF_REQUIRE( offsets->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );                    
  GDF_REQUIRE( indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( (offsets->dtype == indices->dtype), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( ((offsets->dtype == GDF_INT32) || (offsets->dtype == GDF_INT64)), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( (offsets->size > 0), GDF_DATASET_EMPTY ); 
  GDF_REQUIRE( (graph->adjList == nullptr) , GDF_INVALID_API_CALL);

  graph->adjList = new gdf_adj_list;
  graph->adjList->offsets = new gdf_column;
  graph->adjList->indices = new gdf_column;
  graph->adjList->ownership = 0;

  cpy_column_view(offsets, graph->adjList->offsets);
  cpy_column_view(indices, graph->adjList->indices);
  if (edge_data) {
      GDF_REQUIRE( indices->size == edge_data->size, GDF_COLUMN_SIZE_MISMATCH );
      graph->adjList->edge_data = new gdf_column;
      cpy_column_view(edge_data, graph->adjList->edge_data);
  }
  else {
    graph->adjList->edge_data = nullptr;
  }
  return GDF_SUCCESS;
}

gdf_error gdf_adj_list::get_vertex_identifiers(gdf_column *identifiers) {
  GDF_REQUIRE( offsets != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( offsets->data != nullptr , GDF_INVALID_API_CALL);
  cugraph::sequence<int>((int)offsets->size-1, (int*)identifiers->data);
  return GDF_SUCCESS;
}

gdf_error gdf_adj_list::get_source_indices (gdf_column *src_indices) {
  GDF_REQUIRE( offsets != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( offsets->data != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( src_indices->size == indices->size, GDF_COLUMN_SIZE_MISMATCH );
  GDF_REQUIRE( src_indices->dtype == indices->dtype, GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( src_indices->size > 0, GDF_DATASET_EMPTY ); 
  cugraph::offsets_to_indices<int>((int*)offsets->data, offsets->size-1, (int*)src_indices->data);

  return GDF_SUCCESS;
}

gdf_error gdf_edge_list_view(gdf_graph *graph, const gdf_column *src_indices, 
                                 const gdf_column *dest_indices, const gdf_column *edge_data) {
  GDF_REQUIRE( src_indices->size == dest_indices->size, GDF_COLUMN_SIZE_MISMATCH );
  GDF_REQUIRE( src_indices->dtype == dest_indices->dtype, GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( ((src_indices->dtype == GDF_INT32) || (src_indices->dtype == GDF_INT64)), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( src_indices->size > 0, GDF_DATASET_EMPTY ); 
  GDF_REQUIRE( src_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );                    
  GDF_REQUIRE( dest_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( graph->edgeList == nullptr , GDF_INVALID_API_CALL);

  graph->edgeList = new gdf_edge_list;
  graph->edgeList->src_indices = new gdf_column;
  graph->edgeList->dest_indices = new gdf_column;
  graph->edgeList->ownership = 0;

  cpy_column_view(src_indices, graph->edgeList->src_indices);
  cpy_column_view(dest_indices, graph->edgeList->dest_indices);
  if (edge_data) {
      GDF_REQUIRE( src_indices->size == edge_data->size, GDF_COLUMN_SIZE_MISMATCH );
      graph->edgeList->edge_data = new gdf_column;
      cpy_column_view(edge_data, graph->edgeList->edge_data);
  }
  else {
    graph->edgeList->edge_data = nullptr;
  }
  return GDF_SUCCESS;
}

template <typename WT>
gdf_error gdf_add_adj_list_impl (gdf_graph *graph) {
    if (graph->adjList == nullptr) {
      GDF_REQUIRE( graph->edgeList != nullptr , GDF_INVALID_API_CALL);
      int nnz = graph->edgeList->src_indices->size, status = 0;
      graph->adjList = new gdf_adj_list;
      graph->adjList->offsets = new gdf_column;
      graph->adjList->indices = new gdf_column;
      graph->adjList->ownership = 1;

    if (graph->edgeList->edge_data!= nullptr) {
      graph->adjList->edge_data = new gdf_column;

      CSR_Result_Weighted<int,WT> adj_list;
      status = ConvertCOOtoCSR_weighted((int*)graph->edgeList->src_indices->data, (int*)graph->edgeList->dest_indices->data, (WT*)graph->edgeList->edge_data->data, nnz, adj_list);
      
      gdf_column_view(graph->adjList->offsets, adj_list.rowOffsets, 
                            nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->indices, adj_list.colIndices, 
                            nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->edge_data, adj_list.edgeWeights, 
                          nullptr, adj_list.nnz, graph->edgeList->edge_data->dtype);
    }
    else {
      CSR_Result<int> adj_list;
      status = ConvertCOOtoCSR((int*)graph->edgeList->src_indices->data,(int*)graph->edgeList->dest_indices->data, nnz, adj_list);      
      gdf_column_view(graph->adjList->offsets, adj_list.rowOffsets, 
                            nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->indices, adj_list.colIndices, 
                            nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
    }
    if (status !=0) {
      std::cerr << "Could not generate the adj_list" << std::endl;
      return GDF_CUDA_ERROR;
    }
  }
  return GDF_SUCCESS;
}

gdf_error gdf_add_edge_list (gdf_graph *graph) {
    if (graph->edgeList == nullptr) {
      GDF_REQUIRE( graph->adjList != nullptr , GDF_INVALID_API_CALL);
      int *d_src;
      graph->edgeList = new gdf_edge_list;
      graph->edgeList->src_indices = new gdf_column;
      graph->edgeList->dest_indices = new gdf_column;
      graph->edgeList->ownership = 2;

      CUDA_TRY(hipMallocManaged ((void**)&d_src, sizeof(int) * graph->adjList->indices->size));

      cugraph::offsets_to_indices<int>((int*)graph->adjList->offsets->data, 
                                  graph->adjList->offsets->size-1, 
                                  (int*)d_src);

      gdf_column_view(graph->edgeList->src_indices, d_src, 
                      nullptr, graph->adjList->indices->size, graph->adjList->indices->dtype);
      cpy_column_view(graph->adjList->indices, graph->edgeList->dest_indices);
      
      if (graph->adjList->edge_data != nullptr) {
        graph->edgeList->edge_data = new gdf_column;
        cpy_column_view(graph->adjList->edge_data, graph->edgeList->edge_data);
      }
  }
  return GDF_SUCCESS;
}


template <typename WT>
gdf_error gdf_add_transposed_adj_list_impl (gdf_graph *graph) {
    if (graph->transposedAdjList == nullptr ) {
      GDF_REQUIRE( graph->edgeList != nullptr , GDF_INVALID_API_CALL);
      int nnz = graph->edgeList->src_indices->size, status = 0;
      graph->transposedAdjList = new gdf_adj_list;
      graph->transposedAdjList->offsets = new gdf_column;
      graph->transposedAdjList->indices = new gdf_column;
      graph->transposedAdjList->ownership = 1;
    
      if (graph->edgeList->edge_data) {
        graph->transposedAdjList->edge_data = new gdf_column;
        CSR_Result_Weighted<int,WT> adj_list;
        status = ConvertCOOtoCSR_weighted( (int*)graph->edgeList->dest_indices->data, (int*)graph->edgeList->src_indices->data, (WT*)graph->edgeList->edge_data->data, nnz, adj_list);
        gdf_column_view(graph->transposedAdjList->offsets, adj_list.rowOffsets, 
                              nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
        gdf_column_view(graph->transposedAdjList->indices, adj_list.colIndices, 
                              nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
        gdf_column_view(graph->transposedAdjList->edge_data, adj_list.edgeWeights, 
                            nullptr, adj_list.nnz, graph->edgeList->edge_data->dtype);
      }
      else {

        CSR_Result<int> adj_list;
        status = ConvertCOOtoCSR((int*)graph->edgeList->dest_indices->data, (int*)graph->edgeList->src_indices->data, nnz, adj_list);      
        gdf_column_view(graph->transposedAdjList->offsets, adj_list.rowOffsets, 
                              nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
        gdf_column_view(graph->transposedAdjList->indices, adj_list.colIndices, 
                              nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
      }
      if (status !=0) {
        std::cerr << "Could not generate the adj_list" << std::endl;
        return GDF_CUDA_ERROR;
      }
    }
    return GDF_SUCCESS;
}

template <typename WT>
gdf_error gdf_pagerank_impl (gdf_graph *graph,
                      gdf_column *pagerank, float alpha = 0.85,
                      float tolerance = 1e-4, int max_iter = 200,
                      bool has_guess = false) {
  GDF_REQUIRE( graph->edgeList != nullptr, GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( graph->edgeList->src_indices->size == graph->edgeList->dest_indices->size, GDF_COLUMN_SIZE_MISMATCH ); 
  GDF_REQUIRE( graph->edgeList->src_indices->dtype == graph->edgeList->dest_indices->dtype, GDF_UNSUPPORTED_DTYPE );  
  GDF_REQUIRE( graph->edgeList->src_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );                 
  GDF_REQUIRE( graph->edgeList->dest_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );  
  GDF_REQUIRE( pagerank != nullptr , GDF_INVALID_API_CALL ); 
  GDF_REQUIRE( pagerank->data != nullptr , GDF_INVALID_API_CALL ); 
  GDF_REQUIRE( pagerank->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );          
  GDF_REQUIRE( pagerank->size > 0 , GDF_INVALID_API_CALL );         

  int m=pagerank->size, nnz = graph->edgeList->src_indices->size, status = 0;
  WT *d_pr, *d_val = nullptr, *d_leaf_vector = nullptr; 
  WT res = 1.0;
  WT *residual = &res;

  if (graph->transposedAdjList == nullptr) {
    gdf_add_transposed_adj_list(graph);
  }
  hipStream_t stream{nullptr};
  ALLOC_MANAGED_TRY((void**)&d_leaf_vector, sizeof(WT) * m, stream);
  ALLOC_MANAGED_TRY((void**)&d_val, sizeof(WT) * nnz , stream);
  ALLOC_MANAGED_TRY((void**)&d_pr,    sizeof(WT) * m, stream);

  cugraph::HT_matrix_csc_coo(m, nnz, (int*)graph->transposedAdjList->offsets->data, (int*)graph->transposedAdjList->indices->data, d_val, d_leaf_vector);

  if (has_guess)
  {
    GDF_REQUIRE( pagerank->data != nullptr, GDF_VALIDITY_UNSUPPORTED );
    cugraph::copy<WT>(m, (WT*)pagerank->data, d_pr);
  }

  status = cugraph::pagerank<int,WT>( m,nnz, (int*)graph->transposedAdjList->offsets->data, (int*)graph->transposedAdjList->indices->data, 
    d_val, alpha, d_leaf_vector, false, tolerance, max_iter, d_pr, residual);
 
  if (status !=0)
    switch ( status ) { 
      case -1: std::cerr<< "Error : bad parameters in Pagerank"<<std::endl; return GDF_CUDA_ERROR; 
      case 1: std::cerr<< "Warning : Pagerank did not reached the desired tolerance"<<std::endl;  return GDF_CUDA_ERROR; 
      default:  std::cerr<< "Pagerank failed"<<std::endl;  return GDF_CUDA_ERROR; 
    }   
 
  cugraph::copy<WT>(m, d_pr, (WT*)pagerank->data);

  ALLOC_FREE_TRY(d_val, stream);
  ALLOC_FREE_TRY(d_pr, stream);
  ALLOC_FREE_TRY(d_leaf_vector, stream);

  return GDF_SUCCESS;
}

gdf_error gdf_add_adj_list(gdf_graph *graph) {
  if (graph->adjList != nullptr)
    return GDF_SUCCESS;

  GDF_REQUIRE( graph->edgeList != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( graph->adjList == nullptr , GDF_INVALID_API_CALL);

  if (graph->edgeList->edge_data != nullptr) {
    switch (graph->edgeList->edge_data->dtype) {
      case GDF_FLOAT32:   return gdf_add_adj_list_impl<float>(graph);
      case GDF_FLOAT64:   return gdf_add_adj_list_impl<double>(graph);
      default: return GDF_UNSUPPORTED_DTYPE;
    }
  }
  else {
    return gdf_add_adj_list_impl<float>(graph);
  }
}

gdf_error gdf_add_transposed_adj_list(gdf_graph *graph) {
  if (graph->edgeList == nullptr)
    gdf_add_edge_list(graph);
  if (graph->edgeList->edge_data != nullptr) {
    switch (graph->edgeList->edge_data->dtype) {
      case GDF_FLOAT32:   return gdf_add_transposed_adj_list_impl<float>(graph);
      case GDF_FLOAT64:   return gdf_add_transposed_adj_list_impl<double>(graph);
      default: return GDF_UNSUPPORTED_DTYPE;
    }
  }
  else {
    return gdf_add_transposed_adj_list_impl<float>(graph);
  }
}

gdf_error gdf_delete_adj_list(gdf_graph *graph) {
  if (graph->adjList) {
    delete graph->adjList;
  }
  graph->adjList = nullptr;
  return GDF_SUCCESS;
}

gdf_error gdf_delete_edge_list(gdf_graph *graph) {
  if (graph->edgeList) {
    delete graph->edgeList;
  }
  graph->edgeList = nullptr;
  return GDF_SUCCESS;
}

gdf_error gdf_delete_transposed_adj_list(gdf_graph *graph) {
  if (graph->transposedAdjList) {
    delete graph->transposedAdjList;
  }
  graph->transposedAdjList = nullptr;
  return GDF_SUCCESS;
}

gdf_error gdf_pagerank(gdf_graph *graph, gdf_column *pagerank, float alpha, float tolerance, int max_iter, bool has_guess) {
  switch (pagerank->dtype) {
    case GDF_FLOAT32:   return gdf_pagerank_impl<float>(graph, pagerank, alpha, tolerance, max_iter, has_guess);
    case GDF_FLOAT64:   return gdf_pagerank_impl<double>(graph, pagerank, alpha, tolerance, max_iter, has_guess);
    default: return GDF_UNSUPPORTED_DTYPE;
  }
}

gdf_error gdf_bfs(gdf_graph *graph, gdf_column *distances, gdf_column *predecessors, int start_node, bool directed) {
  GDF_REQUIRE(graph->adjList != nullptr || graph->edgeList != nullptr, GDF_INVALID_API_CALL);
  gdf_error err = gdf_add_adj_list(graph);
  if (err != GDF_SUCCESS)
    return err;
  GDF_REQUIRE(graph->adjList->offsets->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(graph->adjList->indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(distances->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(predecessors->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);

  int n = graph->adjList->offsets->size - 1;
  int e = graph->adjList->indices->size;
  int* offsets_ptr = (int*)graph->adjList->offsets->data;
  int* indices_ptr = (int*)graph->adjList->indices->data;
  int* distances_ptr = (int*)distances->data;
  int* predecessors_ptr = (int*)predecessors->data;
  int alpha = 15;
  int beta = 18;

  cugraph::Bfs<int> bfs(n, e, offsets_ptr, indices_ptr, directed, alpha, beta);
  bfs.configure(distances_ptr, predecessors_ptr, nullptr);
  bfs.traverse(start_node);
  return GDF_SUCCESS;
}

gdf_error gdf_jaccard(gdf_graph *graph, void *c_gamma, gdf_column *weights, gdf_column *weight_j) {
  GDF_REQUIRE(graph->adjList != nullptr || graph->edgeList != nullptr, GDF_INVALID_API_CALL);
  gdf_error err = gdf_add_adj_list(graph);
  if (err != GDF_SUCCESS)
    return err;
  //GDF_REQUIRE(weight_j->dtype == GDF_FLOAT32, GDF_UNSUPPORTED_DTYPE);
  
  size_t n = graph->adjList->offsets->size - 1;
  size_t e = graph->adjList->indices->size;

  void* offsets_ptr = graph->adjList->offsets->data;
  void* indices_ptr = graph->adjList->indices->data;
  void* value_ptr = graph->adjList->edge_data? graph->adjList->edge_data->data: NULL;
  void* weight_j_ptr = weight_j->data;

  void* weights_ptr;
  if(weights == NULL)
      weights_ptr = NULL;
  else
      weights_ptr = weights->data;

  auto gdf_to_cudadtype= [](gdf_column *col){
    hipDataType cuda_dtype; 
    switch(col->dtype){
      case GDF_INT8: cuda_dtype = HIP_R_8I; break;
      case GDF_INT32: cuda_dtype = HIP_R_32I; break;
      case GDF_FLOAT32: cuda_dtype = HIP_R_32F; break;
      case GDF_FLOAT64: cuda_dtype = HIP_R_64F; break;
      }return cuda_dtype;
  };

  hipDataType index_type = gdf_to_cudadtype(graph->adjList->indices);
  hipDataType val_type = graph->adjList->edge_data? gdf_to_cudadtype(graph->adjList->edge_data): HIP_R_32F;

  nvgraphJaccard(index_type, val_type, n, e, offsets_ptr, indices_ptr, NULL,
                 0, weights_ptr, c_gamma, weight_j_ptr);
  return GDF_SUCCESS;
}

gdf_error gdf_louvain(gdf_graph *graph, void *final_modularity, void *num_level, gdf_column *louvain_parts) {
  GDF_REQUIRE(graph->adjList != nullptr || graph->edgeList != nullptr, GDF_INVALID_API_CALL);
  gdf_error err = gdf_add_adj_list(graph);
  if (err != GDF_SUCCESS)
    return err;

  size_t n = graph->adjList->offsets->size - 1;
  size_t e = graph->adjList->indices->size;

  void* offsets_ptr = graph->adjList->offsets->data;
  void* indices_ptr = graph->adjList->indices->data;
  void* value_ptr = graph->adjList->edge_data? graph->adjList->edge_data->data: NULL;
  void* louvain_parts_ptr = louvain_parts->data;

  auto gdf_to_cudadtype= [](gdf_column *col){
    hipDataType cuda_dtype;
    switch(col->dtype){
      case GDF_INT8: cuda_dtype = HIP_R_8I; break;
      case GDF_INT32: cuda_dtype = HIP_R_32I; break;
      case GDF_FLOAT32: cuda_dtype = HIP_R_32F; break;
      case GDF_FLOAT64: cuda_dtype = HIP_R_64F; break;
      }return cuda_dtype;
  };

  hipDataType index_type = gdf_to_cudadtype(graph->adjList->indices);
  hipDataType val_type = graph->adjList->edge_data? gdf_to_cudadtype(graph->adjList->edge_data): HIP_R_32F;

  nvgraphLouvain(index_type, val_type, n, e, offsets_ptr, indices_ptr, value_ptr, 1, 0, NULL, 
                 final_modularity, louvain_parts_ptr, num_level);
  return GDF_SUCCESS;
}
