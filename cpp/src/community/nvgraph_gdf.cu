// -*-c++-*-

/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/** ---------------------------------------------------------------------------*
 * @brief Wrapper functions for Nvgraph
 *
 * @file nvgraph_gdf.cu
 * ---------------------------------------------------------------------------**/

#include <cugraph.h>
#include <nvgraph/nvgraph.h>
#include <thrust/device_vector.h>
#include <ctime>
#include "utilities/error_utils.h"
#include "converters/nvgraph.cuh"
#include <rmm_utils.h>

namespace cugraph {

void louvain(Graph *graph, void *final_modularity, void *num_level, void *louvain_parts_ptr, int max_iter) {

  CHECK_GRAPH(graph);

  size_t n = graph->adjList->offsets->size - 1;
  size_t e = graph->adjList->indices->size;

  void* offsets_ptr = graph->adjList->offsets->data;
  void* indices_ptr = graph->adjList->indices->data;

  void* value_ptr;
  rmm::device_vector<float> d_values;
  if(graph->adjList->edge_data) {
      value_ptr = graph->adjList->edge_data->data;
  }
  else {
      hipStream_t stream {nullptr};
      d_values.resize(graph->adjList->indices->size);
      thrust::fill(rmm::exec_policy(stream)->on(stream), d_values.begin(), d_values.end(), 1.0);
      value_ptr = (void * ) thrust::raw_pointer_cast(d_values.data());
  }

  auto gdf_to_cudadtype= [](gdf_column *col){
    hipDataType cuda_dtype;
    switch(col->dtype){
      case GDF_INT8: cuda_dtype = HIP_R_8I; break;
      case GDF_INT32: cuda_dtype = HIP_R_32I; break;
      case GDF_FLOAT32: cuda_dtype = HIP_R_32F; break;
      case GDF_FLOAT64: cuda_dtype = HIP_R_64F; break;
      default: throw new std::invalid_argument("Cannot convert data type");
      }return cuda_dtype;
  };

  hipDataType index_type = gdf_to_cudadtype(graph->adjList->indices);
  hipDataType val_type = graph->adjList->edge_data? gdf_to_cudadtype(graph->adjList->edge_data): HIP_R_32F;

  nvgraphLouvain(index_type, val_type, n, e, offsets_ptr, indices_ptr, value_ptr, 1, 0, NULL,
                 final_modularity, louvain_parts_ptr, num_level, max_iter);
  
}

} //namespace cugraph
