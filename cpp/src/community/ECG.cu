#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/** ---------------------------------------------------------------------------*
 * @brief Wrapper functions for Nvgraph
 *
 * @file nvgraph_gdf.cu
 * ---------------------------------------------------------------------------**/

#include <cugraph.h>
#include <nvgraph/nvgraph.h>
#include <thrust/device_vector.h>
#include <ctime>
#include "utilities/error_utils.h"
#include "converters/nvgraph.cuh"
#include <rmm_utils.h>
#include "utilities/graph_utils.cuh"
#include "converters/COOtoCSR.cuh"

namespace {
struct prg {
  __host__ __device__
  float operator()(n){
    thrust::default_random_engine rng;
    thrust::uniform_real_distribution<float> dist(0.0, 1.0);
    rng.discard(n);
    return dist(rng);
  }
};

template <typename IdxT>
struct permutation_functor{
  IdxT* permutation;
  permutation_functor(IdxT* p):permutation(p){}
  __host__ __device__
  IdxT operator()(IdxT in){
    return permutation[in];
  }
};

template<typename IdxT, typename ValT>
cugraph::Graph* permute_graph(cugraph::Graph* graph, IdxT* permutation) {
  // Get the source indices from the offsets
  IdxT* src_indices;
  IdxT nnz = graph->adjList->indices->size;
  ALLOC_TRY(src_indices, sizeof(IdxT) * graph->adjList->indices->size, nullptr);
  cugraph::detail::offsets_to_indices((IdxT*) graph->adjList->indices->data,
                                      graph->adjList->offsets->size - 1,
                                      src_indices);
  // Permute the src_indices
  permutation_functor<IdxT>pf(permutation);
  thrust::transform(rmm::exec_policy(stream)->on(stream),
                    src_indices,
                    src_indices + nnz,
                    src_indices,
                    pf);

  // Copy the indices and values
  IdxT* dest_indices;
  ALLOC_TRY(dest_indices, sizeof(IdxT) * nnz, nullptr);
  ValT* values;
  ALLOC_TRY(values, sizeof(ValT) * nnz, nullptr);
  thrust::copy(rmm::exec_policy(nullptr)->on(nullptr),
               (IdxT*) graph->adjList->indices->data,
               (IdxT*) graph->adjList->indices->data + nnz,
               dest_indices);
  thrust::copy(rmm::exec_policy(nullptr)->on(nullptr),
               (ValT*) graph->adjList->edge_data->data,
               (ValT*) graph->adjList->edge_data->data + nnz,
               values);

  // Permute the destination indices
  thrust::transform(rmm::exec_policy(nullptr)->on(nullptr),
                    dest_indices,
                    dest_indices + nnz,
                    dest_indices,
                    pf);

  // Call COO2CSR to get the new adjacency
  CSR_result_weighted<IdxT, ValT> new_csr;
  ConvertCOOtoCSR_weighted(src_indices, dest_indices, values, (int64_t)nnz, new_csr);

  // Construct the result graph
  cugraph::Graph* result = new cugraph::Graph;
  result->adjList = new gdf_adj_list;
  result->adjList->offsets = new gdf_column;
  result->adjList->indices = new gdf_column;
  result->adjList->edge_data = new gdf_column;
  result->adjList->ownership = 0;

  gdf_column_view(result->adjList->offsets,
                  new_csr.rowOffsets,
                  nullptr,
                  new_csr.size + 1,
                  graph->adjList->offsets->dtype);
  gdf_column_view(result->adjList->indices,
                  new_csr.colIndices,
                  nullptr,
                  nnz,
                  graph->adjList->offsets->dtype);
  gdf_column_view(result->adjList->edge_data,
                  new_csr.edgeWeights,
                  nullptr,
                  nnz,
                  graph->adjList->edge_data->dtype);

  ALLOC_FREE_TRY(src_indices, nullptr);
  ALLOC_FREE_TRY(dest_indices, nullptr);
  ALLOC_FREE_TRY(values, nullptr);

  return result;
}

template <typename IdxT>
IdxT* get_permutation_vector(IdxT size, IdxT seed) {
  IdxT* output_vector;
  ALLOC_TRY(output_vector, sizeof(IdxT) * size, nullptr);
  float* randoms;
  ALLOC_TRY(randoms, sizeof(float) * size, nullptr);

  thrust::counting_iterator<uint32_t> index(seed);
  thrust::transform(rmm::exec_policy(nullptr)->on(nullptr), index, index + size, randoms, prg());
  thrust::sequence(rmm::exec_policy(nullptr)->on(nullptr), output_vector, output_vector + size, 0);
  thrust::sort_by_key(rmm::exec_policy(nullptr)->on(nullptr), randoms, randoms + size, output_vector);

  ALLOC_FREE_TRY(randoms, nullptr);
  return output_vector;
}

template <typename IdxT, typename ValT>
void ecg_impl(cugraph::Graph* graph,
              double min_weight,
              int ensemble_size,
              gdf_column *ecg_parts) {
  IdxT size = graph->adjList->offsets->size - 1;
  IdxT* offsets = (IdxT*)graph->adjList->offsets->data;
  IdxT* indices = (IdxT*)graph->adjList->indices->data;
  ValT* weights = (ValT*)graph->adjList->edge_data->data;

  for (int i = 0; i < ensemble_size; i++) {
    // Take random permutation of the graph
    IdxT* permutation = getPermutationVector(size, size * i);

    // Run Louvain clustering on the random permutation

    // For each edge in the graph determine whether the endpoints are in the same partition

    // Keep a sum for each edge of the total number of times its endpoints are in the same partition
  }

  // Set weights = min_weight + (1 - min-weight)*sum/ensemble_size

  // Run Louvain on the original graph using the computed weights
}
} // anonymous namespace


namespace cugraph {
void ecg(Graph* graph,
         double min_weight,
         int ensemble_size,
         gdf_column *ecg_parts) {
  CUGRAPH_EXPECTS(graph != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(ecg_parts != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(graph->adjList != nullptr, "Graph must have adjacency list");
  CUGRAPH_EXPECTS(graph->adjList->edge_data != nullptr, "Graph must have weights");
  CUGRAPH_EXPECTS(graph->adfList->offsets->dtype == ecg_parts->dtype, "Output type must match index type!");

  // determine the index type and value type of the graph
  // Call the appropriate templated instance of the implementation
  switch (graph->adjList->offsets->dtype) {
    case GDF_INT32: {
      switch (graph->adjList->edge_data) {
        case GDF_FLOAT32: {
          ecg_impl<int32_t, float>(graph, min_weight, ensemble_size, ecg_parts);
          break;
        }
        case GDF_FLOAT64: {
          ecg_impl<int32_t, double>(graph, min_weight, ensemble_size, ecg_parts);
          break;
        }
      }
      break;
    }
    case GDF_INT64: {
      switch (graph->adjList->edge_data) {
        case GDF_FLOAT32: {
          ecg_impl<int64_t, float>(graph, min_weight, ensemble_size, ecg_parts);
          break;
        }
        case GDF_FLOAT64: {
          ecg_impl<int64_t, double>(graph, min_weight, ensemble_size, ecg_parts);
          break;
        }
      }
    }
  }
}
} // cugraph namespace
