/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <community/louvain.cuh>

namespace cugraph {
namespace detail {

template <typename vertex_t, typename edge_t, typename weight_t>
std::pair<int, weight_t> louvain(GraphCSRView<vertex_t, edge_t, weight_t> const &graph,
                                 vertex_t *louvain_parts,
                                 int max_level,
                                 weight_t resolution,
                                 hipStream_t stream)
{
  CUGRAPH_EXPECTS(graph.edge_data != nullptr, "API error, louvain expects a weighted graph");
  CUGRAPH_EXPECTS(louvain_parts != nullptr, "API error, louvain_parts is null");

  Louvain<GraphCSRView<vertex_t, edge_t, weight_t>> runner(graph, stream);

  return runner.compute(louvain_parts, max_level, resolution);
}

}  // namespace detail

template <typename vertex_t, typename edge_t, typename weight_t>
std::pair<int, weight_t> louvain(GraphCSRView<vertex_t, edge_t, weight_t> const &graph,
                                 vertex_t *louvain_parts,
                                 int max_level,
                                 weight_t resolution)
{
  hipStream_t stream{0};

  return detail::louvain(graph, louvain_parts, max_level, resolution, stream);
}

template std::pair<int, float> louvain(
  GraphCSRView<int32_t, int32_t, float> const &, int32_t *, int, float);
template std::pair<int, double> louvain(
  GraphCSRView<int32_t, int32_t, double> const &, int32_t *, int, double);

}  // namespace cugraph
