#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <graph.hpp>

#include <rmm/thrust_rmm_allocator.h>

#include <nvgraph/include/high_res_clock.h>
#include <nvgraph/include/util.cuh>
#include <utilities/cuda_utils.cuh>
#include <utilities/graph_utils.cuh>

#include <converters/COOtoCSR.cuh>

namespace cugraph {
namespace detail {

template <typename vertex_t, typename edge_t, typename weight_t>
__global__  // __launch_bounds__(CUDA_MAX_KERNEL_THREADS)
  void
  compute_vertex_sums(vertex_t n_vertex,
                      edge_t const *offsets,
                      weight_t const *weights,
                      weight_t *output)
{
  // FIXME:  Do this at a WARP level and do an inline reduce
  //         to better handle high degree vertices?
  //
  int src = blockDim.x * blockIdx.x + threadIdx.x;

  if ((src < n_vertex)) {
    weight_t sum{0.0};

    for (int i = offsets[src]; i < offsets[src + 1]; ++i) { sum += weights[i]; }

    output[src] = sum;
  }
}

template <typename vertex_t, typename edge_t, typename weight_t>
__global__ void kernel_modularity_no_matrix(vertex_t n_vertex,
                                            vertex_t n_clusters,
                                            weight_t m2,
                                            edge_t const *offsets,
                                            vertex_t const *indices,
                                            weight_t const *weights,
                                            vertex_t const *cluster,
                                            weight_t const *vertex_sums,
                                            weight_t const *cluster_sums,
                                            weight_t *Q_arr)
{
  int src = blockIdx.x * blockDim.x + threadIdx.x;

  if (src < n_vertex) {
    weight_t Ai{0.0};

    vertex_t c_i = cluster[src];
    weight_t ki  = vertex_sums[src];

    for (int j = offsets[src]; j < offsets[src + 1]; ++j) {
      if (c_i != cluster[indices[j]]) Ai += weights[j];
    }

    weight_t sum_k = m2 - cluster_sums[c_i];
    Q_arr[src]     = (Ai - ((ki * sum_k) / m2)) / m2;
  }
}

//
//  NEW APPROACH!!!
//
//  Parallelizing Louvain is hard.  There are a bunch of attempts
//  at identifying which work can be done in parallel before doing
//  the parallel work.
//
//  For now, start with the simplest parallel model which will
//  be serial over the vertices in the worst case.  This should get
//  the correct answer while we work an a more optimal implementation.
//
//  TODO:  This might need to be just 1 warp for now...
//
template <typename vertex_t, typename edge_t, typename weight_t>
__global__  // __launch_bounds__(CUDA_MAX_KERNEL_THREADS)
  void
  update_each_assignment_by_delta_modularity(weight_t m2,
                                             vertex_t n_vertex,
                                             edge_t n_edges,
                                             edge_t const *offsets,
                                             vertex_t const *indices,
                                             weight_t const *weights,
                                             weight_t const *vertex_weights,
                                             weight_t volatile *cluster_weights,
                                             vertex_t volatile *cluster,
                                             vertex_t volatile *vertex_ready)
{
  unsigned int tid = threadIdx.x;  // 0 ~ 31

  if (blockIdx.x > 0) return;

  __shared__ edge_t start_idx;
  __shared__ edge_t end_idx;
  __shared__ vertex_t local_max_loc[WARP_SIZE];
  __shared__ weight_t local_max[WARP_SIZE];

  for (vertex_t src = 0; src < n_vertex; ++src) {
    if (tid == 0) {
      start_idx = offsets[src];
      end_idx   = offsets[src + 1];
    }

    __syncwarp();

    local_max[tid]     = -1.0;
    local_max_loc[tid] = -1;

    //
    //  So, first we're going to compute the delta modularity
    //  for each edge associated with this vertex and store
    //  the maximum
    //
    vertex_t old_cluster = cluster[src];
    weight_t degc_totw   = vertex_weights[src] / m2;

    for (int loc = start_idx + tid; loc < end_idx; loc += WARP_SIZE) {
      vertex_t dst         = indices[loc];
      vertex_t new_cluster = cluster[dst];

      if (old_cluster != new_cluster) {
        weight_t delta_mod{-1.0};
        weight_t old_cluster_sum{0.0};
        weight_t new_cluster_sum{0.0};

        //
        // TODO:  This could be computed by the warp once into
        //        a temp array.  TOO MUCH MEMORY.  If we stay
        //        with 1 warp doing all the work could use
        //        rmm::device_vector<weight_t>(num_verts)...
        //        but with multiple warps, each warp would need its
        //        own array of that size.
        //
        for (edge_t i = offsets[src]; i < offsets[src + 1]; ++i) {
          vertex_t j = indices[i];

          if (j != src) {
            vertex_t cluster_j = cluster[j];

#ifdef DEBUG
            if (src == 3)
              printf(
                "---- src = %d, j = %d, old_cluster = %d, new_cluster = %d, cluster_j = %d, "
                "weights = %g\n",
                src,
                j,
                old_cluster,
                new_cluster,
                cluster_j,
                weights[i]);
#endif

            if (cluster_j == new_cluster) {
              new_cluster_sum += weights[i];
            } else if (cluster_j == old_cluster) {
              old_cluster_sum += weights[i];
            }
          }
        }

        delta_mod =
          new_cluster_sum - degc_totw * cluster_weights[new_cluster] -
          (old_cluster_sum - (degc_totw * (cluster_weights[old_cluster] - vertex_weights[src])));

#ifdef DEBUG
        if (src == 3)
          printf(
            "*** src = %d, dst = %d, loc = %d, (%g - (%g * %g)) - (%g - (%g * (%g - %g))) = %g\n",
            src,
            dst,
            loc,
            new_cluster_sum,
            degc_totw,
            cluster_weights[new_cluster],
            old_cluster_sum,
            degc_totw,
            cluster_weights[old_cluster],
            vertex_weights[src],
            delta_mod);
#endif

        if (delta_mod > local_max[tid]) {
          local_max[tid]     = delta_mod;
          local_max_loc[tid] = loc;
        }
      } else {
        if (local_max[tid] < 0.0) {
          local_max[tid]     = 0.0;
          local_max_loc[tid] = loc;
        }
      }
    }

    __syncwarp();

    // Now we'll do a reduction
    unsigned stride = WARP_SIZE / 2;

    while ((tid < stride) && (stride > 0)) {
      if (((tid + stride) < WARP_SIZE) && ((local_max[tid + stride] > local_max[tid]))) {
        local_max[tid]     = local_max[tid + stride];
        local_max_loc[tid] = local_max_loc[tid + stride];
      }

      stride /= 2;
    }

    __syncwarp();

    //
    //  Now we've identified the best new cluster for this
    //  vertex, update it.
    //
    if (tid == 0) {
      if (local_max[0] > weight_t{0.0}) {
        cluster_weights[cluster[src]] -= vertex_weights[src];
        cluster[src] = cluster[indices[local_max_loc[0]]];
        cluster_weights[cluster[src]] += vertex_weights[src];

#ifdef DEBUG
        printf("assigning vertex %d to cluster %d, score = %g, cluster_weights[%d] = %g\n",
               src,
               cluster[src],
               local_max[0],
               cluster[src],
               cluster_weights[cluster[src]]);
#endif
      }

      vertex_ready[src] = 1;
    }
  }
}

template <typename vertex_t, typename edge_t, typename weight_t>
void generate_superverticies_graph(
  cugraph::experimental::GraphCSRView<vertex_t, edge_t, weight_t> &current_graph,
  vertex_t new_number_of_vertices,
  rmm::device_vector<vertex_t> &cluster_v)
{
  hipStream_t stream{0};

  rmm::device_vector<vertex_t> tmp_src_v(current_graph.number_of_edges);
  rmm::device_vector<vertex_t> new_src_v(current_graph.number_of_edges);
  rmm::device_vector<vertex_t> new_dst_v(current_graph.number_of_edges);
  rmm::device_vector<weight_t> new_weight_v(current_graph.number_of_edges);

  vertex_t *d_old_src    = tmp_src_v.data().get();
  vertex_t *d_old_dst    = current_graph.indices;
  weight_t *d_old_weight = current_graph.edge_data;
  vertex_t *d_new_src    = new_src_v.data().get();
  vertex_t *d_new_dst    = new_dst_v.data().get();
  vertex_t *d_clusters   = cluster_v.data().get();
  weight_t *d_new_weight = new_weight_v.data().get();

  //
  //  First, let's expand the CSR sources into a COO
  //
  current_graph.get_source_indices(d_old_src);

  //
  //  Now we'll renumber the COO
  //
  thrust::for_each(
    rmm::exec_policy(stream)->on(stream),
    thrust::make_counting_iterator<edge_t>(0),
    thrust::make_counting_iterator<edge_t>(current_graph.number_of_edges),
    [d_old_src, d_old_dst, d_new_src, d_new_dst, d_clusters, d_new_weight, d_old_weight] __device__(
      edge_t e) {
      d_new_src[e]    = d_clusters[d_old_src[e]];
      d_new_dst[e]    = d_clusters[d_old_dst[e]];
      d_new_weight[e] = d_old_weight[e];
    });

  thrust::stable_sort_by_key(
    rmm::exec_policy(stream)->on(stream),
    d_new_dst,
    d_new_dst + current_graph.number_of_edges,
    thrust::make_zip_iterator(thrust::make_tuple(d_new_src, d_new_weight)));
  thrust::stable_sort_by_key(
    rmm::exec_policy(stream)->on(stream),
    d_new_src,
    d_new_src + current_graph.number_of_edges,
    thrust::make_zip_iterator(thrust::make_tuple(d_new_dst, d_new_weight)));

  //
  //  Now we reduce by key to combine the weights of duplicate
  //  edges.
  //
  auto start     = thrust::make_zip_iterator(thrust::make_tuple(d_new_src, d_new_dst));
  auto new_start = thrust::make_zip_iterator(thrust::make_tuple(d_old_src, d_old_dst));
  auto new_end   = thrust::reduce_by_key(rmm::exec_policy(stream)->on(stream),
                                       start,
                                       start + current_graph.number_of_edges,
                                       d_new_weight,
                                       new_start,
                                       d_old_weight,
                                       thrust::equal_to<thrust::tuple<vertex_t, vertex_t>>(),
                                       thrust::plus<weight_t>());

  current_graph.number_of_edges    = thrust::distance(new_start, new_end.first);
  current_graph.number_of_vertices = new_number_of_vertices;

  detail::fill_offset(d_old_src,
                      current_graph.offsets,
                      new_number_of_vertices,
                      current_graph.number_of_edges,
                      stream);
  CUDA_CHECK_LAST();
}

template <typename vertex_t, typename edge_t, typename weight_t>
void compute_vertex_sums(experimental::GraphCSRView<vertex_t, edge_t, weight_t> const &graph,
                         rmm::device_vector<weight_t> &sums)
{
  dim3 block_size_1d =
    dim3((graph.number_of_vertices + BLOCK_SIZE_1D * 4 - 1) / BLOCK_SIZE_1D * 4, 1, 1);
  dim3 grid_size_1d = dim3(BLOCK_SIZE_1D * 4, 1, 1);

  compute_vertex_sums<vertex_t, edge_t, weight_t><<<block_size_1d, grid_size_1d>>>(
    graph.number_of_vertices, graph.offsets, graph.edge_data, sums.data().get());
}

template <typename vertex_t, typename edge_t, typename weight_t>
weight_t modularity(weight_t m2,
                    experimental::GraphCSRView<vertex_t, edge_t, weight_t> const &graph,
                    rmm::device_vector<vertex_t> const &cluster,
                    rmm::device_vector<weight_t> const &vertex_sums,
                    rmm::device_vector<weight_t> const &cluster_sums,
                    rmm::device_vector<weight_t> &Q_arr)
{
  int nthreads = min(graph.number_of_vertices, CUDA_MAX_KERNEL_THREADS);
  int nblocks  = min((graph.number_of_vertices + nthreads - 1) / nthreads, CUDA_MAX_BLOCKS);
  kernel_modularity_no_matrix<vertex_t, edge_t, weight_t>
    <<<nblocks, nthreads>>>(graph.number_of_vertices,
                            cluster.size(),
                            m2,
                            graph.offsets,
                            graph.indices,
                            graph.edge_data,
                            cluster.data().get(),
                            vertex_sums.data().get(),
                            cluster_sums.data().get(),
                            Q_arr.data().get());

  CUDA_CALL(hipDeviceSynchronize());

  weight_t Q = thrust::reduce(
    thrust::cuda::par, Q_arr.begin(), Q_arr.begin() + graph.number_of_vertices, weight_t{0.0});

  return -Q;
}

template <typename vertex_t, typename edge_t, typename weight_t>
void update_each_assignment_by_delta_modularity(
  weight_t m2,
  experimental::GraphCSRView<vertex_t, edge_t, weight_t> const &graph,
  rmm::device_vector<weight_t> const &vertex_weights,
  rmm::device_vector<weight_t> &cluster_weights,
  rmm::device_vector<vertex_t> &cluster,
  rmm::device_vector<vertex_t> &vertex_ready)
{
  thrust::fill(vertex_ready.begin(), vertex_ready.end(), vertex_t{0});

  // dim3 block_size_1d = dim3((graph.number_of_vertices + WARP_SIZE - 1) / WARP_SIZE, 1, 1);
  dim3 block_size_1d = dim3(graph.number_of_vertices, 1, 1);
  dim3 grid_size_1d  = dim3(WARP_SIZE, 1, 1);

  update_each_assignment_by_delta_modularity<vertex_t, edge_t, weight_t>
    <<<block_size_1d, grid_size_1d>>>(m2,
                                      graph.number_of_vertices,
                                      graph.number_of_edges,
                                      graph.offsets,
                                      graph.indices,
                                      graph.edge_data,
                                      vertex_weights.data().get(),
                                      cluster_weights.data().get(),
                                      cluster.data().get(),
                                      vertex_ready.data().get());

  CUDA_CALL(hipDeviceSynchronize());
}

template <typename vertex_t, typename edge_t, typename weight_t>
void louvain(experimental::GraphCSRView<vertex_t, edge_t, weight_t> const &graph,
             weight_t *final_modularity,
             int *num_level,
             vertex_t *cluster_vec,
             int max_iter)
{
  *num_level = 0;

  vertex_t n_vertex = graph.number_of_vertices;
  vertex_t n_edges  = graph.number_of_edges;

  //
  //  We create a copy of the graph.  We're going to reuse
  //  this memory to edit the graph during the algorithm.
  //
  rmm::device_vector<edge_t> offsets_v(graph.offsets, graph.offsets + n_vertex + 1);
  rmm::device_vector<vertex_t> indices_v(graph.indices, graph.indices + n_edges);
  rmm::device_vector<weight_t> weights_v(graph.edge_data, graph.edge_data + n_edges);

  //
  //  The cluster graph will have n_vertex edges and at most n_vertex
  //  vertices
  //
  rmm::device_vector<vertex_t> cluster_offsets_v(n_vertex + 1);
  rmm::device_vector<vertex_t> cluster_indices_v(n_vertex);

  //
  //  We need to store to sum the weights for each vertex and cluster
  //
  rmm::device_vector<weight_t> vertex_weights_v(n_vertex);
  rmm::device_vector<weight_t> cluster_weights_v(n_vertex);

  //
  //  TODO:  Can I use cluster_vec directly
  //
  rmm::device_vector<vertex_t> cluster_v(n_vertex);
  rmm::device_vector<weight_t> Q_arr_v(n_vertex);  // really a temp space
  rmm::device_vector<vertex_t> tmp_arr_v(n_vertex);
  rmm::device_vector<vertex_t> cluster_inverse_v(n_vertex);  // really a temp space

  //  TODO:  not needed in current version... for one of the parallel variations
  rmm::device_vector<vertex_t> vertex_ready_v(n_vertex);

  HighResClock hr_clock;
  double timed, diff_time;

  weight_t new_Q;
  weight_t cur_Q;

  // TODO:  Would like this to be a cpp file... need to move some things like this into the kernels
  // file...
  weight_t m2              = thrust::reduce(thrust::cuda::par, weights_v.begin(), weights_v.end());
  weight_t best_modularity = -1;

  //
  //  Initialize every cluster to reference each vertex to itself
  //
  thrust::sequence(thrust::cuda::par, cluster_v.begin(), cluster_v.end());
  thrust::sequence(thrust::cuda::par, cluster_offsets_v.begin(), cluster_offsets_v.end());
  thrust::sequence(thrust::cuda::par, cluster_indices_v.begin(), cluster_indices_v.end());

  //
  //  Start with the base clustering (every vertex in its own cluster)
  //
  thrust::copy(cluster_v.begin(), cluster_v.end(), cluster_vec);

  cugraph::experimental::GraphCSRView<vertex_t, edge_t, weight_t> current_graph(
    offsets_v.data().get(), indices_v.data().get(), weights_v.data().get(), n_vertex, n_edges);

  cugraph::experimental::GraphCSRView<vertex_t, edge_t, weight_t> cluster_graph(
    cluster_offsets_v.data().get(), cluster_indices_v.data().get(), nullptr, n_vertex, n_vertex);

  //
  //  Sum the weights of all edges departing a vertex.  This is
  //  loop invariant, so we'll compute it here.
  //
  cugraph::detail::compute_vertex_sums(graph, vertex_weights_v);

  //
  //  Cluster weights are equivalent to vertex weights with this initial
  //  graph
  //
  thrust::copy(vertex_weights_v.begin(), vertex_weights_v.end(), cluster_weights_v.begin());

  hr_clock.start();

  //
  //  Compute modularity of initial clustering
  //
  new_Q = modularity<vertex_t, edge_t, weight_t>(
    m2, current_graph, cluster_v, vertex_weights_v, cluster_weights_v, Q_arr_v);

  hr_clock.stop(&timed);
  diff_time = timed;

  std::cout << "Initial modularity value: " << new_Q << " runtime: " << (diff_time / 1000)
            << std::endl;

  weight_t delta_Q = 1;

  do {
    cur_Q = new_Q;
    weight_t inner_cur_Q;

    while (delta_Q > 0.0001) {
      inner_cur_Q = new_Q;

      hr_clock.start();

      // Compute delta modularity for each edges
      update_each_assignment_by_delta_modularity(
        m2, current_graph, vertex_weights_v, cluster_weights_v, cluster_v, vertex_ready_v);

      hr_clock.stop(&timed);
      diff_time = timed;
      std::cout << "Complete update_cluster_by_delta_modularity  runtime: " << (diff_time / 1000)
                << std::endl;

#ifdef DEBUG
      std::cout << "clustering = (";
      for (int i = 0; i < current_graph.number_of_vertices; ++i) {
        std::cout << (i == 0 ? "" : ", ") << " " << i << ": " << cluster_v[i];
      }
      std::cout << ")" << std::endl;
#endif

      // FIXME:  make a cluster graph object and pass it around
      new_Q = modularity<vertex_t, edge_t, weight_t>(
        m2, current_graph, cluster_v, vertex_weights_v, cluster_weights_v, Q_arr_v);

      delta_Q = new_Q - inner_cur_Q;

      std::cout << "new_Q = " << new_Q << ", delta_Q = " << delta_Q << std::endl;
    }

    if (new_Q > cur_Q) {
      //
      //  Now we're going to renumber the clusters from 0 to (k-1), where k is the number of
      //  clusters in this level of the dendogram.
      //

      // TODO:  move this into a function for better scoping
      //
      thrust::copy(cluster_v.begin(), cluster_v.end(), tmp_arr_v.begin());
      thrust::sort(tmp_arr_v.begin(), tmp_arr_v.end());
      auto tmp_end = thrust::unique(tmp_arr_v.begin(), tmp_arr_v.end());

      vertex_t num_clusters = thrust::distance(tmp_arr_v.begin(), tmp_end);

      cluster_v.resize(num_clusters);
      tmp_arr_v.resize(num_clusters);
      cluster_weights_v.resize(num_clusters);

      vertex_t *d_tmp_arr         = tmp_arr_v.data().get();
      vertex_t *d_cluster_inverse = cluster_inverse_v.data().get();
      vertex_t *d_cluster         = cluster_v.data().get();

      hipStream_t stream{0};

      thrust::fill(cluster_inverse_v.begin(), cluster_inverse_v.end(), vertex_t{-1});

      thrust::for_each(rmm::exec_policy(stream)->on(stream),
                       thrust::make_counting_iterator<vertex_t>(0),
                       thrust::make_counting_iterator<vertex_t>(num_clusters),
                       [d_tmp_arr, d_cluster_inverse] __device__(vertex_t i) {
                         d_cluster_inverse[d_tmp_arr[i]] = i;
                       });

      thrust::for_each(rmm::exec_policy(stream)->on(stream),
                       thrust::make_counting_iterator<vertex_t>(0),
                       thrust::make_counting_iterator<vertex_t>(current_graph.number_of_vertices),
                       [d_cluster, d_cluster_inverse] __device__(vertex_t i) {
                         d_cluster[i] = d_cluster_inverse[d_cluster[i]];
                       });

      thrust::for_each(rmm::exec_policy(stream)->on(stream),
                       thrust::make_counting_iterator<vertex_t>(0),
                       thrust::make_counting_iterator<vertex_t>(current_graph.number_of_vertices),
                       [cluster_vec, d_cluster] __device__(vertex_t i) {
                         cluster_vec[i] = d_cluster[cluster_vec[i]];
                       });

#ifdef DEBUG
      std::cout << "clustering = (";
      for (int i = 0; i < current_graph.number_of_vertices; ++i) {
        std::cout << (i == 0 ? "" : ", ") << " " << i << ": " << cluster_v[i];
      }
      std::cout << ")" << std::endl;
#endif

      // generate super vertices graph
      generate_superverticies_graph(current_graph, num_clusters, cluster_v);

      //
      //  Sum the weights of all edges departing a vertex.  This is
      //  loop invariant, so we'll compute it here.
      //
      cugraph::detail::compute_vertex_sums(current_graph, vertex_weights_v);

      //
      //  Cluster weights are equivalent to vertex weights with this initial
      //  graph
      //
      thrust::copy(vertex_weights_v.begin(), vertex_weights_v.end(), cluster_weights_v.begin());

      thrust::sequence(
        thrust::cuda::par, cluster_v.begin(), cluster_v.begin() + current_graph.number_of_vertices);

      //
      //  Check... modularity should not have changed
      //
      new_Q = modularity<vertex_t, edge_t, weight_t>(
        m2, current_graph, cluster_v, vertex_weights_v, cluster_weights_v, Q_arr_v);

      delta_Q = new_Q - cur_Q;

#ifdef DEBUG
      std::cout << "after creating super vertices, new_Q = " << new_Q << ", delta_Q = " << delta_Q
                << std::endl;

      std::cout << "new offsets = (";
      for (int i = 0; i < current_graph.number_of_vertices + 1; ++i) {
        std::cout << (i == 0 ? "" : ", ") << " " << offsets_v[i];
      }
      std::cout << ")" << std::endl;

      std::cout << "new indices = (";
      for (int i = 0; i < current_graph.number_of_edges; ++i) {
        std::cout << (i == 0 ? "" : ", ") << " " << indices_v[i];
      }
      std::cout << ")" << std::endl;

      std::cout << "new weights = (";
      for (int i = 0; i < current_graph.number_of_edges; ++i) {
        std::cout << (i == 0 ? "" : ", ") << " " << weights_v[i];
      }
      std::cout << ")" << std::endl;

      std::cout << "vertex weights = (";
      for (int i = 0; i < current_graph.number_of_vertices; ++i) {
        std::cout << (i == 0 ? "" : ", ") << " " << vertex_weights_v[i];
      }
      std::cout << ")" << std::endl;

      std::cout << "cluster map = (";
      for (int i = 0; i < current_graph.number_of_vertices; ++i) {
        std::cout << (i == 0 ? "" : ", ") << " " << cluster_v[i];
      }
      std::cout << ")" << std::endl;
#endif

      best_modularity = new_Q;
    }
  } while (delta_Q > 0.001);

  *final_modularity = best_modularity;
}

template void louvain(
  experimental::GraphCSRView<int32_t, int32_t, float> const &, float *, int *, int32_t *, int);
template void louvain(
  experimental::GraphCSRView<int32_t, int32_t, double> const &, double *, int *, int32_t *, int);

}  // namespace detail
}  // namespace cugraph
