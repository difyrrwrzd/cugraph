#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// snmg pagerank
// Author: Alex Fender afender@nvidia.com
 
#include "hipcub/hipcub.hpp"
#include <omp.h>
#include "rmm_utils.h"
#include <cugraph.h>
#include "utilities/graph_utils.cuh"
#include "snmg/utils.cuh"
#include "utilities/cusparse_helper.h"
#include "snmg/blas/spmv.cuh"
#include "snmg/link_analysis/pagerank.cuh"
#include "snmg/degree/degree.cuh"
//#define SNMG_DEBUG

namespace cugraph
{

  template<typename IndexType, typename ValueType>
__global__ void __launch_bounds__(CUDA_MAX_KERNEL_THREADS)
transition_kernel(const size_t e,
                  const IndexType *ind,
                  const IndexType *degree,
                  ValueType *val) {
  for (auto i = threadIdx.x + blockIdx.x * blockDim.x; 
       i < e; 
       i += gridDim.x * blockDim.x)
    val[i] = 1.0 / degree[ind[i]];
}

template <typename IndexType, typename ValueType>
SNMGpagerank<IndexType,ValueType>::SNMGpagerank(SNMGinfo & env_, size_t* part_off_, 
             IndexType * off_, IndexType * ind_) : 
             env(env_), part_off(part_off_), off(off_), ind(ind_) { 
  id = env.get_thread_num();
  nt = env.get_num_threads(); 
  v_glob = part_off[nt];
  v_loc = part_off[id+1]-part_off[id];
  IndexType tmp_e;
  hipMemcpy(&tmp_e, &off[v_loc], sizeof(IndexType),hipMemcpyDeviceToHost);
  cudaCheckError();
  e_loc = tmp_e;
  stream = nullptr;
  is_setup = false;
  ALLOC_TRY ((void**)&bookmark,   sizeof(ValueType) * v_glob, stream);
  ALLOC_TRY ((void**)&val, sizeof(ValueType) * e_loc, stream);

  // intialize cusparse. This can take some time.
  Cusparse::get_handle();
} 

template <typename IndexType, typename ValueType>
SNMGpagerank<IndexType,ValueType>::~SNMGpagerank() { 
  Cusparse::destroy_handle();
  ALLOC_FREE_TRY(bookmark, stream); 
  ALLOC_FREE_TRY(val, stream);
}

template <typename IndexType, typename ValueType>
void SNMGpagerank<IndexType,ValueType>::transition_vals(const IndexType *degree) {
  int threads = min(static_cast<IndexType>(e_loc), 256);
  int blocks = min(static_cast<IndexType>(32*env.get_num_sm()), CUDA_MAX_BLOCKS);
  transition_kernel<IndexType, ValueType> <<<blocks, threads>>> (e_loc, ind, degree, val);
  cudaCheckError();
}

template <typename IndexType, typename ValueType>
void SNMGpagerank<IndexType,ValueType>::flag_leafs(const IndexType *degree) {
  int threads = min(static_cast<IndexType>(v_glob), 256);
  int blocks = min(static_cast<IndexType>(32*env.get_num_sm()), CUDA_MAX_BLOCKS);
  flag_leafs_kernel<IndexType, ValueType> <<<blocks, threads>>> (v_glob, degree, bookmark);
  cudaCheckError();
}    


// Artificially create the google matrix by setting val and bookmark
template <typename IndexType, typename ValueType>
void SNMGpagerank<IndexType,ValueType>::setup(ValueType _alpha, IndexType** degree) {
  if (!is_setup) {

    alpha=_alpha;
    ValueType zero = 0.0; 
    IndexType *degree_loc;
    ALLOC_TRY ((void**)&degree_loc,   sizeof(IndexType) * v_glob, stream);
    degree[id] = degree_loc;
    if (snmg_degree(1, part_off, off, ind, degree))
       throw std::string("SNMG Degree failed in Pagerank");

    // Update dangling node vector
    fill(v_glob, bookmark, zero);
    flag_leafs(degree_loc);
    update_dangling_nodes(v_glob, bookmark, alpha);

    // Transition matrix
    transition_vals(degree_loc);

    //exit
    ALLOC_FREE_TRY(degree_loc, stream);
    is_setup = true;
  }
  else
    throw std::string("Setup can be called only once");
}

// run the power iteration on the google matrix
template <typename IndexType, typename ValueType>
void SNMGpagerank<IndexType,ValueType>::solve (int max_iter, ValueType ** pagerank) {
  if (is_setup) {
    ValueType  dot_res;
    ValueType one = 1.0;
    ValueType *pr = pagerank[id];
    fill(v_glob, pagerank[id], one/v_glob);
    dot_res = dot( v_glob, bookmark, pr);
    SNMGcsrmv<IndexType,ValueType> spmv_solver(env, part_off, off, ind, val, pagerank);
    for (auto i = 0; i < max_iter; ++i) {
      spmv_solver.run(pagerank);
      scal(v_glob, alpha, pr);
      addv(v_glob, dot_res * (one/v_glob) , pr);
      dot_res = dot( v_glob, bookmark, pr);
      scal(v_glob, one/nrm2(v_glob, pr) , pr);
    }
    scal(v_glob, one/nrm1(v_glob,pr), pr);
  }
  else {
      throw std::string("Solve was called before setup");
  }
}

template class SNMGpagerank<int, double>;
template class SNMGpagerank<int, float>;


} //namespace cugraph

__global__ void dummy_Kernel(int* src, int* dst, size_t e, int* res) {
        int i = threadIdx.x+blockIdx.x*blockDim.x;
        if(i<e)
        {
            res[i]= src[i] + dst[i];
        }
}

gdf_error gdf_multi_pagerank(const size_t n_gpus, gdf_column *src_ptrs, gdf_column *dest_ptrs, gdf_column *pr, const float damping_factor, const int max_iter){

    /*const char* p = std::getenv("CUDA_VISIBLE_DEVICES");
    int x=0;
    int a[n_gpus];
    for(int i=0;p[i]!=NULL;i++)
    {
        if (p[i]!=',')
        {a[x]=int(p[i])-int('0');
        x++;}
    }
    std::map<int,int> actual_to_canonical;;
    for(int i =0;i<n_gpus;i++)
    {
    actual_to_canonical[a[i]]=i;
    }

    int prefix_sum[N+1];
    prefix_sum[0] = 0;
    for(int i=0;i<n_gpus;i++)
    {
      prefix_sum[i+1] = prefix_sum[i] + src_ptrs[actual_to_canonical[i]].size;
    }
    int total_length = prefix_sum[n_gpus];
    */
  int prefix_sum[n_gpus+1];
  prefix_sum[0] = 0;
  for(int i=0;i<n_gpus;i++)
  {
      prefix_sum[i+1] = prefix_sum[i] + src_ptrs[i].size;
  }
  int total_length = prefix_sum[n_gpus];


  int* h_result = (int*)malloc(total_length*sizeof(int));
  int *final_result = h_result;
  int *d_result;
  hipMalloc(&d_result, total_length*sizeof(int));

  printf("\nSTART OMP CODE");
       #pragma omp parallel num_threads(n_gpus)
       {
        auto i = omp_get_thread_num();
        auto p = omp_get_num_threads(); 
        printf("\n Excecuting omp thread %d", i);
        /*hipPointerAttribute_t attr;
        hipPointerGetAttributes (&attr, src_ptrs[i].data);
        hipDeviceSynchronize();
        int dev = attr.device;
        printf("\n Device: %d", dev);
        hipSetDevice(dev);*/
        hipSetDevice(i);
        int *ans;
        hipMalloc(&ans, src_ptrs[i].size*sizeof(int));
        
        int e = src_ptrs[i].size;
        dim3 nthreads, nblocks;
        nthreads.x = min(e, CUDA_MAX_KERNEL_THREADS);
        nthreads.y = 1;
        nthreads.z = 1;
        nblocks.x = min((e + nthreads.x - 1) / nthreads.x, CUDA_MAX_BLOCKS);
        nblocks.y = 1;
        nblocks.z = 1;
        dummy_Kernel<<<nblocks,nthreads>>>((int*)src_ptrs[i].data,(int*)dest_ptrs[i].data, e, (int*)ans);
        
        hipDeviceSynchronize();
        hipMemcpy(final_result+prefix_sum[i], ans, src_ptrs[i].size*sizeof(int), hipMemcpyDeviceToHost);
       }
  printf("\n END OMP\n");


  printf("\nRESULT ON HOST:");
  for(int i=0;i<total_length;i++)
  {
      printf("%d\t", h_result[i]);
  }
  printf("\n\n");

  hipMemcpy(d_result,h_result, total_length*sizeof(int), hipMemcpyHostToDevice);
  pr->data = (void*)d_result;
  pr->size = total_length;

  return GDF_SUCCESS;
}
