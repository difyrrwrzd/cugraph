#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cugraph.h>
#include <rmm_utils.h>
#include <db/db_object.cuh>
#include <cub/device/device_run_length_encode.cuh>

namespace cugraph {
  // Define kernel for copying run length encoded values into offset slots.
  template<typename T>
  __global__ void offsetsKernel(T runCounts, T* unique, T* counts, T* offsets) {
    uint64_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < runCounts)
      offsets[unique[tid]] = counts[tid];
  }

  template<typename idx_t>
  db_pattern_entry<idx_t>::db_pattern_entry(std::string variable) {
    is_var = true;
    variableName = variable;
  }

  template<typename idx_t>
  db_pattern_entry<idx_t>::db_pattern_entry(idx_t constant) {
    is_var = false;
    constantValue = constant;
  }

  template<typename idx_t>
  db_pattern_entry<idx_t>::db_pattern_entry(const db_pattern_entry<idx_t>& other) {
    is_var = other.is_var;
    constantValue = other.constantValue;
    variableName = other.variableName;
  }

  template<typename idx_t>
  bool db_pattern_entry<idx_t>::isVariable() const {
    return is_var;
  }

  template<typename idx_t>
  idx_t db_pattern_entry<idx_t>::getConstant() const {
    return constantValue;
  }

  template<typename idx_t>
  std::string db_pattern_entry<idx_t>::getVariable() const {
    return variableName;
  }

  template class db_pattern_entry<int32_t> ;
  template class db_pattern_entry<int64_t> ;

  template<typename idx_t>
  db_pattern<idx_t>::db_pattern() {

  }

  template<typename idx_t>
  db_pattern<idx_t>::db_pattern(const db_pattern<idx_t>& other) {
    for (size_t i = 0; i < other.entries.size(); i++) {
      entries.push_back(other.getEntry(i));
    }
  }

  template<typename idx_t>
  int db_pattern<idx_t>::getSize() const {
    return entries.size();
  }

  template<typename idx_t>
  const db_pattern_entry<idx_t>& db_pattern<idx_t>::getEntry(int position) const {
    return entries[position];
  }

  template<typename idx_t>
  void db_pattern<idx_t>::addEntry(db_pattern_entry<idx_t>& entry) {
    entries.push_back(entry);
  }

  template<typename idx_t>
  bool db_pattern<idx_t>::isAllConstants() {
    for (size_t i = 0; i < entries.size(); i++)
      if (entries[i].isVariable())
        return false;
    return true;
  }

  template class db_pattern<int32_t> ;
  template class db_pattern<int64_t> ;

  template<typename idx_t>
  void db_column_index<idx_t>::deleteData() {
    if (offsets->data != nullptr)
      ALLOC_FREE_TRY(offsets->data, nullptr);
    if (indirection->data != nullptr)
      ALLOC_FREE_TRY(indirection->data, nullptr);
    free(offsets);
    free(indirection);
  }

  template<typename idx_t>
  db_column_index<idx_t>::db_column_index() {
    gdf_column* _offsets = (gdf_column*) malloc(sizeof(gdf_column));
    gdf_col_set_defaults(_offsets);
    gdf_column_view(_offsets,
                    nullptr,
                    nullptr,
                    0,
                    std::is_same<idx_t, int32_t>::value ? GDF_INT32 : GDF_INT64);
    offsets = _offsets;
    gdf_column* _indirection = (gdf_column*) malloc(sizeof(gdf_column));
    gdf_col_set_defaults(_indirection);
    gdf_column_view(_indirection,
                    nullptr,
                    nullptr,
                    0,
                    std::is_same<idx_t, int32_t>::value ? GDF_INT32 : GDF_INT64);
    indirection = _indirection;
  }

  template<typename idx_t>
  db_column_index<idx_t>::db_column_index(gdf_column* _offsets, gdf_column* _indirection) {
    offsets = _offsets;
    indirection = _indirection;
  }

  template<typename idx_t>
  db_column_index<idx_t>::~db_column_index() {
    deleteData();
  }

  template<typename idx_t>
  void db_column_index<idx_t>::resetData(gdf_column* _offsets, gdf_column* _indirection) {
    deleteData();
    offsets = _offsets;
    indirection = _indirection;
  }

  template<typename idx_t>
  gdf_column* db_column_index<idx_t>::getOffsets() {
    return offsets;
  }

  template<typename idx_t>
  gdf_column* db_column_index<idx_t>::getIndirection() {
    return indirection;
  }

  template class db_column_index<int32_t> ;
  template class db_column_index<int64_t> ;

  template<typename idx_t>
  db_result<idx_t>::db_result() {
    dataValid = false;
    columnSize = 0;
  }

  template<typename idx_t>
  db_result<idx_t>::db_result(db_result&& other) {
    dataValid = other.dataValid;
    for (size_t i = 0; i < other.columns.size(); i++)
      columns.push_back(other.columns[i]);
    for (size_t i = 0; i < other.names.size(); i++)
      names.push_back(other.names[i]);
    other.dataValid = false;
  }

  template<typename idx_t>
  db_result<idx_t>& db_result<idx_t>::operator =(db_result<idx_t> && other) {
    dataValid = other.dataValid;
    for (size_t i = 0; i < other.columns.size(); i++)
      columns.push_back(other.columns[i]);
    for (size_t i = 0; i < other.names.size(); i++)
    names.push_back(other.names[i]);
    other.dataValid = false;
    return *this;
  }

  template<typename idx_t>
  db_result<idx_t>::~db_result() {
    deleteData();
  }

  template<typename idx_t>
  void db_result<idx_t>::deleteData() {
    if (dataValid)
      for (size_t i = 0; i < columns.size(); i++)
        ALLOC_FREE_TRY(columns[i], nullptr);
  }

  template<typename idx_t>
  idx_t* db_result<idx_t>::getData(std::string idx) {
    if (!dataValid)
      throw new std::invalid_argument("Data not valid");

    idx_t* returnPtr = nullptr;
    for (size_t i = 0; i < names.size(); i++)
      if (names[i] == idx)
        returnPtr = columns[i];
    return returnPtr;
  }

  template<typename idx_t>
  void db_result<idx_t>::addColumn(std::string columnName) {
    if (dataValid)
      throw new std::invalid_argument("Cannot add a column to an allocated result");
    names.push_back(columnName);
  }

  template<typename idx_t>
  void db_result<idx_t>::allocateColumns(idx_t size) {
    if (dataValid)
      throw new std::invalid_argument("Already allocated columns");
    for (size_t i = 0; i < names.size(); i++) {
      idx_t* colPtr = nullptr;
      ALLOC_TRY(&colPtr, sizeof(idx_t) * size, nullptr);
      columns.push_back(colPtr);
    }
  }

  template class db_result<int32_t>;
  template class db_result<int64_t>;

  template<typename idx_t>
  db_table<idx_t>::db_table() {

  }

  template<typename idx_t>
  db_table<idx_t>::~db_table() {
    for (size_t i = 0; i < columns.size(); i++) {
      if (columns[i]->data != nullptr)
        ALLOC_FREE_TRY(columns[i]->data, nullptr);
      free(columns[i]);
    }
  }

  template<typename idx_t>
  void db_table<idx_t>::addColumn(std::string name) {
    if (columns.size() > (size_t)0 && columns[0]->size > 0)
      throw new std::invalid_argument("Can't add a column to a non-empty table");

    gdf_column* _col = (gdf_column*) malloc(sizeof(gdf_column));
    gdf_col_set_defaults(_col);
    gdf_column_view(_col,
                    nullptr,
                    nullptr,
                    0,
                    std::is_same<idx_t, int32_t>::value ? GDF_INT32 : GDF_INT64);
    columns.push_back(_col);
    names.push_back(name);
    indices.resize(indices.size() + 1);
  }

  template<typename idx_t>
  void db_table<idx_t>::addEntry(db_pattern<idx_t>& pattern) {
    if (!pattern.isAllConstants())
      throw new std::invalid_argument("Can't add an entry that isn't all constants");
    if ((size_t)pattern.getSize() != columns.size())
      throw new std::invalid_argument("Can't add an entry that isn't the right size");
    inputBuffer.push_back(pattern);
  }

  template<typename idx_t>
  void db_table<idx_t>::rebuildIndices() {
    for (size_t i = 0; i < columns.size(); i++) {
      // Copy the column's data to a new array
      idx_t size = columns[i]->size;
      idx_t* tempColumn;
      ALLOC_TRY(&tempColumn, sizeof(idx_t) * size, nullptr);
      hipMemcpy(tempColumn, columns[i]->data, sizeof(idx_t) * size, hipMemcpyDefault);

      // Construct an array of ascending integers
      idx_t* indirection;
      ALLOC_TRY(&indirection, sizeof(idx_t) * size, nullptr);
      thrust::sequence(rmm::exec_policy(nullptr)->on(nullptr), indirection, indirection + size);

      // Sort the arrays together
      thrust::sort_by_key(rmm::exec_policy(nullptr)->on(nullptr),
                          tempColumn,
                          tempColumn + size,
                          indirection);

      // Compute offsets array based on sorted column
      idx_t maxId;
      hipMemcpy(&maxId, tempColumn + size - 1, sizeof(idx_t), hipMemcpyDefault);
      idx_t *unique, *counts, *runCount;
      ALLOC_TRY(&unique, (maxId + 1) * sizeof(idx_t), nullptr);
      ALLOC_TRY(&counts, (maxId + 1) * sizeof(idx_t), nullptr);
      ALLOC_TRY(&runCount, sizeof(idx_t), nullptr);
      void* tmpStorage = nullptr;
      size_t tmpBytes = 0;
      hipcub::DeviceRunLengthEncode::Encode(tmpStorage,
                                         tmpBytes,
                                         tempColumn,
                                         unique,
                                         counts,
                                         runCount,
                                         size);
      ALLOC_TRY(&tmpStorage, tmpBytes, nullptr);
      hipcub::DeviceRunLengthEncode::Encode(tmpStorage,
                                         tmpBytes,
                                         tempColumn,
                                         unique,
                                         counts,
                                         runCount,
                                         size);
      ALLOC_FREE_TRY(tmpStorage, nullptr);
      idx_t runCount_h;
      hipMemcpy(&runCount_h, runCount, sizeof(idx_t), hipMemcpyDefault);
      idx_t* offsets;

      // Allocating the new offsets array
      ALLOC_TRY(&offsets, (maxId + 2) * sizeof(idx_t), nullptr);

      // Filling values in offsets array from the encoded run lengths
      int threadsPerBlock = 1024;
      int numBlocks = (runCount_h + threadsPerBlock - 1) / threadsPerBlock;
      offsetsKernel<<<numBlocks, threadsPerBlock>>>(runCount_h, unique, counts, offsets);
      cudaCheckError();

      // Taking the exclusive scan of the run lengths to get the final offsets.
      thrust::exclusive_scan(rmm::exec_policy(nullptr)->on(nullptr),
                             offsets,
                             offsets + maxId + 2,
                             offsets);
      ALLOC_FREE_TRY(tempColumn, nullptr);
      ALLOC_FREE_TRY(unique, nullptr);
      ALLOC_FREE_TRY(counts, nullptr);
      ALLOC_FREE_TRY(runCount, nullptr);

      // Assign new offsets array and indirection vector to index
      gdf_column* offsetsCol = (gdf_column*) malloc(sizeof(gdf_column));
      gdf_col_set_defaults(offsetsCol);
      gdf_column_view(offsetsCol,
                      offsets,
                      nullptr,
                      maxId + 2,
                      std::is_same<idx_t, int32_t>::value ? GDF_INT32 : GDF_INT64);

      gdf_column* indirectionCol = (gdf_column*) malloc(sizeof(gdf_column));
      gdf_col_set_defaults(indirectionCol);
      gdf_column_view(indirectionCol,
                      indirection,
                      nullptr,
                      size,
                      std::is_same<idx_t, int32_t>::value ? GDF_INT32 : GDF_INT64);

      indices[i].resetData(offsetsCol, indirectionCol);
    }
  }

  template<typename idx_t>
  void db_table<idx_t>::flush_input() {
    if (inputBuffer.size() == (size_t)0)
      return;
    idx_t tempSize = inputBuffer.size();
    std::vector<idx_t*> tempColumns;
    for (size_t i = 0; i < columns.size(); i++) {
      tempColumns.push_back((idx_t*) malloc(sizeof(idx_t) * tempSize));
      for (idx_t j = 0; j < tempSize; j++) {
        tempColumns.back()[j] = inputBuffer[j].getEntry(i).getConstant();
      }
    }
    inputBuffer.clear();
    idx_t currentSize = columns[0]->size;
    idx_t newSize = currentSize + tempSize;
    std::vector<idx_t*> newColumns;
    for (size_t i = 0; i < columns.size(); i++) {
      idx_t* newCol;
      ALLOC_TRY(&newCol, sizeof(idx_t) * newSize, nullptr);
      newColumns.push_back(newCol);
    }
    for (size_t i = 0; i < columns.size(); i++) {
      if (currentSize > 0)
        hipMemcpy(newColumns[i], columns[i]->data, sizeof(idx_t) * currentSize, hipMemcpyDefault);
      hipMemcpy(newColumns[i] + currentSize,
                 tempColumns[i],
                 sizeof(idx_t) * tempSize,
                 hipMemcpyDefault);
      free(tempColumns[i]);
      if (columns[i]->data != nullptr)
        ALLOC_FREE_TRY(columns[i]->data, nullptr);
      columns[i]->data = newColumns[i];
      columns[i]->size = newSize;
    }

    rebuildIndices();
  }

  template<typename idx_t>
  db_column_index<idx_t>& db_table<idx_t>::getIndex(int idx) {
    return indices[idx];
  }

  template<typename idx_t>
  gdf_column* db_table<idx_t>::getColumn(int idx) {
    return columns[idx];
  }

  template class db_table<int32_t> ;
  template class db_table<int64_t> ;

  template<typename idx_t>
  db_object<idx_t>::db_object() {
    next_id = 0;
    relationshipsTable.addColumn("begin");
    relationshipsTable.addColumn("end");
    relationshipsTable.addColumn("type");
    relationshipPropertiesTable.addColumn("id");
    relationshipPropertiesTable.addColumn("name");
    relationshipPropertiesTable.addColumn("value");
  }

  template<typename idx_t>
  std::string db_object<idx_t>::query(std::string query) {
    return "";
  }

  template class db_object<int32_t> ;
  template class db_object<int64_t> ;
}
