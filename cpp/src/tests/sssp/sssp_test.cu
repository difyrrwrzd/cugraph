#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

#include <gtest/gtest.h>
#include <cugraph.h>
#include <algorithm>
#include <unordered_map>
#include <utility>
#include <queue>
#include <nvgraph/nvgraph.h>
#include <nvgraph_gdf.h>
#include "test_utils.h"
#include "high_res_clock.h"

typedef enum graph_type { RMAT, MTX } GraphType;

template <typename MaxEType, typename MaxVType, typename DistType>
void ref_bfs(const std::vector<MaxEType>& rowPtr,
             const std::vector<MaxVType>& colInd,
             const MaxVType source_vertex,
             std::vector<DistType>& distances,
             std::vector<MaxVType>& predecessors) {
  size_t n = rowPtr.size() - 1;
  size_t nnz = colInd.size();

  ASSERT_LE(n, std::numeric_limits<MaxVType>::max() - 1);
  ASSERT_LE(nnz, std::numeric_limits<MaxEType>::max());
  ASSERT_EQ(distances.size(), rowPtr.size() - 1);

  std::fill(distances.begin(),
            distances.end(),
            std::numeric_limits<DistType>::max());
  std::fill(predecessors.begin(), predecessors.end(), -1);

  std::queue<MaxVType> q;
  q.push(source_vertex);
  distances[source_vertex] = 0;

  while (!q.empty()) {
    MaxVType u = q.front();
    q.pop();

    for (auto iCol = rowPtr[u]; iCol != rowPtr[u + 1]; ++iCol) {
      MaxVType v = colInd[iCol];
      // undiscovered
      if (distances[v] == std::numeric_limits<DistType>::max()) {
        distances[v] = distances[u] + 1;
        predecessors[v] = u;
        q.push(v);
      }
    }
  }
}

template <typename MaxEType, typename MaxVType, typename DistType>
void ref_sssp(const std::vector<MaxEType>& rowPtr,
              const std::vector<MaxVType>& colInd,
              const std::vector<DistType>& weights,
              const MaxVType source_vertex,
              std::vector<DistType>& distances,
              std::vector<MaxVType>& predecessors) {
  size_t n = rowPtr.size() - 1;
  size_t nnz = colInd.size();

  ASSERT_LE(n, std::numeric_limits<MaxVType>::max() - 1);
  ASSERT_LE(nnz, std::numeric_limits<MaxEType>::max());
  ASSERT_EQ(nnz, weights.size());
  ASSERT_EQ(distances.size(), rowPtr.size() - 1);

  std::fill(distances.begin(),
            distances.end(),
            std::numeric_limits<DistType>::max());
  std::fill(predecessors.begin(), predecessors.end(), -1);

  std::set<MaxVType> curr_frontier;
  curr_frontier.insert(source_vertex);
  distances[source_vertex] = 0;
  MaxVType nf = 1;

  while (nf > 0) {
    std::set<MaxVType> next_frontier;
    for (auto it = curr_frontier.begin(); it != curr_frontier.end(); ++it) {
      MaxVType u = *it;

      for (auto iCol = rowPtr[u]; iCol != rowPtr[u + 1]; ++iCol) {
        MaxVType v = colInd[iCol];
        // relax
        if (distances[u] + weights[iCol] < distances[v]) {
          distances[v] = distances[u] + weights[iCol];
          next_frontier.insert(v);
          predecessors[v] = u;
        }
      }
    }

    curr_frontier = next_frontier;
    nf = curr_frontier.size();
  }
}

// do the perf measurements
// enabled by command line parameter s'--perf'
static int PERF = 0;

// iterations for perf tests
// enabled by command line parameter '--perf-iters"
static int PERF_MULTIPLIER = 5;

typedef struct SSSP_Usecase_t {
  GraphType type_;
  std::string config_;
  std::string file_path_;
  int src_;
  SSSP_Usecase_t(const GraphType& type,
                 const std::string& config,
                 const int src)
      : type_(type), config_(config), src_(src) {
    // assume relative paths are relative to RAPIDS_DATASET_ROOT_DIR
    // FIXME: Use platform independent stuff from c++14/17 on compiler update
    if (type_ == MTX) {
      const std::string& rapidsDatasetRootDir = get_rapids_dataset_root_dir();
      if ((config_ != "") && (config_[0] != '/')) {
        file_path_ = rapidsDatasetRootDir + "/" + config_;
      } else {
        file_path_ = config_;
      }
    }
  };
} SSSP_Usecase;

class Tests_SSSP : public ::testing::TestWithParam<SSSP_Usecase> {
 public:
  Tests_SSSP() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {
    if (PERF) {
      for (unsigned int i = 0; i < SSSP_time.size(); ++i) {
        std::cout << SSSP_time[i] / PERF_MULTIPLIER << std::endl;
      }
    }
  }
  virtual void SetUp() {}
  virtual void TearDown() {}

  static std::vector<double> SSSP_time;

  template <typename DistType, bool DoDist, bool DoPreds>
  void run_current_test(const SSSP_Usecase& param) {
    gdf_column col_src, col_dest, col_weights, col_distances, col_preds;

    size_t v, e;
    // Input
    col_src.data = nullptr;
    col_src.dtype = GDF_INT32;
    col_src.valid = nullptr;
    col_src.null_count = 0;

    col_dest.data = nullptr;
    col_dest.dtype = GDF_INT32;
    col_dest.valid = nullptr;
    col_dest.null_count = 0;

    col_weights.data = nullptr;
    if (std::is_same<DistType, float>::value)
      col_weights.dtype = GDF_FLOAT32;
    else if (std::is_same<DistType, double>::value)
      col_weights.dtype = GDF_FLOAT64;
    else
      ASSERT_TRUE(0);
    col_weights.valid = nullptr;
    col_weights.null_count = 0;

    // Output
    col_distances.data = nullptr;
    col_distances.size = 0;
    col_preds.data = nullptr;
    col_preds.size = 0;

    if (param.type_ == RMAT) {
      ASSERT_EQ(
          gdf_grmat_gen(
              param.config_.c_str(), v, e, &col_src, &col_dest, &col_weights),
          GDF_SUCCESS);
    } else if (param.type_ == MTX) {
      int m, k, nnz;
      MM_typecode mc;

      FILE* fpin = fopen(param.file_path_.c_str(), "r");

      ASSERT_EQ(mm_properties<int>(fpin, 1, &mc, &m, &k, &nnz), 0)
          << "could not read Matrix Market file properties"
          << "\n";
      ASSERT_TRUE(mm_is_matrix(mc));
      ASSERT_TRUE(mm_is_coordinate(mc));
      ASSERT_FALSE(mm_is_complex(mc));
      ASSERT_FALSE(mm_is_skew(mc));

      // Allocate memory on host
      std::vector<int> cooRowInd(nnz), cooColInd(nnz);
      std::vector<DistType> cooVal;

      // Read weights if given
      if (!mm_is_pattern(mc)) {
        cooVal.resize(nnz);
        ASSERT_EQ(
            (mm_to_coo<int, DistType>(
                fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], &cooVal[0], NULL)),
            0)
            << "could not read matrix data"
            << "\n";
      } else {
        ASSERT_EQ((mm_to_coo<int, DistType>(
                      fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], NULL, NULL)),
                  0)
            << "could not read matrix data"
            << "\n";
        // Set random weights
        if (std::is_same<DistType, float>::value ||
            std::is_same<DistType, double>::value) {
          cooVal.resize(nnz);
          for (auto i = 0; i < nnz; i++) {
            cooVal[i] = static_cast<DistType>(rand()) /
                static_cast<DistType>(RAND_MAX);
          }
        }
      }

      ASSERT_EQ(fclose(fpin), 0);
      // gdf columns
      create_gdf_column(cooRowInd, &col_src);
      create_gdf_column(cooColInd, &col_dest);
      create_gdf_column(cooVal, &col_weights);
      v = m;
      e = nnz;
    } else {
      ASSERT_TRUE(0);
    }

    // std::cout << "v = " << v << "e = " << e << "\n";
    gdf_graph G;
    ASSERT_EQ(gdf_edge_list_view(&G, &col_src, &col_dest, &col_weights),
              GDF_SUCCESS);

    std::vector<DistType> dist_vec;
    std::vector<int32_t> pred_vec;

    if (DoDist) {
      dist_vec =
          std::vector<DistType>(v, std::numeric_limits<DistType>::max());
      create_gdf_column(dist_vec, &col_distances);
    }

    if (DoPreds) {
      pred_vec = std::vector<int32_t>(v, -1);
      create_gdf_column(pred_vec, &col_preds);
    }

    gdf_error ret;
    HighResClock hr_clock;
    double time_tmp;

    hipDeviceSynchronize();
    if (PERF) {
      hr_clock.start();
      for (int i = 0; i < PERF_MULTIPLIER; ++i) {
        ret = gdf_sssp(&G, &col_distances, &col_preds, param.src_);
        hipDeviceSynchronize();
      }
      hr_clock.stop(&time_tmp);
      SSSP_time.push_back(time_tmp);
    } else {
      ret = gdf_sssp(&G, &col_distances, &col_preds, param.src_);
      hipDeviceSynchronize();
    }

    ASSERT_EQ(ret, GDF_SUCCESS);

    // MTX may have zero-degree vertices. So reset v after conversion to CSR
    v = G.adjList->offsets->size - 1;

    if (DoDist)
      hipMemcpy((void*)&dist_vec[0],
                 col_distances.data,
                 sizeof(DistType) * v,
                 hipMemcpyDeviceToHost);

    if (DoPreds)
      hipMemcpy((void*)&pred_vec[0],
                 col_preds.data,
                 sizeof(int32_t) * v,
                 hipMemcpyDeviceToHost);

    // Create ref host structures

    std::vector<int32_t> vlist(v + 1), elist(e);
    std::vector<DistType> ref_distances(v), weights(e);
    std::vector<int32_t> ref_predecessors(v);

    hipMemcpy((void*)&vlist[0],
               G.adjList->offsets->data,
               sizeof(int32_t) * (v + 1),
               hipMemcpyDeviceToHost);
    hipMemcpy((void*)&elist[0],
               G.adjList->indices->data,
               sizeof(int32_t) * (e),
               hipMemcpyDeviceToHost);
    hipMemcpy((void*)&weights[0],
               G.adjList->edge_data->data,
               sizeof(DistType) * (e),
               hipMemcpyDeviceToHost);

    std::unordered_map<uint64_t, DistType> min_edge_map;

    if (DoPreds) {
      for (auto i = 0; i < v; ++i) {
        for (auto e = vlist[i]; e < vlist[i + 1]; ++e) {
          DistType weight = weights[e];
          uint64_t key = (uint64_t)i << 32 | (uint64_t)elist[e];
          if (min_edge_map.find(key) != min_edge_map.end()) {
            min_edge_map[key] == std::min(weight, min_edge_map[key]);
          } else {
            min_edge_map[key] = weight;
          }
        }
      }
    }

    ref_sssp(
        vlist, elist, weights, param.src_, ref_distances, ref_predecessors);

    for (auto i = 0; i < v; ++i) {
      if (DoDist)
        ASSERT_EQ(dist_vec[i], ref_distances[i])
            << "vid: " << i << "ref dist " << ref_distances[i]
            << " actual dist " << dist_vec[i];

      if (DoPreds) {
        if (pred_vec[i] != -1) {
          uint64_t key = (uint64_t)pred_vec[i] << 32 | (uint64_t)i;
          DistType min_edge_weight = min_edge_map.at(key);
          ASSERT_EQ(ref_distances[pred_vec[i]] + min_edge_weight,
                    ref_distances[i])
              << "vid: " << i << "pred " << pred_vec[i] << " ref dist "
              << ref_distances[i] << " observed " << ref_distances[pred_vec[i]]
              << " + " << min_edge_weight << " = "
              << ref_distances[pred_vec[i]] + min_edge_weight << "\n";
        } else {
          ASSERT_EQ(pred_vec[i], ref_predecessors[i])
              << "vid: " << i << "ref pred " << ref_predecessors[i]
              << " actual " << pred_vec[i];
        }
      }
    }
  }
};

std::vector<double> Tests_SSSP::SSSP_time;

TEST_P(Tests_SSSP, CheckFP32_DIST_NO_PREDS) {
  run_current_test<float, true, false>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP32_NO_DIST_PREDS) {
  run_current_test<float, false, true>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP32_DIST_PREDS) {
  run_current_test<float, true, true>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP64_DIST_NO_PREDS) {
  run_current_test<double, true, false>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP64_NO_DIST_PREDS) {
  run_current_test<double, false, true>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP64_DIST_PREDS) {
  run_current_test<double, true, true>(GetParam());
}

// --gtest_filter=*simple_test*

INSTANTIATE_TEST_CASE_P(
    simple_test,
    Tests_SSSP,
    ::testing::Values(
        SSSP_Usecase(RMAT,
                     "grmat --rmat_scale=10 --rmat_edgefactor=16 --device=0  "
                     "--normalized --quiet",
                     0),
        SSSP_Usecase(RMAT,
                     "grmat --rmat_scale=12 --rmat_edgefactor=8 --device=0  "
                     "--normalized --quiet",
                     10),
        SSSP_Usecase(MTX, "test/datasets/dblp.mtx", 100),
        SSSP_Usecase(MTX, "test/datasets/wiki2003.mtx", 100000),
        SSSP_Usecase(MTX, "test/datasets/karate.mtx", 1)
        //,SSSP_Usecase(MTX, "test/datasets/cit-Patents.mtx", 100)
        ));

int main(int argc, char** argv) {
  srand(42);
  ::testing::InitGoogleTest(&argc, argv);

  for (int i = 0; i < argc; i++) {
    if (strcmp(argv[i], "--perf") == 0) {
      PERF = 1;
    }
    if (strcmp(argv[i], "--perf-iters") == 0) {
      PERF_MULTIPLIER = atoi(argv[i + 1]);
    }
  }
  return RUN_ALL_TESTS();
}
