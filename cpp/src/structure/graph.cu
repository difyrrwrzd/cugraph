#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "utilities/cuda_utils.cuh"
#include "utilities/error_utils.h"
#include "utilities/graph_utils.cuh"

#include <graph.hpp>

namespace {

template <typename vertex_t, typename edge_t>
void degree_from_offsets(vertex_t number_of_vertices, edge_t const *offsets,
                         edge_t *degree, hipStream_t stream) {

  // Computes out-degree for x = 0 and x = 2
  thrust::for_each(rmm::exec_policy(stream)->on(stream),
                   thrust::make_counting_iterator<vertex_t>(0),
                   thrust::make_counting_iterator<vertex_t>(number_of_vertices),
                   [offsets, degree] __device__(vertex_t v) {
                     degree[v] = offsets[v + 1] - offsets[v];
                   });
}

template <typename vertex_t, typename edge_t>
void degree_from_vertex_ids(const cugraph::experimental::Comm &comm,
                            vertex_t number_of_vertices, edge_t number_of_edges,
                            vertex_t const *indices, edge_t *degree,
                            hipStream_t stream) {

  thrust::for_each(rmm::exec_policy(stream)->on(stream),
                   thrust::make_counting_iterator<edge_t>(0),
                   thrust::make_counting_iterator<edge_t>(number_of_edges),
                   [indices, degree] __device__(edge_t e) {
                     cugraph::atomicAdd(degree + indices[e], 1);
                   });
  comm.allreduce(number_of_vertices, degree, degree,
                 cugraph::experimental::ReduceOp::SUM);
}

} // namespace

namespace cugraph {
namespace experimental {

template <typename VT, typename ET, typename WT>
void GraphBase<VT, ET, WT>::get_vertex_identifiers(VT *identifiers) const {
  cugraph::detail::sequence<VT>(number_of_vertices, identifiers);
}

template <typename VT, typename ET, typename WT>
void GraphCompressedSparseBase<VT, ET, WT>::get_source_indices(
    VT *src_indices) const {
  CUGRAPH_EXPECTS(offsets != nullptr, "No graph specified");
  cugraph::detail::offsets_to_indices<VT>(
      offsets, GraphBase<VT, ET, WT>::number_of_vertices, src_indices);
}

template <typename VT, typename ET, typename WT>
void GraphCOO<VT, ET, WT>::degree(ET *degree, DegreeDirection direction) const {
  //
  // NOTE:  We assume offsets/indices are a CSR.  If a CSC is passed
  //        in then x should be modified to reflect the expected direction.
  //        (e.g. if you have a CSC and you want in-degree (x=1) then pass
  //        the offsets/indices and request an out-degree (x=2))
  //
  hipStream_t stream{nullptr};

  if (direction != DegreeDirection::IN) {
    if (GraphBase<VT, ET, WT>::comm.get_p()) // FIXME retrieve global source
                                             // indexing for the allreduce work
      CUGRAPH_FAIL("OPG degree not implemented for OUT degree");
    degree_from_vertex_ids(
        GraphBase<VT, ET, WT>::comm, GraphBase<VT, ET, WT>::number_of_vertices,
        GraphBase<VT, ET, WT>::number_of_edges, src_indices, degree, stream);
  }

  if (direction != DegreeDirection::OUT) {
    degree_from_vertex_ids(
        GraphBase<VT, ET, WT>::comm, GraphBase<VT, ET, WT>::number_of_vertices,
        GraphBase<VT, ET, WT>::number_of_edges, dst_indices, degree, stream);
  }
}

template <typename VT, typename ET, typename WT>
void GraphCompressedSparseBase<VT, ET, WT>::degree(
    ET *degree, DegreeDirection direction) const {
  //
  // NOTE:  We assume offsets/indices are a CSR.  If a CSC is passed
  //        in then x should be modified to reflect the expected direction.
  //        (e.g. if you have a CSC and you want in-degree (x=1) then pass
  //        the offsets/indices and request an out-degree (x=2))
  //
  hipStream_t stream{nullptr};

  if (direction != DegreeDirection::IN) {
    if (GraphBase<VT, ET, WT>::comm.get_p())
      CUGRAPH_FAIL(
          "OPG degree not implemented for OUT degree"); // FIXME retrieve global
                                                        // source indexing for
                                                        // the allreduce to work
    degree_from_offsets(GraphBase<VT, ET, WT>::number_of_vertices, offsets,
                        degree, stream);
  }

  if (direction != DegreeDirection::OUT) {
    degree_from_vertex_ids(
        GraphBase<VT, ET, WT>::comm, GraphBase<VT, ET, WT>::number_of_vertices,
        GraphBase<VT, ET, WT>::number_of_edges, indices, degree, stream);
  }
}

// explicit instantiation
template class GraphBase<int32_t, int32_t, float>;
template class GraphBase<int32_t, int32_t, double>;
template class GraphCOO<int32_t, int32_t, float>;
template class GraphCOO<int32_t, int32_t, double>;
template class GraphCompressedSparseBase<int32_t, int32_t, float>;
template class GraphCompressedSparseBase<int32_t, int32_t, double>;
} // namespace experimental
} // namespace cugraph
