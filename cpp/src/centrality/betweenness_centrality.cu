#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <vector>

#include <thrust/transform.h>

#include <algorithms.hpp>
#include <graph.hpp>
#include "rmm_utils.h"

#include <utilities/error_utils.h>

#include <gunrock/gunrock.h>

#include "betweenness_centrality.cuh"

namespace cugraph {

namespace detail {
template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::setup() {
    // --- Set up parameters from graph adjList ---
    number_vertices  = graph.number_of_vertices;
    number_edges = graph.number_of_edges;
    offsets_ptr = graph.offsets;
    indices_ptr = graph.indices;
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::configure(result_t *_betweenness, bool _normalize,
                                         VT const *_sample_seeds,
                                         VT _number_of_sample_seeds) {
    // --- Bind betweenness output vector to internal ---
    betweenness = _betweenness;
    apply_normalization = _normalize;
    sample_seeds = _sample_seeds;
    number_of_sample_seeds =  _number_of_sample_seeds;

    // --- Working data allocation ---
    ALLOC_TRY(&distances, number_vertices * sizeof(VT), nullptr);
    ALLOC_TRY(&predecessors, number_vertices * sizeof(VT), nullptr);
    ALLOC_TRY(&nodes, number_vertices * sizeof(VT), nullptr);
    ALLOC_TRY(&sp_counters, number_vertices * sizeof(int), nullptr);
    ALLOC_TRY(&deltas, number_vertices * sizeof(result_t), nullptr);
    // --- Confirm that configuration went through ---
    configured = true;
}
template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::clean() {
    ALLOC_FREE_TRY(distances, nullptr);
    ALLOC_FREE_TRY(predecessors, nullptr);
    ALLOC_FREE_TRY(nodes, nullptr);
    ALLOC_FREE_TRY(sp_counters, nullptr);
    ALLOC_FREE_TRY(deltas, nullptr);
    // ---  Betweenness is not ours ---
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::normalize() {
    printf("[DBG] Being normalized\n");
    thrust::device_vector<result_t> normalizer(number_vertices);
    thrust::fill(normalizer.begin(), normalizer.end(), ((number_vertices - 1) * (number_vertices - 2)));

    if (typeid(result_t) == typeid(float)) {
        thrust::transform(rmm::exec_policy(stream)->on(stream), betweenness, betweenness + number_vertices, normalizer.begin(), betweenness, thrust::divides<float>());
    } else if (typeid(result_t) == typeid(double)) {
        thrust::transform(rmm::exec_policy(stream)->on(stream), betweenness, betweenness + number_vertices, normalizer.begin(), betweenness, thrust::divides<double>());
    }
}

/* TODO(xcadet) Use an iteration based node system, to process nodes of the same level at the same time
** For now all the work is done on the first thread */
template <typename VT, typename ET, typename WT, typename result_t>
__global__ void accumulation_kernel_old(result_t *betweenness, VT number_vertices,
                                  VT const *indices, ET const *offsets,
                                  VT *distances,
                                  int *sp_counters,
                                  result_t *deltas, VT source, VT depth) {
 //int gid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int gid = blockIdx.x * blockDim.x + threadIdx.x; gid < number_vertices;
       gid += gridDim.x * blockDim.x) {
  //for (int gid = blockIdx.x * blockDim.x + threadIdx.x;
       //gid < number_vertices; gid += blockDim.x * gridDim.x) {
    VT v = gid;
    // TODO(xcadet) Use a for loop using strides
    if (distances[v] == depth) { // Process nodes at this depth
      ET edge_start = offsets[v];
      ET edge_end = offsets[v + 1];
      ET edge_count = edge_end - edge_start;
      for (ET edge_idx = 0; edge_idx < edge_count; ++edge_idx) { // Visit neighbors
        VT w =  indices[edge_start + edge_idx];
        if (distances[w] == depth + 1) { // Current node is a predecessor
          result_t factor = (static_cast<result_t>(1.0) + deltas[w]) / static_cast<result_t>(sp_counters[w]);
          atomicAdd(&deltas[v], static_cast<result_t>(sp_counters[v]) * factor);
        }
      }
        atomicAdd(&betweenness[v], deltas[v]);
    }
  }
}
// Dependecy Accumulation: McLaughlin and Bader, 2018
template <typename VT, typename ET, typename WT, typename result_t>
__global__ void accumulation_kernel(result_t *betweenness, VT number_vertices,
                                  VT const *indices, ET const *offsets,
                                  VT *distances,
                                  int *sp_counters,
                                  result_t *deltas, VT source, VT depth) {
  for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < number_vertices;
       tid += gridDim.x * blockDim.x) {
    VT w = tid;
    result_t dsw = 0;
    result_t sw = static_cast<result_t>(sp_counters[w]);
    if (distances[w] == depth) { // Process nodes at this depth
      ET edge_start = offsets[w];
      ET edge_end = offsets[w + 1];
      ET edge_count = edge_end - edge_start;
      for (ET edge_idx = 0; edge_idx < edge_count; ++edge_idx) { // Visit neighbors
        VT v = indices[edge_start + edge_idx];
        if (distances[v] == distances[w] + 1) {
          result_t factor = (static_cast<result_t>(1) + deltas[v]) / static_cast<result_t>(sp_counters[v]);
          dsw += sw * factor;
        }
      }
      deltas[w] = dsw;
    }
  }
}

// TODO(xcadet) We might be able to handle different nodes with a kernel
// With BFS distances can be used to handle accumulation,
template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::accumulate(result_t *betweenness, VT* distances,
                                          VT *sp_counters,
                                          result_t *deltas, VT source, VT max_depth) {
    dim3 grid, block;
    block.x = 1; // TODO(xcadet) Replace these values, only for debugging
    grid.x = 1;
  // Step 1) Dependencies (deltas) are initialized to 0 before starting
  thrust::fill(rmm::exec_policy(stream)->on(stream), deltas,
               deltas + number_vertices, static_cast<result_t>(0));
  // Step 2) Process each node, -1 is used to notify unreached nodes in the sssp
  for (VT depth = max_depth; depth > 0; --depth) {
    //std::cout << "\t[ACC] Processing depth: " << depth << std::endl;
    accumulation_kernel<VT, ET, WT, result_t>
                     <<<grid, block, 0, stream>>>(betweenness, number_vertices,
                                             graph.indices, graph.offsets,
                                             distances, sp_counters,
                                             deltas, source, depth);
    hipDeviceSynchronize();
  }

  thrust::transform(rmm::exec_policy(stream)->on(stream),
    deltas, deltas + number_vertices, betweenness, betweenness, thrust::plus<result_t>());
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::check_input() {
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::compute() {
    CUGRAPH_EXPECTS(configured, "BC must be configured before computation");
    thrust::device_vector<VT> d_sp_counters(number_vertices, 0);
    thrust::device_vector<VT> d_distances(number_vertices, 0);
    thrust::device_vector<result_t> d_deltas(number_vertices, 0);
    for (int source_vertex = 0; source_vertex < number_vertices;
         ++source_vertex) {
        // Step 1) Singe-source shortest-path problem
        cugraph::bfs(graph, thrust::raw_pointer_cast(d_distances.data()), predecessors, thrust::raw_pointer_cast(d_sp_counters.data()), source_vertex,
                     graph.prop.directed);
        hipDeviceSynchronize();

        //TODO(xcadet) Remove that with a BC specific class to gather
        //             information during traversal
        // NOTE: REPLACE INFINITY BY -1 otherwise the max depth will be maximal
        //       value!
        thrust::replace(rmm::exec_policy(stream)->on(stream), d_distances.begin(),
                        d_distances.end(),
                        std::numeric_limits<VT>::max(),
                        static_cast<VT>(-1));
        auto value = thrust::max_element(d_distances.begin(), d_distances.end());

        accumulate(betweenness, thrust::raw_pointer_cast(d_distances.data()), thrust::raw_pointer_cast(d_sp_counters.data()), thrust::raw_pointer_cast(d_deltas.data()), source_vertex, *value);
        /*
        std::cout << "Deltas" << std::endl;
        thrust::copy(d_deltas.begin(), d_deltas.end(), std::ostream_iterator<result_t>(std::cout, ", "));
        std::cout << std::endl;
        */
    }
    hipDeviceSynchronize();
    if (apply_normalization) {
        normalize();
    }
}
  /**
  * ---------------------------------------------------------------------------*
  * @brief Native betweenness centrality
  *
  * @file betweenness_centrality.cu
  * --------------------------------------------------------------------------*/
  template <typename VT, typename ET, typename WT, typename result_t>
  void betweenness_centrality(experimental::GraphCSR<VT,ET,WT> const &graph,
                            result_t *result,
                            bool normalize,
                            VT const *sample_seeds = nullptr,
                            VT number_of_sample_seeds = 0) {
    printf("[DBG][BC] BETWEENNESS CENTRALITY NATIVE_CUGPRAPH\n");
    CUGRAPH_EXPECTS(result != nullptr, "Invalid API parameter: output betwenness is nullptr");
    if (typeid(VT) != typeid(int)) {
      CUGRAPH_FAIL("Unsupported vertex id data type, please use int");
    }
    if (typeid(ET) != typeid(int)) {
      CUGRAPH_FAIL("Unsupported edge id data type, please use int");
    }
    if (typeid(WT) != typeid(float) && typeid(WT) != typeid(double)) {
      CUGRAPH_FAIL("Unsupported weight data type, please use float or double");
    }

    CUGRAPH_EXPECTS(sample_seeds == nullptr, "Sampling seeds is currently not supported");
    // Current Implementation relies on BFS
    // FIXME: For SSSP version
    // Brandes Algorithm excpets non negative weights for the accumulation
    cugraph::detail::BC<VT, ET, WT, result_t> bc(graph);
    bc.configure(result, normalize, sample_seeds, number_of_sample_seeds);
    bc.compute();
  }
} // !cugraph::detail

namespace gunrock {

template <typename VT, typename ET, typename WT, typename result_t>
void betweenness_centrality(experimental::GraphCSR<VT,ET,WT> const &graph,
                            result_t *result,
                            bool normalize,
                            VT const *sample_seeds = nullptr,
                            VT number_of_sample_seeds = 0) {

  hipStream_t stream{nullptr};

  //
  //  gunrock currently (as of 2/28/2020) only operates on a graph and results in
  //  host memory.  [That is, the first step in gunrock is to allocate device memory
  //  and copy the data into device memory, the last step is to allocate host memory
  //  and copy the results into the host memory]
  //
  //  They are working on fixing this.  In the meantime, to get the features into
  //  cuGraph we will first copy the graph back into local memory and when we are finished
  //  copy the result back into device memory.
  //
  std::vector<ET>        v_offsets(graph.number_of_vertices + 1);
  std::vector<VT>        v_indices(graph.number_of_edges);
  std::vector<result_t>  v_result(graph.number_of_vertices);
  std::vector<float>     v_sigmas(graph.number_of_vertices);
  std::vector<int>       v_labels(graph.number_of_vertices);
  
  // fill them
  CUDA_TRY(hipMemcpy(v_offsets.data(), graph.offsets, sizeof(ET) * (graph.number_of_vertices + 1), hipMemcpyDeviceToHost));
  CUDA_TRY(hipMemcpy(v_indices.data(), graph.indices, sizeof(VT) * graph.number_of_edges, hipMemcpyDeviceToHost));

  if (sample_seeds == nullptr) {
    bc(graph.number_of_vertices,
       graph.number_of_edges,
       v_offsets.data(),
       v_indices.data(),
       -1,
       v_result.data(),
       v_sigmas.data(),
       v_labels.data());
  } else {
    //
    //  Gunrock, as currently implemented
    //  doesn't support this method.
    //
    CUGRAPH_FAIL("gunrock doesn't currently support sampling seeds");
  }

  // copy to results
  CUDA_TRY(hipMemcpy(result, v_result.data(), sizeof(result_t) * graph.number_of_vertices, hipMemcpyHostToDevice));

  // normalize result
  if (normalize) {
    float denominator = (graph.number_of_vertices - 1) * (graph.number_of_vertices - 2);

    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      result, result + graph.number_of_vertices, result,
                      [denominator] __device__ (float f) {
                        return (f * 2) / denominator;
                      });
  } else {
    //
    //  gunrock answer needs to be doubled to match networkx
    //
    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      result, result + graph.number_of_vertices, result,
                      [] __device__ (float f) {
                        return (f * 2);
                      });
  }
}

} // namespace detail

template <typename VT, typename ET, typename WT, typename result_t>
void betweenness_centrality(experimental::GraphCSR<VT,ET,WT> const &graph,
                            result_t *result,
                            bool normalize,
                            bool endpoints,
                            WT const *weight,
                            VT k,
                            VT const *vertices) {

  //
  // NOTE:  gunrock implementation doesn't yet support the unused parameters:
  //     - endpoints
  //     - weight
  //     - k
  //     - vertices
  //
  // These parameters are present in the API to support future features.
  //
  //gunrock::betweenness_centrality(graph, result, normalize);
  detail::betweenness_centrality(graph, result, normalize);
}

template void betweenness_centrality<int, int, float, float>(experimental::GraphCSR<int,int,float> const &, float*, bool, bool, float const *, int, int const *);

} //namespace cugraph

