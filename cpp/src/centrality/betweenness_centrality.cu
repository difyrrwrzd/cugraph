#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <vector>

#include <thrust/transform.h>

#include <raft/cudart_utils.h>

#include <algorithms.hpp>
#include <graph.hpp>
#include <utilities/error.hpp>

#include <raft/handle.hpp>
#include "betweenness_centrality.cuh"
#include "betweenness_centrality_kernels.cuh"

namespace cugraph {
namespace detail {
namespace {
template <typename VT, typename ET, typename WT, typename result_t>
void betweenness_centrality_impl(raft::handle_t const &handle,
                                 GraphCSRView<VT, ET, WT> const &graph,
                                 result_t *result,
                                 bool normalize,
                                 bool endpoints,
                                 WT const *weight,
                                 VT number_of_sources,
                                 VT const *sources,
                                 VT total_number_of_sources)
{
  // Current Implementation relies on BFS
  // FIXME: For SSSP version
  // Brandes Algorithm expects non negative weights for the accumulation
  bool is_edge_betweenness = false;
  verify_betweenness_centrality_input<VT, ET, WT, result_t>(
    result, is_edge_betweenness, normalize, endpoints, weight, number_of_sources, sources);
  cugraph::detail::BC<VT, ET, WT, result_t> bc(handle, graph);
  bc.configure(
    result, is_edge_betweenness, normalize, endpoints, weight, sources, number_of_sources);
  bc.compute();
  bc.rescale_by_total_sources_used(total_number_of_sources);
}

template <typename VT, typename ET, typename WT, typename result_t>
void edge_betweenness_centrality_impl(raft::handle_t const &handle,
                                      GraphCSRView<VT, ET, WT> const &graph,
                                      result_t *result,
                                      bool normalize,
                                      WT const *weight,
                                      VT number_of_sources,
                                      VT const *sources,
                                      VT total_number_of_sources)
{
  // Current Implementation relies on BFS
  // FIXME: For SSSP version
  // Brandes Algorithm expects non negative weights for the accumulation
  bool is_edge_betweenness = true;
  bool endpoints           = false;
  verify_betweenness_centrality_input<VT, ET, WT, result_t>(
    result, is_edge_betweenness, normalize, endpoints, weight, number_of_sources, sources);
  cugraph::detail::BC<VT, ET, WT, result_t> bc(handle, graph);
  bc.configure(
    result, is_edge_betweenness, normalize, endpoints, weight, sources, number_of_sources);
  bc.compute();
  // NOTE: As of 07/2020 NetworkX does not apply rescaling based on number
  // of sources
  // bc.rescale_by_total_sources_used(total_number_of_sources);
}
}  // namespace

template <typename VT, typename ET, typename WT, typename result_t>
void verify_betweenness_centrality_input(result_t *result,
                                         bool is_edge_betweenness,
                                         bool normalize,
                                         bool endpoints,
                                         WT const *weights,
                                         VT const number_of_sources,
                                         VT const *sources)
{
  static_assert(std::is_same<VT, int>::value, "VT should be int");
  static_assert(std::is_same<ET, int>::value, "ET should be int");
  static_assert(std::is_same<WT, float>::value || std::is_same<WT, double>::value,
                "WT should be float or double");
  static_assert(std::is_same<result_t, float>::value || std::is_same<result_t, double>::value,
                "result_t should be float or double");

  CUGRAPH_EXPECTS(result != nullptr, "Invalid API parameter: betwenness pointer is NULL");
  CUGRAPH_EXPECTS(number_of_sources >= 0, "Number of sources must be positive or equal to 0.");
  if (number_of_sources != 0) {
    CUGRAPH_EXPECTS(sources != nullptr,
                    "Sources cannot be NULL if number_of_source is different from 0.");
  }
  if (is_edge_betweenness) {
    CUGRAPH_EXPECTS(!endpoints, "Endpoints is not supported for edge betweenness centrality.");
  }
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::setup()
{
  number_of_vertices_ = graph_.number_of_vertices;
  number_of_edges_    = graph_.number_of_edges;
  offsets_ptr_        = graph_.offsets;
  indices_ptr_        = graph_.indices;
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::configure(result_t *betweenness,
                                         bool is_edge_betweenness,
                                         bool normalized,
                                         bool endpoints,
                                         WT const *weights,
                                         VT const *sources,
                                         VT number_of_sources)
{
  // --- Bind betweenness output vector to internal ---
  betweenness_         = betweenness;
  normalized_          = normalized;
  endpoints_           = endpoints;
  sources_             = sources;
  number_of_sources_   = number_of_sources;
  edge_weights_ptr_    = weights;
  is_edge_betweenness_ = is_edge_betweenness;

  // --- Working data allocation ---
  initialize_work_vectors();
  initialize_pointers_to_vectors();

  // --- Get Device Information ---
  initialize_device_information();

  // --- Confirm that configuration went through ---
  configured_ = true;
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::initialize_work_vectors()
{
  distances_vec_.resize(number_of_vertices_);
  predecessors_vec_.resize(number_of_vertices_);
  sp_counters_vec_.resize(number_of_vertices_);
  deltas_vec_.resize(number_of_vertices_);
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::initialize_pointers_to_vectors()
{
  distances_    = distances_vec_.data().get();
  predecessors_ = predecessors_vec_.data().get();
  sp_counters_  = sp_counters_vec_.data().get();
  deltas_       = deltas_vec_.data().get();
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::initialize_device_information()
{
  max_grid_dim_1D_  = handle_.get_device_properties().maxGridSize[0];
  max_block_dim_1D_ = handle_.get_device_properties().maxThreadsDim[0];
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::compute()
{
  CUGRAPH_EXPECTS(configured_, "BC must be configured before computation");
  if (sources_) {
    for (VT source_idx = 0; source_idx < number_of_sources_; ++source_idx) {
      VT source_vertex = sources_[source_idx];
      compute_single_source(source_vertex);
    }
  } else {
    for (VT source_vertex = 0; source_vertex < number_of_vertices_; ++source_vertex) {
      compute_single_source(source_vertex);
    }
  }
  rescale();
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::compute_single_source(VT source_vertex)
{
  // Step 1) Singe-source shortest-path problem
  cugraph::bfs(
    handle_, graph_, distances_, predecessors_, sp_counters_, source_vertex, graph_.prop.directed);

  // FIXME: Remove that with a BC specific class to gather
  //        information during traversal

  // Numeric max value is replaced by -1 as we look for the maximal depth of
  // the traversal, this value is avalaible within the bfs implementation and
  // there could be a way to access it directly and avoid both replace and the
  // max
  thrust::replace(rmm::exec_policy(stream_)->on(stream_),
                  distances_,
                  distances_ + number_of_vertices_,
                  std::numeric_limits<VT>::max(),
                  static_cast<VT>(-1));
  auto current_max_depth = thrust::max_element(
    rmm::exec_policy(stream_)->on(stream_), distances_, distances_ + number_of_vertices_);
  VT max_depth = 0;
  CUDA_TRY(hipMemcpy(&max_depth, current_max_depth, sizeof(VT), hipMemcpyDeviceToHost));
  // Step 2) Dependency accumulation
  accumulate(source_vertex, max_depth);
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::accumulate(VT source_vertex, VT max_depth)
{
  dim3 grid_configuration, block_configuration;
  block_configuration.x = max_block_dim_1D_;
  grid_configuration.x  = min(max_grid_dim_1D_, (number_of_edges_ / block_configuration.x + 1));

  initialize_dependencies();

  if (is_edge_betweenness_) {
    accumulate_edges(max_depth, grid_configuration, block_configuration);
  } else if (endpoints_) {
    accumulate_vertices_with_endpoints(
      source_vertex, max_depth, grid_configuration, block_configuration);
  } else {
    accumulate_vertices(max_depth, grid_configuration, block_configuration);
  }
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::initialize_dependencies()
{
  thrust::fill(rmm::exec_policy(stream_)->on(stream_),
               deltas_,
               deltas_ + number_of_vertices_,
               static_cast<result_t>(0));
}
template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::accumulate_edges(VT max_depth,
                                                dim3 grid_configuration,
                                                dim3 block_configuration)
{
  for (VT depth = max_depth; depth >= 0; --depth) {
    edges_accumulation_kernel<VT, ET, WT, result_t>
      <<<grid_configuration, block_configuration, 0, stream_>>>(betweenness_,
                                                                number_of_vertices_,
                                                                graph_.indices,
                                                                graph_.offsets,
                                                                distances_,
                                                                sp_counters_,
                                                                deltas_,
                                                                depth);
  }
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::accumulate_vertices_with_endpoints(VT source_vertex,
                                                                  VT max_depth,
                                                                  dim3 grid_configuration,
                                                                  dim3 block_configuration)
{
  for (VT depth = max_depth; depth > 0; --depth) {
    endpoints_accumulation_kernel<VT, ET, WT, result_t>
      <<<grid_configuration, block_configuration, 0, stream_>>>(betweenness_,
                                                                number_of_vertices_,
                                                                graph_.indices,
                                                                graph_.offsets,
                                                                distances_,
                                                                sp_counters_,
                                                                deltas_,
                                                                depth);
  }
  add_reached_endpoints_to_source_betweenness(source_vertex);
  add_vertices_dependencies_to_betweenness();
}

// Distances should contain -1 for unreached nodes,

// FIXME: There might be a cleaner way to add a value to a single
//        score in the betweenness vector
template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::add_reached_endpoints_to_source_betweenness(VT source_vertex)
{
  VT number_of_unvisited_vertices = thrust::count(
    rmm::exec_policy(stream_)->on(stream_), distances_, distances_ + number_of_vertices_, -1);
  VT number_of_visited_vertices_except_source =
    number_of_vertices_ - number_of_unvisited_vertices - 1;
  rmm::device_vector<VT> buffer(1);
  buffer[0] = number_of_visited_vertices_except_source;
  thrust::transform(rmm::exec_policy(stream_)->on(stream_),
                    buffer.begin(),
                    buffer.end(),
                    betweenness_ + source_vertex,
                    betweenness_ + source_vertex,
                    thrust::plus<result_t>());
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::add_vertices_dependencies_to_betweenness()
{
  thrust::transform(rmm::exec_policy(stream_)->on(stream_),
                    deltas_,
                    deltas_ + number_of_vertices_,
                    betweenness_,
                    betweenness_,
                    thrust::plus<result_t>());
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::accumulate_vertices(VT max_depth,
                                                   dim3 grid_configuration,
                                                   dim3 block_configuration)
{
  for (VT depth = max_depth; depth > 0; --depth) {
    accumulation_kernel<VT, ET, WT, result_t>
      <<<grid_configuration, block_configuration, 0, stream_>>>(betweenness_,
                                                                number_of_vertices_,
                                                                graph_.indices,
                                                                graph_.offsets,
                                                                distances_,
                                                                sp_counters_,
                                                                deltas_,
                                                                depth);
  }
  add_vertices_dependencies_to_betweenness();
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::rescale()
{
  bool modified           = false;
  result_t rescale_factor = static_cast<result_t>(1);
  if (normalized_) {
    if (is_edge_betweenness_) {
      rescale_edges_betweenness_centrality(rescale_factor, modified);
    } else {
      rescale_vertices_betweenness_centrality(rescale_factor, modified);
    }
  } else {
    if (!graph_.prop.directed) {
      rescale_factor /= static_cast<result_t>(2);
      modified = true;
    }
  }
  apply_rescale_factor_to_betweenness(rescale_factor);
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::rescale_edges_betweenness_centrality(result_t &rescale_factor,
                                                                    bool &modified)
{
  result_t casted_number_of_vertices_ = static_cast<result_t>(number_of_vertices_);
  if (number_of_vertices_ > 1) {
    rescale_factor /= ((casted_number_of_vertices_) * (casted_number_of_vertices_ - 1));
    modified = true;
  }
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::rescale_vertices_betweenness_centrality(result_t &rescale_factor,
                                                                       bool &modified)
{
  result_t casted_number_of_vertices = static_cast<result_t>(number_of_vertices_);
  if (number_of_vertices_ > 2) {
    if (endpoints_) {
      rescale_factor /= (casted_number_of_vertices * (casted_number_of_vertices - 1));
    } else {
      rescale_factor /= ((casted_number_of_vertices - 1) * (casted_number_of_vertices - 2));
    }
    modified = true;
  }
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::apply_rescale_factor_to_betweenness(result_t rescale_factor)
{
  size_t result_size = number_of_vertices_;
  if (is_edge_betweenness_) result_size = number_of_edges_;
  thrust::transform(rmm::exec_policy(stream_)->on(stream_),
                    betweenness_,
                    betweenness_ + result_size,
                    thrust::make_constant_iterator(rescale_factor),
                    betweenness_,
                    thrust::multiplies<result_t>());
}

template <typename VT, typename ET, typename WT, typename result_t>
void BC<VT, ET, WT, result_t>::rescale_by_total_sources_used(VT total_number_of_sources_used)
{
  result_t rescale_factor = static_cast<result_t>(1);
  result_t casted_total_number_of_sources_used =
    static_cast<result_t>(total_number_of_sources_used);
  result_t casted_number_of_vertices = static_cast<result_t>(number_of_vertices_);

  if (normalized_) {
    if (number_of_vertices_ > 2 && total_number_of_sources_used > 0) {
      rescale_factor *= (casted_number_of_vertices / casted_total_number_of_sources_used);
    }
  } else if (!graph_.prop.directed) {
    if (number_of_vertices_ > 2 && total_number_of_sources_used > 0) {
      rescale_factor *= (casted_number_of_vertices / casted_total_number_of_sources_used);
    }
  }
  apply_rescale_factor_to_betweenness(rescale_factor);
}
}  // namespace detail

template <typename VT, typename ET, typename WT, typename result_t>
void betweenness_centrality(raft::handle_t const &handle,
                            GraphCSRView<VT, ET, WT> const *graph,
                            result_t *result,
                            bool normalize,
                            bool endpoints,
                            WT const *weight,
                            VT k,
                            VT const *vertices,
                            VT total_number_of_sources_used)
{
  if (handle.comms_initialized()) {
    int rank = handle.get_comms().get_rank();
    cugraph::opg::DSGGraphCSR<VT, ET, WT> local_holder(handle, graph);
    local_holder.distribute();

    rmm::device_vector<result_t> betweenness(local_holder.graph.number_of_vertices, 0);
    detail::betweenness_centrality_impl(handle,
                                        local_holder.graph,
                                        betweenness.data().get(),
                                        normalize,
                                        endpoints,
                                        weight,
                                        k,
                                        vertices,
                                        total_number_of_sources_used);
    handle.get_comms().reduce(
      betweenness.data().get(), result, betweenness.size(), raft::comms::op_t::SUM, 0, 0);
  } else {
    detail::betweenness_centrality_impl(handle,
                                        *graph,
                                        result,
                                        normalize,
                                        endpoints,
                                        weight,
                                        k,
                                        vertices,
                                        total_number_of_sources_used);
  }
}

template void betweenness_centrality<int, int, float, float>(const raft::handle_t &,
                                                             GraphCSRView<int, int, float> const *,
                                                             float *,
                                                             bool,
                                                             bool,
                                                             float const *,
                                                             int,
                                                             int const *,
                                                             int);
template void betweenness_centrality<int, int, double, double>(
  const raft::handle_t &,
  GraphCSRView<int, int, double> const *,
  double *,
  bool,
  bool,
  double const *,
  int,
  int const *,
  int);

template <typename VT, typename ET, typename WT, typename result_t>
void edge_betweenness_centrality(raft::handle_t const &handle,
                                 GraphCSRView<VT, ET, WT> const *graph,
                                 result_t *result,
                                 bool normalize,
                                 WT const *weight,
                                 VT k,
                                 VT const *vertices,
                                 VT total_number_of_sources_used)
{
  if (handle.comms_initialized()) {
    cugraph::opg::DSGGraphCSR<VT, ET, WT> local_holder(handle, graph);
    local_holder.distribute();

    rmm::device_vector<result_t> betweenness(local_holder.graph.number_of_edges, 0);
    detail::edge_betweenness_centrality_impl(handle,
                                             local_holder.graph,
                                             betweenness.data().get(),
                                             normalize,
                                             weight,
                                             k,
                                             vertices,
                                             total_number_of_sources_used);
    handle.get_comms().reduce(
      betweenness.data().get(), result, betweenness.size(), raft::comms::op_t::SUM, 0, 0);
  } else {
    detail::edge_betweenness_centrality_impl(
      handle, *graph, result, normalize, weight, k, vertices, total_number_of_sources_used);
  }
}

template void edge_betweenness_centrality<int, int, float, float>(
  const raft::handle_t &,
  GraphCSRView<int, int, float> const *,
  float *,
  bool,
  float const *,
  int,
  int const *,
  int);

template void edge_betweenness_centrality<int, int, double, double>(
  raft::handle_t const &handle,
  GraphCSRView<int, int, double> const *,
  double *,
  bool,
  double const *,
  int,
  int const *,
  int);
}  // namespace cugraph
