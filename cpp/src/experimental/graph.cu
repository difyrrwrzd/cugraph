#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <experimental/graph.hpp>
#include <utilities/error.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <raft/device_atomics.cuh>
#include <raft/handle.hpp>
#include <rmm/device_uvector.hpp>

#include <thrust/adjacent_difference.h>
#include <thrust/binary_search.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <tuple>

namespace cugraph {
namespace experimental {

namespace {

template <bool store_transposed, typename vertex_t, typename edge_t, typename weight_t>
std::
  tuple<rmm::device_uvector<edge_t>, rmm::device_uvector<vertex_t>, rmm::device_uvector<weight_t>>
  edge_list_to_compressed_sparse(raft::handle_t const &handle,
                                 edgelist_t<vertex_t, edge_t, weight_t> const &edgelist,
                                 vertex_t row_first,
                                 vertex_t row_last,
                                 vertex_t col_first,
                                 vertex_t col_last)
{
  rmm::device_uvector<edge_t> offsets(
    store_transposed ? (row_last - row_first) + 1 : (col_last - col_first) + 1,
    handle.get_stream());
  rmm::device_uvector<vertex_t> indices(edgelist.number_of_edges, handle.get_stream());
  rmm::device_uvector<weight_t> weights(
    edgelist.p_edge_weights != nullptr ? edgelist.number_of_edges : 0, handle.get_stream());
  thrust::fill(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
               offsets.begin(),
               offsets.end(),
               edge_t{0});
  thrust::fill(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
               indices.begin(),
               indices.end(),
               vertex_t{0});

  // FIXME: need to performance test this code with R-mat graphs having highly-skewed degree
  // distribution. If there is a small number of vertices with very large degrees, atomicAdd can
  // sequentialize execution. CUDA9+ & Kepler+ provide complier/architectural optimizations to
  // mitigate this impact
  // (https://developer.nvidia.com/blog/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/),
  // and we need to check this thrust::for_each based approach delivers the expected performance.

  // FIXME: also need to verify this approach is at least not significantly slower than the sorting
  // based approach (this approach does not use extra memory, so better stick to this approach
  // unless performance is significantly worse).

  auto p_offsets = offsets.data();
  auto p_indices = indices.data();
  auto p_weights =
    edgelist.p_edge_weights != nullptr ? weights.data() : static_cast<weight_t *>(nullptr);

  auto major_first = store_transposed ? row_first : col_first;
  thrust::for_each(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                   store_transposed ? edgelist.p_src_vertices : edgelist.p_dst_vertices,
                   store_transposed ? edgelist.p_src_vertices + edgelist.number_of_edges
                                    : edgelist.p_dst_vertices + edgelist.number_of_edges,
                   [p_offsets, major_first] __device__(auto v) {
                     atomicAdd(p_offsets + (v - major_first), edge_t{1});
                   });

  thrust::exclusive_scan(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                         offsets.begin(),
                         offsets.end(),
                         offsets.begin());

  if (edgelist.p_edge_weights != nullptr) {
    auto edge_first = thrust::make_zip_iterator(thrust::make_tuple(
      edgelist.p_src_vertices, edgelist.p_dst_vertices, edgelist.p_edge_weights));
    thrust::for_each(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                     edge_first,
                     edge_first + edgelist.number_of_edges,
                     [p_offsets, p_indices, p_weights, major_first] __device__(auto e) {
                       auto s      = thrust::get<0>(e);
                       auto d      = thrust::get<1>(e);
                       auto w      = thrust::get<2>(e);
                       auto major  = store_transposed ? s : d;
                       auto minor  = store_transposed ? d : s;
                       auto start  = p_offsets[major - major_first];
                       auto degree = p_offsets[(major - major_first) + 1] - start;
                       auto idx    = atomicAdd(p_indices + (degree - 1),
                                            vertex_t{1});  // use the last element as a counter
                       // FIXME: we can actually store minor - minor_first instead of minor to save
                       // memory if minor can be larger than 32 bit but minor - minor_first fits
                       // within 32 bit
                       p_indices[start + idx] =
                         minor;  // overwrite the counter only if idx == degree - 1 (no race)
                       p_weights[start + idx] = w;
                     });
  } else {
    auto edge_first = thrust::make_zip_iterator(
      thrust::make_tuple(edgelist.p_src_vertices, edgelist.p_dst_vertices));
    thrust::for_each(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                     edge_first,
                     edge_first + edgelist.number_of_edges,
                     [p_offsets, p_indices, p_weights, major_first] __device__(auto e) {
                       auto s      = thrust::get<0>(e);
                       auto d      = thrust::get<1>(e);
                       auto major  = store_transposed ? s : d;
                       auto minor  = store_transposed ? d : s;
                       auto start  = p_offsets[major - major_first];
                       auto degree = p_offsets[(major - major_first) + 1] - start;
                       auto idx    = atomicAdd(p_indices + (degree - 1),
                                            vertex_t{1});  // use the last element as a counter
                       // FIXME: we can actually store minor - minor_first instead of minor to save
                       // memory if minor can be larger than 32 bit but minor - minor_first fits
                       // within 32 bit
                       p_indices[start + idx] =
                         minor;  // overwrite the counter only if idx == degree - 1 (no race)
                     });
  }

  return std::make_tuple(std::move(offsets), std::move(indices), std::move(weights));
}

// FIXME: better move this elsewhere, this can be reused in graph_device_view.cuh to compute degree
// as well.
// compute the numbers of nonzeros in rows of the (transposed) graph adjacency matrix
template <typename vertex_t, typename edge_t>
rmm::device_uvector<edge_t> compute_row_degree(
  raft::handle_t const &handle,
  std::vector<rmm::device_uvector<edge_t>> const &adj_matrix_partition_offsets,
  partition_t<vertex_t> const &partition)
{
  auto &comm_p_row     = handle.get_subcomm(comm_p_row_key);
  auto comm_p_row_rank = comm_p_row.get_rank();
  auto comm_p_row_size = comm_p_row.get_size();
  auto &comm_p_col     = handle.get_subcomm(comm_p_col_key);
  auto comm_p_col_rank = comm_p_col.get_rank();
  auto comm_p_col_size = comm_p_col.get_size();

  rmm::device_uvector<edge_t> local_degrees(0, handle.get_stream());
  rmm::device_uvector<edge_t> degrees(0, handle.get_stream());

  vertex_t max_num_local_degrees{0};
  for (int i = 0; i < comm_p_col_size; ++i) {
    auto vertex_partition_id = partition.hypergraph_partitioned
                                 ? comm_p_row_size * i + comm_p_row_rank
                                 : comm_p_col_size * comm_p_row_rank + i;
    auto row_first        = partition.vertex_partition_offsets[vertex_partition_id];
    auto row_last         = partition.vertex_partition_offsets[vertex_partition_id + 1];
    max_num_local_degrees = std::max(max_num_local_degrees, row_last - row_first);
    if (i == comm_p_col_rank) { degrees.resize(row_last - row_first, handle.get_stream()); }
  }
  local_degrees.resize(max_num_local_degrees, handle.get_stream());
  for (int i = 0; i < comm_p_col_size; ++i) {
    auto vertex_partition_id = partition.hypergraph_partitioned
                                 ? comm_p_row_size * i + comm_p_row_rank
                                 : comm_p_col_size * comm_p_row_rank + i;
    auto row_first = partition.vertex_partition_offsets[vertex_partition_id];
    auto row_last  = partition.vertex_partition_offsets[vertex_partition_id + 1];
    auto p_offsets =
      partition.hypergraph_partitioned
        ? adj_matrix_partition_offsets[i].data()
        : adj_matrix_partition_offsets[0].data() +
            (row_first - partition.vertex_partition_offsets[comm_p_col_size * comm_p_row_rank]);
    thrust::transform(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                      thrust::make_counting_iterator(vertex_t{0}),
                      thrust::make_counting_iterator(row_last - row_first),
                      local_degrees.data(),
                      [p_offsets] __device__(auto i) { return p_offsets[i + 1] - p_offsets[i]; });
    comm_p_row.reduce(local_degrees.data(),
                      i == comm_p_col_rank ? degrees.data() : static_cast<edge_t *>(nullptr),
                      degrees.size(),
                      raft::comms::op_t::SUM,
                      comm_p_col_rank,
                      handle.get_stream());
  }

  return degrees;
}

template <typename vertex_t, typename DegreeIterator, typename ThresholdIterator>
std::vector<vertex_t> segment_degree_sorted_vertex_partition(raft::handle_t const &handle,
                                                             DegreeIterator degree_first,
                                                             DegreeIterator degree_last,
                                                             ThresholdIterator threshold_first,
                                                             ThresholdIterator threshold_last)
{
  auto num_elements = thrust::distance(degree_first, degree_last);
  auto num_segments = thrust::distance(threshold_first, threshold_last) + 1;

  std::vector<vertex_t> h_segment_offsets(num_segments + 1);
  h_segment_offsets[0]     = 0;
  h_segment_offsets.back() = num_elements;

  rmm::device_uvector<vertex_t> d_segment_offsets(num_segments - 1, handle.get_stream());

  thrust::upper_bound(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                      degree_first,
                      degree_last,
                      threshold_first,
                      threshold_last,
                      d_segment_offsets.begin());

  raft::update_host(h_segment_offsets.begin() + 1,
                    d_segment_offsets.begin(),
                    d_segment_offsets.size(),
                    handle.get_stream());

  return h_segment_offsets;
}

}  // namespace

template <typename vertex_t,
          typename edge_t,
          typename weight_t,
          bool store_transposed,
          bool multi_gpu>
graph_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu, std::enable_if_t<multi_gpu>>::
  graph_t(raft::handle_t const &handle,
          std::vector<edgelist_t<vertex_t, edge_t, weight_t>> const &edgelists,
          partition_t<vertex_t> const &partition,
          vertex_t number_of_vertices,
          edge_t number_of_edges,
          bool is_symmetric,
          bool is_multigraph,
          bool is_weighted,
          bool sorted_by_global_degree_within_vertex_partition,
          bool do_expensive_check)
  : graph_base_t<vertex_t, edge_t, weight_t>(
      handle, number_of_vertices, number_of_edges, is_symmetric, is_multigraph, is_weighted),
    partition_(partition)
{
  auto &comm_p_row     = this->get_handle_ptr()->get_subcomm(comm_p_row_key);
  auto comm_p_row_rank = comm_p_row.get_rank();
  auto comm_p_row_size = comm_p_row.get_size();
  auto &comm_p_col     = this->get_handle_ptr()->get_subcomm(comm_p_col_key);
  auto comm_p_col_rank = comm_p_col.get_rank();
  auto comm_p_col_size = comm_p_col.get_size();
  auto default_stream  = this->get_handle_ptr()->get_stream();

  // convert edge list (COO) to compressed sparse format (CSR or CSC)

  adj_matrix_partition_offsets_.reserve(edgelists.size());
  adj_matrix_partition_indices_.reserve(edgelists.size());
  adj_matrix_partition_weights_.reserve(edgelists.size());
  for (size_t i = 0; i < edgelists.size(); ++i) {
    CUGRAPH_EXPECTS((is_weighted == false) || (edgelists[i].p_edge_weights != nullptr),
                    "Invalid API parameter, edgelists[i].p_edge_weights shoud not be nullptr if "
                    "is_weighted == true");

    auto row_first = partition_.hypergraph_partitioned
                       ? partition_.vertex_partition_offsets[comm_p_row_size * i + comm_p_row_rank]
                       : partition_.vertex_partition_offsets[comm_p_col_size * comm_p_row_rank];
    auto row_last =
      partition_.hypergraph_partitioned
        ? partition_.vertex_partition_offsets[comm_p_row_size * i + comm_p_row_rank + 1]
        : partition_.vertex_partition_offsets[comm_p_col_size * (comm_p_row_rank + 1)];
    auto col_first = partition_.vertex_partition_offsets[comm_p_row_size * comm_p_col_rank];
    auto col_last  = partition_.vertex_partition_offsets[comm_p_row_size * (comm_p_col_rank + 1)];

    rmm::device_uvector<edge_t> offsets(0, this->get_handle_ptr()->get_stream());
    rmm::device_uvector<vertex_t> indices(0, this->get_handle_ptr()->get_stream());
    rmm::device_uvector<weight_t> weights(0, this->get_handle_ptr()->get_stream());
    std::tie(offsets, indices, weights) = edge_list_to_compressed_sparse<store_transposed>(
      *(this->get_handle_ptr()), edgelists[i], row_first, row_last, col_first, col_last);
    adj_matrix_partition_offsets_.push_back(std::move(offsets));
    adj_matrix_partition_indices_.push_back(std::move(indices));
    adj_matrix_partition_weights_.push_back(std::move(weights));
  }

  // update degree-based segment offsets (to be used for graph analytics kernel optimization)

  auto degrees =
    compute_row_degree(*(this->get_handle_ptr()), adj_matrix_partition_offsets_, partition_);

  static_assert(num_segments_per_vertex_partition == 3);
  static_assert((low_degree_threshold <= mid_degree_threshold) &&
                (mid_degree_threshold <= std::numeric_limits<edge_t>::max()));
  rmm::device_uvector<edge_t> d_thresholds(num_segments_per_vertex_partition - 1, default_stream);
  std::vector<edge_t> h_thresholds = {static_cast<edge_t>(low_degree_threshold),
                                      static_cast<edge_t>(mid_degree_threshold)};
  raft::update_device(
    d_thresholds.data(), h_thresholds.data(), h_thresholds.size(), default_stream);

  rmm::device_uvector<vertex_t> segment_offsets(num_segments_per_vertex_partition + 1,
                                                default_stream);
  segment_offsets.set_element_async(0, 0, default_stream);
  segment_offsets.set_element_async(
    num_segments_per_vertex_partition, degrees.size(), default_stream);

  thrust::upper_bound(rmm::exec_policy(default_stream)->on(default_stream),
                      degrees.begin(),
                      degrees.end(),
                      d_thresholds.begin(),
                      d_thresholds.end(),
                      segment_offsets.begin() + 1);

  rmm::device_uvector<vertex_t> aggregate_segment_offsets(comm_p_row_size * segment_offsets.size(),
                                                          default_stream);
  comm_p_row.allgather(segment_offsets.data(),
                       aggregate_segment_offsets.data(),
                       segment_offsets.size(),
                       default_stream);

  vertex_partition_segment_offsets_.resize(comm_p_row_size * (segment_offsets.size()));
  raft::update_host(vertex_partition_segment_offsets_.data(),
                    aggregate_segment_offsets.data(),
                    aggregate_segment_offsets.size(),
                    default_stream);
}

template <typename vertex_t,
          typename edge_t,
          typename weight_t,
          bool store_transposed,
          bool multi_gpu>
graph_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu, std::enable_if_t<!multi_gpu>>::
  graph_t(raft::handle_t const &handle,
          edgelist_t<vertex_t, edge_t, weight_t> const &edgelist,
          vertex_t number_of_vertices,
          edge_t number_of_edges,
          bool is_symmetric,
          bool is_multigraph,
          bool is_weighted,
          bool sorted_by_global_degree,
          bool do_expensive_check)
  : graph_base_t<vertex_t, edge_t, weight_t>(
      handle, number_of_vertices, number_of_edges, is_symmetric, is_multigraph, is_weighted),
    offsets_(rmm::device_uvector<edge_t>(0, handle.get_stream())),
    indices_(rmm::device_uvector<vertex_t>(0, handle.get_stream())),
    weights_(rmm::device_uvector<weight_t>(0, handle.get_stream()))
{
  auto default_stream = this->get_handle_ptr()->get_stream();

  // convert edge list (COO) to compressed sparse format (CSR or CSC)

  CUGRAPH_EXPECTS(
    (is_weighted == false) || (edgelist.p_edge_weights != nullptr),
    "Invalid API parameter, edgelist.p_edge_weights shoud not be nullptr if is_weighted == true");

  std::tie(offsets_, indices_, weights_) =
    edge_list_to_compressed_sparse<store_transposed>(*(this->get_handle_ptr()),
                                                     edgelist,
                                                     vertex_t{0},
                                                     this->get_number_of_vertices(),
                                                     vertex_t{0},
                                                     this->get_number_of_vertices());

  // update degree-based segment offsets (to be used for graph analytics kernel optimization)

  rmm::device_uvector<edge_t> degrees(this->get_number_of_vertices(), default_stream);
  thrust::adjacent_difference(rmm::exec_policy(default_stream)->on(default_stream),
                              offsets_.begin() + 1,
                              offsets_.end(),
                              degrees.begin());

  static_assert(num_segments_per_vertex_partition == 3);
  static_assert((low_degree_threshold <= mid_degree_threshold) &&
                (mid_degree_threshold <= std::numeric_limits<edge_t>::max()));
  rmm::device_uvector<edge_t> d_thresholds(num_segments_per_vertex_partition - 1, default_stream);
  std::vector<edge_t> h_thresholds = {static_cast<edge_t>(low_degree_threshold),
                                      static_cast<edge_t>(mid_degree_threshold)};
  raft::update_device(
    d_thresholds.data(), h_thresholds.data(), h_thresholds.size(), default_stream);

  rmm::device_uvector<vertex_t> segment_offsets(num_segments_per_vertex_partition + 1,
                                                default_stream);
  segment_offsets.set_element_async(0, 0, default_stream);
  segment_offsets.set_element_async(
    num_segments_per_vertex_partition, degrees.size(), default_stream);

  thrust::upper_bound(rmm::exec_policy(default_stream)->on(default_stream),
                      degrees.begin(),
                      degrees.end(),
                      d_thresholds.begin(),
                      d_thresholds.end(),
                      segment_offsets.begin() + 1);

  raft::update_host(
    segment_offsets_.data(), segment_offsets.data(), segment_offsets.size(), default_stream);
}

// explicit instantiation

template class graph_t<int32_t, int32_t, float, true, true>;
template class graph_t<int32_t, int32_t, float, false, true>;

template class graph_t<int32_t, int32_t, float, true, false>;
template class graph_t<int32_t, int32_t, float, false, false>;

}  // namespace experimental
}  // namespace cugraph