/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <experimental/detail/graph_utils.cuh>
#include <experimental/graph.hpp>
#include <experimental/graph_view.hpp>
#include <patterns/copy_to_adj_matrix_row_col.cuh>
#include <utilities/comm_utils.cuh>
#include <utilities/error.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <raft/handle.hpp>
#include <rmm/device_uvector.hpp>

#include <thrust/copy.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/sort.h>
#include <thrust/tuple.h>
#include <cuco/detail/hash_functions.cuh>
#include <cuco/static_map.cuh>

#include <algorithm>
#include <iterator>
#include <numeric>
#include <tuple>
#include <utility>

namespace cugraph {
namespace experimental {

namespace {

template <typename vertex_t, typename edge_t, typename weight_t>
std::
  tuple<rmm::device_uvector<vertex_t>, rmm::device_uvector<vertex_t>, rmm::device_uvector<weight_t>>
  compressed_sparse_to_edgelist(edge_t const *compressed_sparse_offsets,
                                vertex_t const *compressed_sparse_indices,
                                weight_t const *compressed_sparse_weights,
                                vertex_t major_first,
                                vertex_t major_last,
                                hipStream_t stream)
{
  edge_t number_of_edges{0};
  raft::update_host(
    &number_of_edges, compressed_sparse_offsets + (major_last - major_first), 1, stream);
  CUDA_TRY(hipStreamSynchronize(stream));
  rmm::device_uvector<vertex_t> edgelist_major_vertices(number_of_edges, stream);
  rmm::device_uvector<vertex_t> edgelist_minor_vertices(number_of_edges, stream);
  rmm::device_uvector<weight_t> edgelist_weights(
    compressed_sparse_weights != nullptr ? number_of_edges : 0, stream);

  // FIXME: this is highly inefficient for very high-degree vertices, for better performance, we can
  // fill high-degree vertices using one CUDA block per vertex, mid-degree vertices using one CUDA
  // warp per vertex, and low-degree vertices using one CUDA thread per block
  thrust::for_each(rmm::exec_policy(stream)->on(stream),
                   thrust::make_counting_iterator(major_first),
                   thrust::make_counting_iterator(major_last),
                   [compressed_sparse_offsets,
                    major_first,
                    p_majors = edgelist_major_vertices.begin()] __device__(auto v) {
                     auto first = compressed_sparse_offsets[v - major_first];
                     auto last  = compressed_sparse_offsets[v - major_first + 1];
                     thrust::fill(thrust::seq, p_majors + first, p_majors + last, v);
                   });
  thrust::copy(rmm::exec_policy(stream)->on(stream),
               compressed_sparse_indices,
               compressed_sparse_indices + number_of_edges,
               edgelist_minor_vertices.begin());
  if (compressed_sparse_weights != nullptr) {
    thrust::copy(rmm::exec_policy(stream)->on(stream),
                 compressed_sparse_weights,
                 compressed_sparse_weights + number_of_edges,
                 edgelist_weights.data());
  }

  return std::make_tuple(std::move(edgelist_major_vertices),
                         std::move(edgelist_minor_vertices),
                         std::move(edgelist_weights));
}

template <typename vertex_t, typename weight_t>
void sort_and_coarsen_edgelist(rmm::device_uvector<vertex_t> &edgelist_major_vertices /* [INOUT] */,
                               rmm::device_uvector<vertex_t> &edgelist_minor_vertices /* [INOUT] */,
                               rmm::device_uvector<weight_t> &edgelist_weights /* [INOUT] */,
                               hipStream_t stream)
{
  auto pair_first = thrust::make_zip_iterator(
    thrust::make_tuple(edgelist_major_vertices.begin(), edgelist_minor_vertices.begin()));

  size_t number_of_edges{0};
  if (edgelist_weights.size() > 0) {
    thrust::sort_by_key(rmm::exec_policy(stream)->on(stream),
                        pair_first,
                        pair_first + edgelist_major_vertices.size(),
                        edgelist_weights.begin());

    rmm::device_uvector<vertex_t> tmp_edgelist_major_vertices(edgelist_major_vertices.size(),
                                                              stream);
    rmm::device_uvector<vertex_t> tmp_edgelist_minor_vertices(tmp_edgelist_major_vertices.size(),
                                                              stream);
    rmm::device_uvector<weight_t> tmp_edgelist_weights(tmp_edgelist_major_vertices.size(), stream);
    auto it = thrust::reduce_by_key(
      rmm::exec_policy(stream)->on(stream),
      pair_first,
      pair_first + edgelist_major_vertices.size(),
      edgelist_weights.begin(),
      thrust::make_zip_iterator(thrust::make_tuple(tmp_edgelist_major_vertices.begin(),
                                                   tmp_edgelist_minor_vertices.begin())),
      tmp_edgelist_weights.begin());
    number_of_edges = thrust::distance(tmp_edgelist_weights.begin(), thrust::get<1>(it));

    CUDA_TRY(hipStreamSynchronize(
      stream));  // memory blocks owned by edgelist_(major_vertices,minor_vertices,weights) will be
                 // freed after the assignments below

    edgelist_major_vertices = std::move(tmp_edgelist_major_vertices);
    edgelist_minor_vertices = std::move(tmp_edgelist_minor_vertices);
    edgelist_weights        = std::move(tmp_edgelist_weights);
  } else {
    thrust::sort(rmm::exec_policy(stream)->on(stream),
                 pair_first,
                 pair_first + edgelist_major_vertices.size());
    auto it         = thrust::unique(rmm::exec_policy(stream)->on(stream),
                             pair_first,
                             pair_first + edgelist_major_vertices.size());
    number_of_edges = thrust::distance(pair_first, it);
  }

  edgelist_major_vertices.resize(number_of_edges, stream);
  edgelist_minor_vertices.resize(number_of_edges, stream);
  edgelist_weights.resize(number_of_edges, stream);
  edgelist_major_vertices.shrink_to_fit(stream);
  edgelist_minor_vertices.shrink_to_fit(stream);
  edgelist_weights.shrink_to_fit(stream);

  return;
}

template <typename vertex_t, typename edge_t, typename weight_t>
std::
  tuple<rmm::device_uvector<vertex_t>, rmm::device_uvector<vertex_t>, rmm::device_uvector<weight_t>>
  compressed_sparse_to_relabeled_and_sorted_and_coarsened_edgelist(
    edge_t const *compressed_sparse_offsets,
    vertex_t const *compressed_sparse_indices,
    weight_t const *compressed_sparse_weights,
    vertex_t const *p_major_labels,
    vertex_t const *p_minor_labels,
    vertex_t major_first,
    vertex_t major_last,
    vertex_t minor_first,
    vertex_t minor_last,
    hipStream_t stream)
{
  // FIXME: it might be possible to directly create relabled & coarsened edgelist from the
  // compressed sparse format to save memory

  rmm::device_uvector<vertex_t> edgelist_major_vertices(0, stream);
  rmm::device_uvector<vertex_t> edgelist_minor_vertices(0, stream);
  rmm::device_uvector<weight_t> edgelist_weights(0, stream);
  std::tie(edgelist_major_vertices, edgelist_minor_vertices, edgelist_weights) =
    compressed_sparse_to_edgelist(compressed_sparse_offsets,
                                  compressed_sparse_indices,
                                  compressed_sparse_weights,
                                  major_first,
                                  major_last,
                                  stream);

  auto pair_first = thrust::make_zip_iterator(
    thrust::make_tuple(edgelist_major_vertices.begin(), edgelist_minor_vertices.begin()));
  thrust::transform(
    rmm::exec_policy(stream)->on(stream),
    pair_first,
    pair_first + edgelist_major_vertices.size(),
    pair_first,
    [p_major_labels, p_minor_labels, major_first, minor_first] __device__(auto val) {
      return thrust::make_tuple(p_major_labels[thrust::get<0>(val) - major_first],
                                p_minor_labels[thrust::get<1>(val) - minor_first]);
    });

  sort_and_coarsen_edgelist(
    edgelist_major_vertices, edgelist_minor_vertices, edgelist_weights, stream);

  return std::make_tuple(std::move(edgelist_major_vertices),
                         std::move(edgelist_minor_vertices),
                         std::move(edgelist_weights));
}

template <typename vertex_t, typename edge_t, bool multi_gpu>
rmm::device_uvector<vertex_t> compute_renumber_map(
  raft::handle_t const &handle,
  rmm::device_uvector<vertex_t> const &edgelist_major_vertices,
  rmm::device_uvector<vertex_t> const &edgelist_minor_vertices)
{
  // FIXME: compare this sort based approach with hash based approach in both speed and memory
  // footprint

  // 1. acquire (unique major label, count) pairs

  rmm::device_uvector<vertex_t> tmp_labels(edgelist_major_vertices, handle.get_stream());
  thrust::sort(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
               tmp_labels.begin(),
               tmp_labels.end());
  rmm::device_uvector<vertex_t> major_labels(tmp_labels.size(), handle.get_stream());
  rmm::device_uvector<edge_t> major_counts(major_labels.size(), handle.get_stream());
  auto major_pair_it =
    thrust::reduce_by_key(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                          tmp_labels.begin(),
                          tmp_labels.end(),
                          thrust::make_constant_iterator(edge_t{1}),
                          major_labels.begin(),
                          major_counts.begin());
  tmp_labels.resize(0, handle.get_stream());
  tmp_labels.shrink_to_fit(handle.get_stream());
  major_labels.resize(thrust::distance(major_labels.begin(), thrust::get<0>(major_pair_it)),
                      handle.get_stream());
  major_counts.resize(major_labels.size(), handle.get_stream());
  major_labels.shrink_to_fit(handle.get_stream());
  major_counts.shrink_to_fit(handle.get_stream());

  // 2. acquire unique minor labels

  rmm::device_uvector<vertex_t> minor_labels(edgelist_minor_vertices, handle.get_stream());
  thrust::sort(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
               minor_labels.begin(),
               minor_labels.end());
  auto minor_label_it =
    thrust::unique(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                   minor_labels.begin(),
                   minor_labels.end());
  minor_labels.resize(thrust::distance(minor_labels.begin(), minor_label_it), handle.get_stream());
  minor_labels.shrink_to_fit(handle.get_stream());

  // 3. merge major and minor labels

  rmm::device_uvector<vertex_t> merged_labels(major_labels.size() + minor_labels.size(),
                                              handle.get_stream());
  rmm::device_uvector<edge_t> merged_counts(merged_labels.size(), handle.get_stream());
  thrust::merge_by_key(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                       major_labels.begin(),
                       major_labels.end(),
                       minor_labels.begin(),
                       minor_labels.end(),
                       major_counts.begin(),
                       thrust::make_constant_iterator(edge_t{0}),
                       merged_labels.begin(),
                       merged_counts.begin());
  major_labels.resize(0, handle.get_stream());
  major_counts.resize(0, handle.get_stream());
  minor_labels.resize(0, handle.get_stream());
  major_labels.shrink_to_fit(handle.get_stream());
  major_counts.shrink_to_fit(handle.get_stream());
  minor_labels.shrink_to_fit(handle.get_stream());
  rmm::device_uvector<vertex_t> labels(merged_labels.size(), handle.get_stream());
  rmm::device_uvector<edge_t> counts(labels.size(), handle.get_stream());
  auto pair_it =
    thrust::reduce_by_key(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                          merged_labels.begin(),
                          merged_labels.end(),
                          merged_counts.begin(),
                          labels.begin(),
                          counts.begin());
  merged_labels.resize(0, handle.get_stream());
  merged_counts.resize(0, handle.get_stream());
  merged_labels.shrink_to_fit(handle.get_stream());
  merged_counts.shrink_to_fit(handle.get_stream());
  labels.resize(thrust::distance(labels.begin(), thrust::get<0>(pair_it)), handle.get_stream());
  counts.resize(labels.size(), handle.get_stream());
  labels.shrink_to_fit(handle.get_stream());
  counts.shrink_to_fit(handle.get_stream());

  // 4. if multi-GPU, shuffle and reduce (label, count) pairs

  if (multi_gpu) {
    auto &comm           = handle.get_comms();
    auto const comm_size = comm.get_size();

    auto pair_first = thrust::make_zip_iterator(thrust::make_tuple(labels.begin(), counts.begin()));
    auto key_func   = detail::compute_gpu_id_from_vertex_t<vertex_t>{comm_size};
    thrust::sort(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                 pair_first,
                 pair_first + labels.size(),
                 [key_func] __device__(auto lhs, auto rhs) {
                   return key_func(thrust::get<0>(lhs)) < key_func(thrust::get<0>(rhs));
                 });
    auto key_first = thrust::make_transform_iterator(
      labels.begin(), [key_func] __device__(auto val) { return key_func(val); });
    rmm::device_uvector<size_t> tx_value_counts(comm_size, handle.get_stream());
    thrust::reduce_by_key(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                          key_first,
                          key_first + labels.size(),
                          thrust::make_constant_iterator(size_t{1}),
                          thrust::make_discard_iterator(),
                          tx_value_counts.begin());

    rmm::device_uvector<vertex_t> rx_labels(0, handle.get_stream());
    rmm::device_uvector<edge_t> rx_counts(0, handle.get_stream());

    std::tie(rx_labels, rx_counts, std::ignore) = cugraph::experimental::detail::shuffle_values(
      handle.get_comms(), pair_first, tx_value_counts, handle.get_stream());

    labels.resize(rx_labels.size(), handle.get_stream());
    counts.resize(labels.size(), handle.get_stream());
    thrust::sort_by_key(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                        rx_labels.begin(),
                        rx_labels.end(),
                        rx_counts.begin());
    pair_it = thrust::reduce_by_key(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                                    rx_labels.begin(),
                                    rx_labels.end(),
                                    rx_counts.begin(),
                                    labels.begin(),
                                    counts.begin());
    rx_labels.resize(0, handle.get_stream());
    rx_counts.resize(0, handle.get_stream());
    rx_labels.shrink_to_fit(handle.get_stream());
    rx_counts.shrink_to_fit(handle.get_stream());
    labels.resize(thrust::distance(labels.begin(), thrust::get<0>(pair_it)), handle.get_stream());
    counts.resize(labels.size(), handle.get_stream());
    labels.shrink_to_fit(handle.get_stream());
    labels.shrink_to_fit(handle.get_stream());
  }

  // 5. sort by degree

  thrust::sort_by_key(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                      counts.begin(),
                      counts.end(),
                      labels.begin(),
                      thrust::greater<edge_t>());

  CUDA_TRY(
    hipStreamSynchronize(handle.get_stream()));  // temporary rmm::devicec_uvector objects become
                                                  // out-of-scope once this function returns.

  return std::move(labels);
}

}  // namespace

template <typename vertex_t, typename edge_t, bool multi_gpu>
std::enable_if_t<multi_gpu,
                 std::tuple<rmm::device_uvector<vertex_t>, partition_t<vertex_t>, vertex_t, edge_t>>
renumber_edgelist(raft::handle_t const &handle,
                  rmm::device_uvector<vertex_t> &edgelist_major_vertices /* [INOUT] */,
                  rmm::device_uvector<vertex_t> &edgelist_minor_vertices /* [INOUT] */,
                  bool is_hypergraph_partitioned)
{
  auto &comm               = handle.get_comms();
  auto const comm_size     = comm.get_size();
  auto const comm_rank     = comm.get_rank();
  auto &row_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().row_name());
  auto const row_comm_size = row_comm.get_size();
  auto const row_comm_rank = row_comm.get_rank();
  auto &col_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().col_name());
  auto const col_comm_size = col_comm.get_size();
  auto const col_comm_rank = col_comm.get_rank();

  // 1. compute renumber map

  auto renumber_map_labels = compute_renumber_map<vertex_t, edge_t, multi_gpu>(
    handle, edgelist_major_vertices, edgelist_minor_vertices);

  // 2. initialize partition_t object, number_of_vertices, and number_of_edges for the coarsened
  // graph

  auto vertex_partition_counts = host_scalar_allgather(
    comm, static_cast<vertex_t>(renumber_map_labels.size()), handle.get_stream());
  std::vector<vertex_t> vertex_partition_offsets(comm_size + 1, 0);
  std::partial_sum(vertex_partition_counts.begin(),
                   vertex_partition_counts.end(),
                   vertex_partition_offsets.begin() + 1);

  partition_t<vertex_t> partition(vertex_partition_offsets,
                                  is_hypergraph_partitioned,
                                  row_comm_size,
                                  col_comm_size,
                                  row_comm_rank,
                                  col_comm_rank);

  auto number_of_vertices = vertex_partition_offsets.back();
  auto number_of_edges    = host_scalar_allreduce(
    comm, static_cast<edge_t>(edgelist_major_vertices.size()), handle.get_stream());

  // 3. renumber edges

  if (is_hypergraph_partitioned) {
    CUGRAPH_FAIL("unimplemented.");
  } else {
    double constexpr load_factor = 0.7;

    // FIXME: compare this hash based approach with a binary search based approach in both memory
    // footprint and execution time

    {
      vertex_t major_first{};
      vertex_t major_last{};
      std::tie(major_first, major_last) = partition.get_matrix_partition_major_range(0);
      rmm::device_uvector<vertex_t> renumber_map_major_labels(major_last - major_first,
                                                              handle.get_stream());
      std::vector<size_t> recvcounts(row_comm_size);
      for (int i = 0; i < row_comm_size; ++i) {
        recvcounts[i] = partition.get_vertex_partition_size(row_comm_rank * row_comm_size + i);
      }
      std::vector<size_t> displacements(row_comm_size, 0);
      std::partial_sum(recvcounts.begin(), recvcounts.end() - 1, displacements.begin() + 1);
      device_allgatherv(row_comm,
                        renumber_map_labels.begin(),
                        renumber_map_major_labels.begin(),
                        recvcounts,
                        displacements,
                        handle.get_stream());

      CUDA_TRY(hipStreamSynchronize(
        handle.get_stream()));  // cuco::static_map currently does not take stream

      cuco::static_map<vertex_t, vertex_t> renumber_map{
        static_cast<size_t>(static_cast<double>(renumber_map_major_labels.size()) / load_factor),
        invalid_vertex_id<vertex_t>::value,
        invalid_vertex_id<vertex_t>::value};
      auto pair_first = thrust::make_transform_iterator(
        thrust::make_zip_iterator(thrust::make_tuple(renumber_map_major_labels.begin(),
                                                     thrust::make_counting_iterator(major_first))),
        [] __device__(auto val) {
          return thrust::make_pair(thrust::get<0>(val), thrust::get<1>(val));
        });
      renumber_map.insert(pair_first, pair_first + renumber_map_major_labels.size());
      renumber_map.find(edgelist_major_vertices.begin(),
                        edgelist_major_vertices.end(),
                        edgelist_major_vertices.begin());
    }

    {
      vertex_t minor_first{};
      vertex_t minor_last{};
      std::tie(minor_first, minor_last) = partition.get_matrix_partition_minor_range();
      rmm::device_uvector<vertex_t> renumber_map_minor_labels(minor_last - minor_first,
                                                              handle.get_stream());

      // FIXME: this P2P is unnecessary if we apply the partitioning scheme used with hypergraph
      // partitioning
      auto comm_src_rank = row_comm_rank * col_comm_size + col_comm_rank;
      auto comm_dst_rank = (comm_rank % col_comm_size) * row_comm_size + comm_rank / col_comm_size;
      // FIXME: this branch may be no longer necessary with NCCL backend
      if (comm_src_rank == comm_rank) {
        assert(comm_dst_rank == comm_rank);
        thrust::copy(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                     renumber_map_labels.begin(),
                     renumber_map_labels.end(),
                     renumber_map_minor_labels.begin() +
                       (partition.get_vertex_partition_first(comm_src_rank) -
                        partition.get_vertex_partition_first(row_comm_rank * col_comm_size)));
      } else {
        device_sendrecv(comm,
                        renumber_map_labels.begin(),
                        renumber_map_labels.size(),
                        comm_dst_rank,
                        renumber_map_minor_labels.begin() +
                          (partition.get_vertex_partition_first(comm_src_rank) -
                           partition.get_vertex_partition_first(row_comm_rank * col_comm_size)),
                        static_cast<size_t>(partition.get_vertex_partition_size(comm_src_rank)),
                        comm_src_rank,
                        handle.get_stream());
      }

      // FIXME: these broadcast operations can be placed between ncclGroupStart() and
      // ncclGroupEnd()
      for (int i = 0; i < col_comm_size; ++i) {
        auto offset = partition.get_vertex_partition_first(row_comm_rank * col_comm_size + i) -
                      partition.get_vertex_partition_first(row_comm_rank * col_comm_size);
        auto count = partition.get_vertex_partition_size(row_comm_rank * col_comm_size + i);
        device_bcast(col_comm,
                     renumber_map_minor_labels.begin() + offset,
                     renumber_map_minor_labels.begin() + offset,
                     count,
                     i,
                     handle.get_stream());
      }

      CUDA_TRY(hipStreamSynchronize(
        handle.get_stream()));  // cuco::static_map currently does not take stream

      cuco::static_map<vertex_t, vertex_t> renumber_map{
        static_cast<size_t>(static_cast<double>(renumber_map_minor_labels.size()) / load_factor),
        invalid_vertex_id<vertex_t>::value,
        invalid_vertex_id<vertex_t>::value};
      auto pair_first = thrust::make_transform_iterator(
        thrust::make_zip_iterator(thrust::make_tuple(renumber_map_minor_labels.begin(),
                                                     thrust::make_counting_iterator(minor_first))),
        [] __device__(auto val) {
          return thrust::make_pair(thrust::get<0>(val), thrust::get<1>(val));
        });
      renumber_map.insert(pair_first, pair_first + renumber_map_minor_labels.size());
      renumber_map.find(edgelist_minor_vertices.begin(),
                        edgelist_minor_vertices.end(),
                        edgelist_minor_vertices.begin());
    }
  }

  return std::make_tuple(
    std::move(renumber_map_labels), partition, number_of_vertices, number_of_edges);
}

template <typename vertex_t, typename edge_t, bool multi_gpu>
std::enable_if_t<!multi_gpu, rmm::device_uvector<vertex_t>> renumber_edgelist(
  raft::handle_t const &handle,
  rmm::device_uvector<vertex_t> &edgelist_major_vertices /* [INOUT] */,
  rmm::device_uvector<vertex_t> &edgelist_minor_vertices /* [INOUT] */)
{
  auto renumber_map_labels = compute_renumber_map<vertex_t, edge_t, multi_gpu>(
    handle, edgelist_major_vertices, edgelist_minor_vertices);

  double constexpr load_factor = 0.7;

  // FIXME: compare this hash based approach with a binary search based approach in both memory
  // footprint and execution time

  cuco::static_map<vertex_t, vertex_t> renumber_map{
    static_cast<size_t>(static_cast<double>(renumber_map_labels.size()) / load_factor),
    invalid_vertex_id<vertex_t>::value,
    invalid_vertex_id<vertex_t>::value};
  auto pair_first = thrust::make_transform_iterator(
    thrust::make_zip_iterator(
      thrust::make_tuple(renumber_map_labels.begin(), thrust::make_counting_iterator(vertex_t{0}))),
    [] __device__(auto val) {
      return thrust::make_pair(thrust::get<0>(val), thrust::get<1>(val));
    });
  renumber_map.insert(pair_first, pair_first + renumber_map_labels.size());
  renumber_map.find(edgelist_major_vertices.begin(),
                    edgelist_major_vertices.end(),
                    edgelist_major_vertices.begin());
  renumber_map.find(edgelist_minor_vertices.begin(),
                    edgelist_minor_vertices.end(),
                    edgelist_minor_vertices.begin());

  return std::move(renumber_map_labels);
}

template <typename vertex_t,
          typename edge_t,
          typename weight_t,
          bool store_transposed,
          bool multi_gpu>
std::enable_if_t<
  multi_gpu,
  std::tuple<std::unique_ptr<graph_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>>,
             rmm::device_uvector<vertex_t>>>
coarsen_graph(
  raft::handle_t const &handle,
  graph_view_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu> const &graph_view,
  vertex_t const *labels)
{
  auto &comm               = handle.get_comms();
  auto const comm_size     = comm.get_size();
  auto const comm_rank     = comm.get_rank();
  auto &row_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().row_name());
  auto const row_comm_size = row_comm.get_size();
  auto const row_comm_rank = row_comm.get_rank();
  auto &col_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().col_name());
  auto const col_comm_size = col_comm.get_size();
  auto const col_comm_rank = col_comm.get_rank();

  // 1. locally construct coarsened edge list

  // FIXME: we don't need adj_matrix_major_labels if we apply the same partitioning scheme
  // regardless of hypergraph partitioning is applied or not
  rmm::device_uvector<vertex_t> adj_matrix_major_labels(
    store_transposed ? graph_view.get_number_of_local_adj_matrix_partition_cols()
                     : graph_view.get_number_of_local_adj_matrix_partition_rows(),
    handle.get_stream());
  rmm::device_uvector<vertex_t> adj_matrix_minor_labels(
    store_transposed ? graph_view.get_number_of_local_adj_matrix_partition_rows()
                     : graph_view.get_number_of_local_adj_matrix_partition_cols(),
    handle.get_stream());
  if (store_transposed) {
    copy_to_adj_matrix_col(handle, graph_view, labels, adj_matrix_major_labels.data());
    copy_to_adj_matrix_row(handle, graph_view, labels, adj_matrix_minor_labels.data());
  } else {
    copy_to_adj_matrix_row(handle, graph_view, labels, adj_matrix_major_labels.data());
    copy_to_adj_matrix_col(handle, graph_view, labels, adj_matrix_minor_labels.data());
  }

  rmm::device_uvector<vertex_t> coarsened_edgelist_major_vertices(0, handle.get_stream());
  rmm::device_uvector<vertex_t> coarsened_edgelist_minor_vertices(0, handle.get_stream());
  rmm::device_uvector<weight_t> coarsened_edgelist_weights(0, handle.get_stream());
  // FIXME: we may compare performance/memory footprint with the hash_based approach especially when
  // cuco::dynamic_map becomes available (so we don't need to preallocate memory assuming the worst
  // case). We may be able to limit the memory requirement close to the final coarsened edgelist
  // with the hash based approach.
  for (size_t i = 0; i < graph_view.get_number_of_local_adj_matrix_partitions(); ++i) {
    // get edge list

    rmm::device_uvector<vertex_t> edgelist_major_vertices(0, handle.get_stream());
    rmm::device_uvector<vertex_t> edgelist_minor_vertices(0, handle.get_stream());
    rmm::device_uvector<weight_t> edgelist_weights(0, handle.get_stream());
    std::tie(edgelist_major_vertices, edgelist_minor_vertices, edgelist_weights) =
      compressed_sparse_to_relabeled_and_sorted_and_coarsened_edgelist(
        graph_view.offsets(i),
        graph_view.indices(i),
        graph_view.weights(i),
        adj_matrix_major_labels.begin() +
          (store_transposed ? graph_view.get_local_adj_matrix_partition_col_value_start_offset(i)
                            : graph_view.get_local_adj_matrix_partition_row_value_start_offset(i)),
        adj_matrix_minor_labels.begin(),
        store_transposed ? graph_view.get_local_adj_matrix_partition_col_first(i)
                         : graph_view.get_local_adj_matrix_partition_row_first(i),
        store_transposed ? graph_view.get_local_adj_matrix_partition_col_last(i)
                         : graph_view.get_local_adj_matrix_partition_row_last(i),
        store_transposed ? graph_view.get_local_adj_matrix_partition_row_first(i)
                         : graph_view.get_local_adj_matrix_partition_col_first(i),
        store_transposed ? graph_view.get_local_adj_matrix_partition_row_last(i)
                         : graph_view.get_local_adj_matrix_partition_col_last(i),
        handle.get_stream());

    auto cur_size = coarsened_edgelist_major_vertices.size();
    // FIXME: this can lead to frequent costly reallocation; we may be able to avoid this if we can
    // reserve address space to avoid expensive reallocation.
    // https://devblogs.nvidia.com/introducing-low-level-gpu-virtual-memory-management
    coarsened_edgelist_major_vertices.resize(cur_size + edgelist_major_vertices.size(),
                                             handle.get_stream());
    coarsened_edgelist_minor_vertices.resize(coarsened_edgelist_major_vertices.size(),
                                             handle.get_stream());
    coarsened_edgelist_weights.resize(
      graph_view.is_weighted() ? coarsened_edgelist_major_vertices.size() : 0, handle.get_stream());

    if (graph_view.is_weighted()) {
      auto src_edge_first =
        thrust::make_zip_iterator(thrust::make_tuple(edgelist_major_vertices.begin(),
                                                     edgelist_minor_vertices.begin(),
                                                     edgelist_weights.begin()));
      auto dst_edge_first =
        thrust::make_zip_iterator(thrust::make_tuple(coarsened_edgelist_major_vertices.begin(),
                                                     coarsened_edgelist_minor_vertices.begin(),
                                                     coarsened_edgelist_weights.begin())) +
        cur_size;
      thrust::copy(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                   src_edge_first,
                   src_edge_first + edgelist_major_vertices.size(),
                   dst_edge_first);
    } else {
      auto src_edge_first = thrust::make_zip_iterator(
        thrust::make_tuple(edgelist_major_vertices.begin(), edgelist_minor_vertices.begin()));
      auto dst_edge_first =
        thrust::make_zip_iterator(thrust::make_tuple(coarsened_edgelist_major_vertices.begin(),
                                                     coarsened_edgelist_minor_vertices.begin())) +
        cur_size;
      thrust::copy(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                   src_edge_first,
                   src_edge_first + edgelist_major_vertices.size(),
                   dst_edge_first);
    }

    CUDA_TRY(hipStreamSynchronize(
      handle.get_stream()));  // edgelist_(major_vertices,minor_vertices,weights)
                              // will become out-of-scope
  }

  sort_and_coarsen_edgelist(coarsened_edgelist_major_vertices,
                            coarsened_edgelist_minor_vertices,
                            coarsened_edgelist_weights,
                            handle.get_stream());

  // 2. globally shuffle edge list and re-coarsen

  {
    auto edge_first =
      thrust::make_zip_iterator(thrust::make_tuple(coarsened_edgelist_major_vertices.begin(),
                                                   coarsened_edgelist_minor_vertices.begin(),
                                                   coarsened_edgelist_weights.begin()));
    auto key_func = detail::compute_gpu_id_from_edge_t<vertex_t, store_transposed>{
      graph_view.is_hypergraph_partitioned(),
      comm.get_size(),
      row_comm.get_size(),
      col_comm.get_size()};
    thrust::sort(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                 edge_first,
                 edge_first + coarsened_edgelist_major_vertices.size(),
                 [key_func] __device__(auto lhs, auto rhs) {
                   return store_transposed ? (key_func(thrust::get<1>(lhs), thrust::get<0>(lhs)) <
                                              key_func(thrust::get<1>(rhs), thrust::get<0>(rhs)))
                                           : (key_func(thrust::get<0>(lhs), thrust::get<1>(lhs)) <
                                              key_func(thrust::get<0>(rhs), thrust::get<1>(rhs)));
                 });
    auto key_first = thrust::make_transform_iterator(edge_first, [key_func] __device__(auto val) {
      return store_transposed ? key_func(thrust::get<1>(val), thrust::get<0>(val))
                              : key_func(thrust::get<0>(val), thrust::get<1>(val));
    });
    rmm::device_uvector<size_t> tx_value_counts(comm.get_size(), handle.get_stream());
    thrust::reduce_by_key(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                          key_first,
                          key_first + coarsened_edgelist_major_vertices.size(),
                          thrust::make_constant_iterator(size_t{1}),
                          thrust::make_discard_iterator(),
                          tx_value_counts.begin());

    rmm::device_uvector<vertex_t> rx_edgelist_major_vertices(0, handle.get_stream());
    rmm::device_uvector<vertex_t> rx_edgelist_minor_vertices(0, handle.get_stream());
    rmm::device_uvector<weight_t> rx_edgelist_weights(0, handle.get_stream());

    std::tie(
      rx_edgelist_major_vertices, rx_edgelist_minor_vertices, rx_edgelist_weights, std::ignore) =
      detail::shuffle_values(handle.get_comms(), edge_first, tx_value_counts, handle.get_stream());

    sort_and_coarsen_edgelist(rx_edgelist_major_vertices,
                              rx_edgelist_minor_vertices,
                              rx_edgelist_weights,
                              handle.get_stream());

    CUDA_TRY(hipStreamSynchronize(
      handle.get_stream()));  // memory blocks owned by
                              // coarsened_edgelist_(major_vertices,minor_vertices,weights)
                              // will be freed after the assignments below

    coarsened_edgelist_major_vertices = std::move(rx_edgelist_major_vertices);
    coarsened_edgelist_minor_vertices = std::move(rx_edgelist_minor_vertices);
    coarsened_edgelist_weights        = std::move(rx_edgelist_weights);
  }

  rmm::device_uvector<vertex_t> renumber_map_labels(0, handle.get_stream());
  partition_t<vertex_t> partition(
    std::vector<vertex_t>{}, graph_view.is_hypergraph_partitioned(), 0, 0, 0, 0);
  vertex_t number_of_vertices{};
  edge_t number_of_edges{};
  std::tie(renumber_map_labels, partition, number_of_vertices, number_of_edges) =
    renumber_edgelist<vertex_t, edge_t, multi_gpu>(handle,
                                                   coarsened_edgelist_major_vertices,
                                                   coarsened_edgelist_minor_vertices,
                                                   graph_view.is_hypergraph_partitioned());

  // 4. build a graph

  std::vector<edgelist_t<vertex_t, edge_t, weight_t>> edgelists{};
  if (graph_view.is_hypergraph_partitioned()) {
    CUGRAPH_FAIL("unimplemented.");
  } else {
    edgelists.resize(1);
    edgelists[0].p_src_vertices = store_transposed ? coarsened_edgelist_minor_vertices.data()
                                                   : coarsened_edgelist_major_vertices.data();
    edgelists[0].p_dst_vertices = store_transposed ? coarsened_edgelist_major_vertices.data()
                                                   : coarsened_edgelist_minor_vertices.data();
    edgelists[0].p_edge_weights  = coarsened_edgelist_weights.data();
    edgelists[0].number_of_edges = static_cast<edge_t>(coarsened_edgelist_major_vertices.size());
  }

  return std::make_tuple(
    std::make_unique<graph_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>>(
      handle,
      edgelists,
      partition,
      number_of_vertices,
      number_of_edges,
      graph_properties_t{graph_view.is_symmetric(), false},
      true),
    std::move(renumber_map_labels));
}

template <typename vertex_t,
          typename edge_t,
          typename weight_t,
          bool store_transposed,
          bool multi_gpu>
std::enable_if_t<
  !multi_gpu,
  std::tuple<std::unique_ptr<graph_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>>,
             rmm::device_uvector<vertex_t>>>
coarsen_graph(
  raft::handle_t const &handle,
  graph_view_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu> const &graph_view,
  vertex_t const *labels)
{
  rmm::device_uvector<vertex_t> coarsened_edgelist_major_vertices(0, handle.get_stream());
  rmm::device_uvector<vertex_t> coarsened_edgelist_minor_vertices(0, handle.get_stream());
  rmm::device_uvector<weight_t> coarsened_edgelist_weights(0, handle.get_stream());
  std::tie(coarsened_edgelist_major_vertices,
           coarsened_edgelist_minor_vertices,
           coarsened_edgelist_weights) =
    compressed_sparse_to_relabeled_and_sorted_and_coarsened_edgelist(
      graph_view.offsets(),
      graph_view.indices(),
      graph_view.weights(),
      labels,
      labels,
      vertex_t{0},
      graph_view.get_number_of_vertices(),
      vertex_t{0},
      graph_view.get_number_of_vertices(),
      handle.get_stream());

  sort_and_coarsen_edgelist(coarsened_edgelist_major_vertices,
                            coarsened_edgelist_minor_vertices,
                            coarsened_edgelist_weights,
                            handle.get_stream());

  auto renumber_map_labels = renumber_edgelist<vertex_t, edge_t, multi_gpu>(
    handle, coarsened_edgelist_major_vertices, coarsened_edgelist_minor_vertices);

  edgelist_t<vertex_t, edge_t, weight_t> edgelist{};
  edgelist.p_src_vertices = store_transposed ? coarsened_edgelist_minor_vertices.data()
                                             : coarsened_edgelist_major_vertices.data();
  edgelist.p_dst_vertices = store_transposed ? coarsened_edgelist_major_vertices.data()
                                             : coarsened_edgelist_minor_vertices.data();
  edgelist.p_edge_weights  = coarsened_edgelist_weights.data();
  edgelist.number_of_edges = static_cast<edge_t>(coarsened_edgelist_major_vertices.size());

  return std::make_tuple(
    std::make_unique<graph_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>>(
      handle,
      edgelist,
      static_cast<vertex_t>(renumber_map_labels.size()),
      graph_properties_t{graph_view.is_symmetric(), false},
      true),
    std::move(renumber_map_labels));
}

template <typename vertex_t, bool multi_gpu>
rmm::device_uvector<vertex_t> relabel(
  raft::handle_t const &handle,
  rmm::device_uvector<vertex_t> const &old_labels,
  std::tuple<rmm::device_uvector<vertex_t>, rmm::device_uvector<vertex_t>> const
    &old_new_label_pairs)
{
  double constexpr load_factor = 0.7;

  rmm::device_uvector<vertex_t> new_labels(0, handle.get_stream());

  if (multi_gpu) {
    auto &comm           = handle.get_comms();
    auto const comm_size = comm.get_size();

    auto key_func = detail::compute_gpu_id_from_vertex_t<vertex_t>{comm_size};

    // find unique old labels (to be relabeled)

    rmm::device_uvector<vertex_t> unique_old_labels(old_labels, handle.get_stream());
    thrust::sort(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                 unique_old_labels.begin(),
                 unique_old_labels.end());
    auto it = thrust::unique(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                             unique_old_labels.begin(),
                             unique_old_labels.end());
    unique_old_labels.resize(thrust::distance(unique_old_labels.begin(), it), handle.get_stream());
    unique_old_labels.shrink_to_fit(handle.get_stream());

    // collect new labels for the unique old labels

    rmm::device_uvector<vertex_t> new_labels_for_unique_old_labels(0, handle.get_stream());
    {
      // shuffle the old_new_label_pairs based on applying the compute_gpu_id_from_vertex_t functor
      // to the old labels

      rmm::device_uvector<vertex_t> rx_label_pair_old_labels(0, handle.get_stream());
      rmm::device_uvector<vertex_t> rx_label_pair_new_labels(0, handle.get_stream());
      {
        rmm::device_uvector<vertex_t> label_pair_old_labels(thrust::get<0>(old_new_label_pairs),
                                                            handle.get_stream());
        rmm::device_uvector<vertex_t> label_pair_new_labels(thrust::get<1>(old_new_label_pairs),
                                                            handle.get_stream());
        auto pair_first = thrust::make_zip_iterator(
          thrust::make_tuple(label_pair_old_labels.begin(), label_pair_new_labels.begin()));
        thrust::sort(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                     pair_first,
                     pair_first + thrust::get<0>(old_new_label_pairs).size(),
                     [key_func] __device__(auto lhs, auto rhs) {
                       return key_func(thrust::get<0>(lhs)) < key_func(thrust::get<0>(rhs));
                     });
        auto key_first = thrust::make_transform_iterator(
          label_pair_old_labels.begin(), [key_func] __device__(auto val) { return key_func(val); });
        rmm::device_uvector<size_t> tx_value_counts(comm_size, handle.get_stream());
        thrust::reduce_by_key(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                              key_first,
                              key_first + label_pair_old_labels.size(),
                              thrust::make_constant_iterator(size_t{1}),
                              thrust::make_discard_iterator(),
                              tx_value_counts.begin());

        std::tie(rx_label_pair_old_labels, rx_label_pair_new_labels, std::ignore) =
          cugraph::experimental::detail::shuffle_values(
            handle.get_comms(), pair_first, tx_value_counts, handle.get_comms());

        CUDA_TRY(hipStreamSynchronize(
          handle.get_stream()));  // label_pair_old_labels and label_pair_new_labels will become
                                  // out-of-scope
      }

      // update intermediate relabel map

      cuco::static_map<vertex_t, vertex_t> relabel_map{
        static_cast<size_t>(static_cast<double>(rx_label_pair_old_labels.size()) / load_factor),
        invalid_vertex_id<vertex_t>::value,
        invalid_vertex_id<vertex_t>::value};

      auto pair_first = thrust::make_transform_iterator(
        thrust::make_zip_iterator(
          thrust::make_tuple(rx_label_pair_old_labels.begin(), rx_label_pair_new_labels.begin())),
        [] __device__(auto val) {
          return thrust::make_pair(thrust::get<0>(val), thrust::get<1>(val));
        });
      relabel_map.insert(pair_first, pair_first + rx_label_pair_old_labels.size());

      rx_label_pair_old_labels.resize(0, handle.get_stream());
      rx_label_pair_new_labels.resize(0, handle.get_stream());
      rx_label_pair_old_labels.shrink_to_fit(handle.get_stream());
      rx_label_pair_new_labels.shrink_to_fit(handle.get_stream());

      // shuffle unique_old_labels, relabel using the intermediate relabel map, and shuffle back

      {
        thrust::sort(
          rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
          unique_old_labels.begin(),
          unique_old_labels.end(),
          [key_func] __device__(auto lhs, auto rhs) { return key_func(lhs) < key_func(rhs); });

        auto key_first = thrust::make_transform_iterator(
          unique_old_labels.begin(), [key_func] __device__(auto val) { return key_func(val); });
        rmm::device_uvector<size_t> tx_value_counts(comm_size, handle.get_stream());
        thrust::reduce_by_key(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                              key_first,
                              key_first + unique_old_labels.size(),
                              thrust::make_constant_iterator(size_t{1}),
                              thrust::make_discard_iterator(),
                              tx_value_counts.begin());

        rmm::device_uvector<size_t> rx_unique_old_labels(0, handle.get_stream());
        rmm::device_uvector<size_t> rx_value_counts(0, handle.get_stream());

        std::tie(rx_unique_old_labels, rx_value_counts) =
          cugraph::experimental::detail::shuffle_values(
            handle.get_comms(), unique_old_labels.begin(), tx_value_counts, handle.get_stream());

        CUDA_TRY(hipStreamSynchronize(
          handle.get_stream()));  // cuco::static_map currently does not take stream

        relabel_map.find(
          rx_unique_old_labels.begin(),
          rx_unique_old_labels.end(),
          rx_unique_old_labels
            .begin());  // now rx_unique_old_lables hold new labels for the corresponding old labels

        std::tie(new_labels_for_unique_old_labels, std::ignore) =
          cugraph::experimental::detail::shuffle_values(
            handle.get_comms(), rx_unique_old_labels.begin(), rx_value_counts, handle.get_stream());

        CUDA_TRY(hipStreamSynchronize(
          handle.get_stream()));  // tx_value_counts & rx_value_counts will become out-of-scope
      }
    }

    cuco::static_map<vertex_t, vertex_t> relabel_map(
      static_cast<size_t>(static_cast<double>(unique_old_labels.size()) / load_factor),
      invalid_vertex_id<vertex_t>::value,
      invalid_vertex_id<vertex_t>::value);

    auto pair_first = thrust::make_transform_iterator(
      thrust::make_zip_iterator(
        thrust::make_tuple(unique_old_labels.begin(), new_labels_for_unique_old_labels.begin())),
      [] __device__(auto val) {
        return thrust::make_pair(thrust::get<0>(val), thrust::get<1>(val));
      });

    relabel_map.insert(pair_first, pair_first + unique_old_labels.size());
    new_labels.resize(old_labels.size(), handle.get_stream());
    relabel_map.find(old_labels.begin(), old_labels.end(), new_labels.begin());
  } else {
    cuco::static_map<vertex_t, vertex_t> relabel_map(
      static_cast<size_t>(static_cast<double>(old_new_label_pairs.size()) / load_factor),
      invalid_vertex_id<vertex_t>::value,
      invalid_vertex_id<vertex_t>::value);

    auto pair_first = thrust::make_transform_iterator(
      thrust::make_zip_iterator(thrust::make_tuple(std::get<0>(old_new_label_pairs).begin(),
                                                   std::get<1>(old_new_label_pairs).begin())),
      [] __device__(auto val) {
        return thrust::make_pair(thrust::get<0>(val), thrust::get<1>(val));
      });

    relabel_map.insert(pair_first, pair_first + old_new_label_pairs.size());
    new_labels.resize(old_labels.size(), handle.get_stream());
    relabel_map.find(old_labels.begin(), old_labels.end(), new_labels.begin());
  }

  return std::move(new_labels);
}

// explicit instantiation

template std::tuple<std::unique_ptr<graph_t<int32_t, int32_t, float, false, true>>,
                    rmm::device_uvector<int32_t>>
coarsen_graph(raft::handle_t const &handle,
              graph_view_t<int32_t, int32_t, float, false, true> const &graph_view,
              int32_t const *labels);

template std::tuple<std::unique_ptr<graph_t<int32_t, int32_t, float, false, false>>,
                    rmm::device_uvector<int32_t>>
coarsen_graph(raft::handle_t const &handle,
              graph_view_t<int32_t, int32_t, float, false, false> const &graph_view,
              int32_t const *labels);

}  // namespace experimental
}  // namespace cugraph
