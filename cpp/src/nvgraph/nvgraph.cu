#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdio>
#include <cstdlib>
#include <climits>
#include <cfloat>
#include <vector>

#include <hipsolver.h>

#include <nvgraph/nvgraph.h>   // public header **This is NVGRAPH C API**

#include "include/nvlouvain.cuh"
#include "include/nvgraph_error.hxx"
#include "include/rmm_shared_ptr.hxx"
#include "include/valued_csr_graph.hxx"
#include "include/multi_valued_csr_graph.hxx"
#include "include/nvgraph_vector.hxx"
#include "include/nvgraph_cusparse.hxx"
#include "include/nvgraph_cublas.hxx"
#include "include/nvgraph_csrmv.hxx"
#include "include/partition.hxx"
#include "include/size2_selector.hxx"
#include "include/modularity_maximization.hxx"
#include "include/csrmv_cub.h"
#include "include/nvgraphP.h"  // private header, contains structures, and potentially other things, used in the public C API that should never be exposed.
#include "include/nvgraph_experimental.h"  // experimental header, contains hidden API entries, can be shared only under special circumstances without reveling internal things
#include "include/debug_macros.h"

static inline int check_context(const nvgraphHandle_t h) {
    int ret = 0;
    if (h == NULL || !h->nvgraphIsInitialized)
        ret = 1;
    return ret;
}

static inline int check_graph(const nvgraphGraphDescr_t d) {
    int ret = 0;
    if (d == NULL || d->graphStatus == IS_EMPTY)
        ret = 1;
    return ret;
}
static inline int check_topology(const nvgraphGraphDescr_t d) {
    int ret = 0;
    if (d->graphStatus == IS_EMPTY)
        ret = 1;
    return ret;
}

static inline int check_int_size(size_t sz) {
    int ret = 0;
    if (sz >= INT_MAX)
        ret = 1;
    return ret;
}

static inline int check_int_ptr(const int* p) {
    int ret = 0;
    if (!p)
        ret = 1;
    return ret;
}

static inline int check_uniform_type_array(const hipDataType * t, size_t sz) {
    int ret = 0;
    hipDataType uniform_type = t[0];
    for (size_t i = 1; i < sz; i++)
            {
        if (t[i] != uniform_type)
            ret = 1;
    }
    return ret;
}

template<typename T>
bool check_ptr(const T* p) {
    bool ret = false;
    if (!p)
        ret = true;
    return ret;
}

namespace nvgraph
{
    nvgraphStatus_t getCAPIStatusForError(NVGRAPH_ERROR err) {
        nvgraphStatus_t ret = NVGRAPH_STATUS_SUCCESS;

        switch (err) {
            case NVGRAPH_OK:
                ret = NVGRAPH_STATUS_SUCCESS;
                break;
            case NVGRAPH_ERR_BAD_PARAMETERS:
                ret = NVGRAPH_STATUS_INVALID_VALUE;
                break;
            case NVGRAPH_ERR_UNKNOWN:
                ret = NVGRAPH_STATUS_INTERNAL_ERROR;
                break;
            case NVGRAPH_ERR_CUDA_FAILURE:
                ret = NVGRAPH_STATUS_EXECUTION_FAILED;
                break;
            case NVGRAPH_ERR_THRUST_FAILURE:
                ret = NVGRAPH_STATUS_EXECUTION_FAILED;
                break;
            case NVGRAPH_ERR_IO:
                ret = NVGRAPH_STATUS_INTERNAL_ERROR;
                break;
            case NVGRAPH_ERR_NOT_IMPLEMENTED:
                ret = NVGRAPH_STATUS_INVALID_VALUE;
                break;
            case NVGRAPH_ERR_NO_MEMORY:
                ret = NVGRAPH_STATUS_ALLOC_FAILED;
                break;
            case NVGRAPH_ERR_NOT_CONVERGED:
                ret = NVGRAPH_STATUS_NOT_CONVERGED;
                break;
            default:
                ret = NVGRAPH_STATUS_INTERNAL_ERROR;
        }
        return ret;
    }

    extern "C" {
        const char* nvgraphStatusGetString(nvgraphStatus_t status) {
            switch (status) {
                case NVGRAPH_STATUS_SUCCESS:
                    return "Success";
                case NVGRAPH_STATUS_NOT_INITIALIZED:
                    return "nvGRAPH not initialized";
                case NVGRAPH_STATUS_ALLOC_FAILED:
                    return "nvGRAPH alloc failed";
                case NVGRAPH_STATUS_INVALID_VALUE:
                    return "nvGRAPH invalid value";
                case NVGRAPH_STATUS_ARCH_MISMATCH:
                    return "nvGRAPH arch mismatch";
                case NVGRAPH_STATUS_MAPPING_ERROR:
                    return "nvGRAPH mapping error";
                case NVGRAPH_STATUS_EXECUTION_FAILED:
                    return "nvGRAPH execution failed";
                case NVGRAPH_STATUS_INTERNAL_ERROR:
                    return "nvGRAPH internal error";
                case NVGRAPH_STATUS_TYPE_NOT_SUPPORTED:
                    return "nvGRAPH type not supported";
                case NVGRAPH_STATUS_NOT_CONVERGED:
                    return "nvGRAPH algorithm failed to converge";
                case NVGRAPH_STATUS_GRAPH_TYPE_NOT_SUPPORTED:
                    return "nvGRAPH graph type not supported";
                default:
                    return "Unknown nvGRAPH Status";
            }
        }
    }

    static nvgraphStatus_t nvgraphCreate_impl(struct nvgraphContext **outCtx) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            // First, initialize NVGraph's context

            auto ctx = static_cast<struct nvgraphContext*>(calloc(1, sizeof(struct nvgraphContext)));
            if (ctx == nullptr) {
                FatalError("Cannot allocate NVGRAPH context.", NVGRAPH_ERR_UNKNOWN);
            }

            // Now NVGraph assumes that RMM is initialized outside NVGraph
            // if RMM is unintialized, RMM_ALLOC/RMM_FREE are just aliases for hipMalloc/hipFree

            ctx->stream = nullptr;
            ctx->nvgraphIsInitialized = true;

             if (outCtx != nullptr) {
                 *outCtx = ctx;
             }

            // Second, initialize Cublas and Cusparse (get_handle() creates a new handle
            // if there is no existing handle).

            nvgraph::Cusparse::get_handle();
            nvgraph::Cublas::get_handle();
        }
        NVGRAPH_CATCHES(rc)

        return getCAPIStatusForError(rc);
    }

    static nvgraphStatus_t nvgraphDestroy_impl(nvgraphHandle_t handle) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle))
                FatalError("Cannot initialize memory manager.", NVGRAPH_ERR_NO_MEMORY);

            // First, destroy Cublas and Cusparse

            nvgraph::Cusparse::destroy_handle();
            nvgraph::Cublas::destroy_handle();

            // Second, destroy NVGraph's context

            free(handle);
        }
        NVGRAPH_CATCHES(rc)

        return getCAPIStatusForError(rc);
    }

    static nvgraphStatus_t nvgraphCreateGraphDescr_impl(nvgraphHandle_t handle,
                                                        struct nvgraphGraphDescr **outGraphDescr) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);

            struct nvgraphGraphDescr *descrG = NULL;
            descrG = (struct nvgraphGraphDescr*) malloc(sizeof(*descrG));
            if (!descrG)
            {
                FatalError("Cannot allocate graph descriptor.", NVGRAPH_ERR_UNKNOWN);
            }
            descrG->graphStatus = IS_EMPTY;
            if (outGraphDescr)
            {
                *outGraphDescr = descrG;
            }
        }
        NVGRAPH_CATCHES(rc)

        return getCAPIStatusForError(rc);
    }

    static nvgraphStatus_t nvgraphDestroyGraphDescr_impl(nvgraphHandle_t handle,
                                                         struct nvgraphGraphDescr *descrG) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);

            if (descrG) {
                    switch (descrG->graphStatus) {
                        case IS_EMPTY: {
                            break;
                        }
                        case HAS_TOPOLOGY: {
                            nvgraph::CsrGraph<int> *CSRG =
                                    static_cast<nvgraph::CsrGraph<int>*>(descrG->graph_handle);
                            delete CSRG;
                            break;
                        }
                        case HAS_VALUES: {
                            if (descrG->T == HIP_R_32F) {
                                nvgraph::MultiValuedCsrGraph<int, float> *MCSRG =
                                        static_cast<nvgraph::MultiValuedCsrGraph<int, float>*>(descrG->graph_handle);
                                delete MCSRG;
                            }
                            else if (descrG->T == HIP_R_64F) {
                                nvgraph::MultiValuedCsrGraph<int, double> *MCSRG =
                                        static_cast<nvgraph::MultiValuedCsrGraph<int, double>*>(descrG->graph_handle);
                                delete MCSRG;
                            }
                            else if (descrG->T == HIP_R_32I) {
                                nvgraph::MultiValuedCsrGraph<int, int> *MCSRG =
                                        static_cast<nvgraph::MultiValuedCsrGraph<int, int>*>(descrG->graph_handle);
                                delete MCSRG;
                            }
                            else
                                return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;
                            break;
                        }
                        default:
                            return NVGRAPH_STATUS_INVALID_VALUE;
                    }
                free(descrG);
            }
            else
                return NVGRAPH_STATUS_INVALID_VALUE;
        }
        NVGRAPH_CATCHES(rc)

        return getCAPIStatusForError(rc);
    }

    nvgraphStatus_t NVGRAPH_API nvgraphSetStream_impl(nvgraphHandle_t handle, hipStream_t stream) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);
            // nvgraph handle
            handle->stream = stream;
            //Cublas and Cusparse
            nvgraph::Cublas::setStream(stream);
            nvgraph::Cusparse::setStream(stream);
        }
        NVGRAPH_CATCHES(rc)

        return getCAPIStatusForError(rc);
    }

    nvgraphStatus_t NVGRAPH_API nvgraphSetGraphStructure_impl(nvgraphHandle_t handle,
                                                              nvgraphGraphDescr_t descrG,
                                                              void* topologyData,
                                                              nvgraphTopologyType_t TT) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);
            if (descrG->graphStatus != IS_EMPTY)
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);
            if (check_ptr(topologyData))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);

            if (TT == NVGRAPH_CSR_32 || TT == NVGRAPH_CSC_32)
                    {
                int v = 0, e = 0, *neighborhood = NULL, *edgedest = NULL;
                switch (TT)
                {
                    case NVGRAPH_CSR_32:
                        {
                        nvgraphCSRTopology32I_t t = static_cast<nvgraphCSRTopology32I_t>(topologyData);
                        if (!t->nvertices || !t->nedges || check_ptr(t->source_offsets)
                                || check_ptr(t->destination_indices))
                            return NVGRAPH_STATUS_INVALID_VALUE;
                        v = t->nvertices;
                        e = t->nedges;
                        neighborhood = t->source_offsets;
                        edgedest = t->destination_indices;
                        break;
                    }
                    case NVGRAPH_CSC_32:
                        {
                        nvgraphCSCTopology32I_t t = static_cast<nvgraphCSCTopology32I_t>(topologyData);
                        if (!t->nvertices || !t->nedges || check_ptr(t->destination_offsets)
                                || check_ptr(t->source_indices))
                            return NVGRAPH_STATUS_INVALID_VALUE;
                        v = t->nvertices;
                        e = t->nedges;
                        neighborhood = t->destination_offsets;
                        edgedest = t->source_indices;
                        break;
                    }
                    default:
                        return NVGRAPH_STATUS_INVALID_VALUE;
                }

                descrG->TT = TT;

                // Create the internal CSR representation
                nvgraph::CsrGraph<int> * CSRG = new nvgraph::CsrGraph<int>(v, e, handle->stream);

                CHECK_CUDA(hipMemcpy(CSRG->get_raw_row_offsets(),
                                      neighborhood,
                                      (size_t )((CSRG->get_num_vertices() + 1) * sizeof(int)),
                                      hipMemcpyDefault));

                CHECK_CUDA(hipMemcpy(CSRG->get_raw_column_indices(),
                                      edgedest,
                                      (size_t )((CSRG->get_num_edges()) * sizeof(int)),
                                      hipMemcpyDefault));

                // Set the graph handle
                descrG->graph_handle = CSRG;
                descrG->graphStatus = HAS_TOPOLOGY;
            }
            else
            {
                return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;
            }

        }
        NVGRAPH_CATCHES(rc)

        return getCAPIStatusForError(rc);

    }

    nvgraphStatus_t NVGRAPH_API nvgraphAttachGraphStructure_impl(nvgraphHandle_t handle,
                                                            nvgraphGraphDescr_t descrG,
                                                            void* topologyData,
                                                            nvgraphTopologyType_t TT) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);
            if (descrG->graphStatus != IS_EMPTY)
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);
            if (check_ptr(topologyData))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);

            if (TT == NVGRAPH_CSR_32 || TT == NVGRAPH_CSC_32)
                    {
                int v = 0, e = 0, *neighborhood = NULL, *edgedest = NULL;
                switch (TT)
                {
                    case NVGRAPH_CSR_32:
                        {
                        nvgraphCSRTopology32I_t t = static_cast<nvgraphCSRTopology32I_t>(topologyData);
                        if (!t->nvertices || !t->nedges || check_ptr(t->source_offsets)
                                || check_ptr(t->destination_indices))
                            return NVGRAPH_STATUS_INVALID_VALUE;
                        v = t->nvertices;
                        e = t->nedges;
                        neighborhood = t->source_offsets;
                        edgedest = t->destination_indices;
                        break;
                    }
                    case NVGRAPH_CSC_32:
                        {
                        nvgraphCSCTopology32I_t t = static_cast<nvgraphCSCTopology32I_t>(topologyData);
                        if (!t->nvertices || !t->nedges || check_ptr(t->destination_offsets)
                                || check_ptr(t->source_indices))
                            return NVGRAPH_STATUS_INVALID_VALUE;
                        v = t->nvertices;
                        e = t->nedges;
                        neighborhood = t->destination_offsets;
                        edgedest = t->source_indices;
                        break;
                    }
                    default:
                        return NVGRAPH_STATUS_INVALID_VALUE;
                }

                descrG->TT = TT;

                // Create the internal CSR representation
                nvgraph::CsrGraph<int> * CSRG = new nvgraph::CsrGraph<int>(v, e, handle->stream);

                CSRG->set_raw_row_offsets(neighborhood);
                CSRG->set_raw_column_indices(edgedest);

                // Set the graph handle
                descrG->graph_handle = CSRG;
                descrG->graphStatus = HAS_TOPOLOGY;
            }
            else
            {
                return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;
            }

        }
        NVGRAPH_CATCHES(rc)

        return getCAPIStatusForError(rc);

    }

    nvgraphStatus_t NVGRAPH_API nvgraphGetGraphStructure_impl(nvgraphHandle_t handle,
                                                              nvgraphGraphDescr_t descrG,
                                                              void* topologyData,
                                                              nvgraphTopologyType_t* TT) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle) || check_graph(descrG) || check_topology(descrG))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);

            nvgraphTopologyType_t graphTType = descrG->TT;

            if (TT != NULL)
                *TT = graphTType;

            if (topologyData != NULL) {
                nvgraph::CsrGraph<int> *CSRG =
                        static_cast<nvgraph::CsrGraph<int> *>(descrG->graph_handle);
                int v = static_cast<int>(CSRG->get_num_vertices());
                int e = static_cast<int>(CSRG->get_num_edges());
                int *neighborhood = NULL, *edgedest = NULL;

                switch (graphTType)
                {
                    case NVGRAPH_CSR_32:
                        {
                        nvgraphCSRTopology32I_t t = static_cast<nvgraphCSRTopology32I_t>(topologyData);
                        t->nvertices = static_cast<int>(v);
                        t->nedges = static_cast<int>(e);
                        neighborhood = t->source_offsets;
                        edgedest = t->destination_indices;
                        break;
                    }
                    case NVGRAPH_CSC_32:
                        {
                        nvgraphCSCTopology32I_t t = static_cast<nvgraphCSCTopology32I_t>(topologyData);
                        t->nvertices = static_cast<int>(v);
                        t->nedges = static_cast<int>(e);
                        neighborhood = t->destination_offsets;
                        edgedest = t->source_indices;
                        break;
                    }
                    default:
                        return NVGRAPH_STATUS_INTERNAL_ERROR;
                }

                if (neighborhood != NULL) {
                    CHECK_CUDA(hipMemcpy(neighborhood,
                                          CSRG->get_raw_row_offsets(),
                                          (size_t )((v + 1) * sizeof(int)),
                                          hipMemcpyDefault));
                }

                if (edgedest != NULL) {
                    CHECK_CUDA(hipMemcpy(edgedest,
                                          CSRG->get_raw_column_indices(),
                                          (size_t )((e) * sizeof(int)),
                                          hipMemcpyDefault));
                }

            }
        }
        NVGRAPH_CATCHES(rc)
        return getCAPIStatusForError(rc);
    }

    nvgraphStatus_t NVGRAPH_API nvgraphAllocateVertexData_impl(nvgraphHandle_t handle,
                                                               nvgraphGraphDescr_t descrG,
                                                               size_t numsets,
                                                               hipDataType *settypes) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle) || check_graph(descrG) || check_int_size(numsets)
                    || check_ptr(settypes))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);
            if (check_uniform_type_array(settypes, numsets))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);

            if (descrG->graphStatus == HAS_TOPOLOGY) // need to convert CsrGraph to MultiValuedCsrGraph first
                    {
                if (*settypes == HIP_R_32F)
                        {
                    nvgraph::CsrGraph<int> *CSRG =
                            static_cast<nvgraph::CsrGraph<int>*>(descrG->graph_handle);
                    nvgraph::MultiValuedCsrGraph<int, float> *MCSRG = new nvgraph::MultiValuedCsrGraph<
                            int, float>(*CSRG);
                    descrG->graph_handle = MCSRG;
                }
                else if (*settypes == HIP_R_64F)
                        {
                    nvgraph::CsrGraph<int> *CSRG =
                            static_cast<nvgraph::CsrGraph<int>*>(descrG->graph_handle);
                    nvgraph::MultiValuedCsrGraph<int, double> *MCSRG = new nvgraph::MultiValuedCsrGraph<
                            int, double>(*CSRG);
                    descrG->graph_handle = MCSRG;
                }
                else if (*settypes == HIP_R_32I)
                        {
                    nvgraph::CsrGraph<int> *CSRG =
                            static_cast<nvgraph::CsrGraph<int>*>(descrG->graph_handle);
                    nvgraph::MultiValuedCsrGraph<int, int> *MCSRG = new nvgraph::MultiValuedCsrGraph<int,
                            int>(*CSRG);
                    descrG->graph_handle = MCSRG;
                }
                else
                    return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;
                descrG->T = *settypes;
                descrG->graphStatus = HAS_VALUES;
            }
            else if (descrG->graphStatus == HAS_VALUES) // Already in MultiValuedCsrGraph, just need to check the type
                    {
                if (*settypes != descrG->T)
                    return NVGRAPH_STATUS_INVALID_VALUE;
            }
            else
                return NVGRAPH_STATUS_INVALID_VALUE;

            // Allocate and transfer
            if (*settypes == HIP_R_32F)
                    {
                nvgraph::MultiValuedCsrGraph<int, float> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, float>*>(descrG->graph_handle);
                MCSRG->allocateVertexData(numsets, NULL);
            }
            else if (*settypes == HIP_R_64F)
                    {
                nvgraph::MultiValuedCsrGraph<int, double> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, double>*>(descrG->graph_handle);
                MCSRG->allocateVertexData(numsets, NULL);
            }
            else if (*settypes == HIP_R_32I)
                    {
                nvgraph::MultiValuedCsrGraph<int, int> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, int>*>(descrG->graph_handle);
                MCSRG->allocateVertexData(numsets, NULL);
            }
            else
                return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;
        }
        NVGRAPH_CATCHES(rc)

        return getCAPIStatusForError(rc);
    }

    nvgraphStatus_t NVGRAPH_API nvgraphAllocateEdgeData_impl(nvgraphHandle_t handle,
                                                             nvgraphGraphDescr_t descrG,
                                                             size_t numsets,
                                                             hipDataType *settypes) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle) || check_graph(descrG) || check_int_size(numsets)
                    || check_ptr(settypes))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);
            if (check_uniform_type_array(settypes, numsets))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);
            // Look at what kind of graph we have
            if (descrG->graphStatus == HAS_TOPOLOGY) // need to convert CsrGraph to MultiValuedCsrGraph first
                    {
                if (*settypes == HIP_R_32F)
                        {
                    nvgraph::CsrGraph<int> *CSRG =
                            static_cast<nvgraph::CsrGraph<int>*>(descrG->graph_handle);
                    nvgraph::MultiValuedCsrGraph<int, float> *MCSRG = new nvgraph::MultiValuedCsrGraph<
                            int, float>(*CSRG);
                    descrG->graph_handle = MCSRG;
                }
                else if (*settypes == HIP_R_64F)
                        {
                    nvgraph::CsrGraph<int> *CSRG =
                            static_cast<nvgraph::CsrGraph<int>*>(descrG->graph_handle);
                    nvgraph::MultiValuedCsrGraph<int, double> *MCSRG = new nvgraph::MultiValuedCsrGraph<
                            int, double>(*CSRG);
                    descrG->graph_handle = MCSRG;
                }
                else if (*settypes == HIP_R_32I)
                        {
                    nvgraph::CsrGraph<int> *CSRG =
                            static_cast<nvgraph::CsrGraph<int>*>(descrG->graph_handle);
                    nvgraph::MultiValuedCsrGraph<int, int> *MCSRG = new nvgraph::MultiValuedCsrGraph<int,
                            int>(*CSRG);
                    descrG->graph_handle = MCSRG;
                }
                else
                    return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;
                descrG->T = *settypes;
                descrG->graphStatus = HAS_VALUES;
            }
            else if (descrG->graphStatus == HAS_VALUES) // Already in MultiValuedCsrGraph, just need to check the type
                    {
                if (*settypes != descrG->T)
                    return NVGRAPH_STATUS_INVALID_VALUE;
            }
            else
                return NVGRAPH_STATUS_INVALID_VALUE;

            // Allocate and transfer
            if (*settypes == HIP_R_32F)
                    {
                nvgraph::MultiValuedCsrGraph<int, float> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, float>*>(descrG->graph_handle);
                MCSRG->allocateEdgeData(numsets, NULL);
            }
            else if (*settypes == HIP_R_64F)
                    {
                nvgraph::MultiValuedCsrGraph<int, double> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, double>*>(descrG->graph_handle);
                MCSRG->allocateEdgeData(numsets, NULL);
            }
            else if (*settypes == HIP_R_32I)
                    {
                nvgraph::MultiValuedCsrGraph<int, int> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, int>*>(descrG->graph_handle);
                MCSRG->allocateEdgeData(numsets, NULL);
            }
            else
                return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;

        }
        NVGRAPH_CATCHES(rc)

        return getCAPIStatusForError(rc);
    }

    nvgraphStatus_t NVGRAPH_API nvgraphAttachEdgeData_impl(nvgraphHandle_t handle,
                                                           nvgraphGraphDescr_t descrG,
                                                           size_t setnum,
                                                           hipDataType settype,
                                                           void *edgeData) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle) || check_graph(descrG) || check_int_size(setnum))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);
            // Look at what kind of graph we have
            if (descrG->graphStatus == HAS_TOPOLOGY) // need to convert CsrGraph to MultiValuedCsrGraph first
                    {
                if (settype == HIP_R_32F)
                        {
                    nvgraph::CsrGraph<int> *CSRG =
                            static_cast<nvgraph::CsrGraph<int>*>(descrG->graph_handle);
                    nvgraph::MultiValuedCsrGraph<int, float> *MCSRG = new nvgraph::MultiValuedCsrGraph<
                            int, float>(*CSRG);
                    descrG->graph_handle = MCSRG;
                }
                else if (settype == HIP_R_64F)
                        {
                    nvgraph::CsrGraph<int> *CSRG =
                            static_cast<nvgraph::CsrGraph<int>*>(descrG->graph_handle);
                    nvgraph::MultiValuedCsrGraph<int, double> *MCSRG = new nvgraph::MultiValuedCsrGraph<
                            int, double>(*CSRG);
                    descrG->graph_handle = MCSRG;
                }
                else if (settype == HIP_R_32I)
                        {
                    nvgraph::CsrGraph<int> *CSRG =
                            static_cast<nvgraph::CsrGraph<int>*>(descrG->graph_handle);
                    nvgraph::MultiValuedCsrGraph<int, int> *MCSRG = new nvgraph::MultiValuedCsrGraph<int,
                            int>(*CSRG);
                    descrG->graph_handle = MCSRG;
                }
                else
                    return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;
                descrG->T = settype;
                descrG->graphStatus = HAS_VALUES;
            }
            else if (descrG->graphStatus == HAS_VALUES) // Already in MultiValuedCsrGraph, just need to check the type
                    {
                if (settype != descrG->T)
                    return NVGRAPH_STATUS_INVALID_VALUE;
            }
            else
                return NVGRAPH_STATUS_INVALID_VALUE;

            // Allocate and transfer
            if (settype == HIP_R_32F)
                    {
                nvgraph::MultiValuedCsrGraph<int, float> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, float>*>(descrG->graph_handle);
                MCSRG->attachEdgeData(setnum, (float*)edgeData, NULL);
            }
            else if (settype == HIP_R_64F)
                    {
                nvgraph::MultiValuedCsrGraph<int, double> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, double>*>(descrG->graph_handle);
                MCSRG->attachEdgeData(setnum, (double*)edgeData, NULL);
            }
            else if (settype == HIP_R_32I)
                    {
                nvgraph::MultiValuedCsrGraph<int, int> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, int>*>(descrG->graph_handle);
                MCSRG->attachEdgeData(setnum, (int*)edgeData, NULL);
            }
            else
                return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;

        }
        NVGRAPH_CATCHES(rc)

        return getCAPIStatusForError(rc);
    }

    nvgraphStatus_t NVGRAPH_API nvgraphSetVertexData_impl(nvgraphHandle_t handle,
                                                          nvgraphGraphDescr_t descrG,
                                                          void *vertexData,
                                                          size_t setnum) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle) || check_graph(descrG) || check_int_size(setnum)
                    || check_ptr(vertexData))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);

            if (descrG->graphStatus != HAS_VALUES) // need a MultiValuedCsrGraph
                FatalError("Graph should have allocated values.", NVGRAPH_ERR_BAD_PARAMETERS);

            if (descrG->T == HIP_R_32F)
                    {
                nvgraph::MultiValuedCsrGraph<int, float> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, float>*>(descrG->graph_handle);
                if (setnum >= MCSRG->get_num_vertex_dim()) // base index is 0
                    return NVGRAPH_STATUS_INVALID_VALUE;
                hipMemcpy(MCSRG->get_raw_vertex_dim(setnum),
                           (float*) vertexData,
                           (size_t) ((MCSRG->get_num_vertices()) * sizeof(float)),
                           hipMemcpyDefault);
            }
            else if (descrG->T == HIP_R_64F)
                    {
                nvgraph::MultiValuedCsrGraph<int, double> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, double>*>(descrG->graph_handle);
                if (setnum >= MCSRG->get_num_vertex_dim()) // base index is 0
                    return NVGRAPH_STATUS_INVALID_VALUE;
                hipMemcpy(MCSRG->get_raw_vertex_dim(setnum),
                           (double*) vertexData,
                           (size_t) ((MCSRG->get_num_vertices()) * sizeof(double)),
                           hipMemcpyDefault);
            }
            else if (descrG->T == HIP_R_32I)
                    {
                nvgraph::MultiValuedCsrGraph<int, int> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, int>*>(descrG->graph_handle);
                if (setnum >= MCSRG->get_num_vertex_dim()) // base index is 0
                    return NVGRAPH_STATUS_INVALID_VALUE;
                hipMemcpy(MCSRG->get_raw_vertex_dim(setnum),
                           (int*) vertexData,
                           (size_t) ((MCSRG->get_num_vertices()) * sizeof(int)),
                           hipMemcpyDefault);
            }
            else
                return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;

            cudaCheckError();
        }
        NVGRAPH_CATCHES(rc)

        return getCAPIStatusForError(rc);
    }

    nvgraphStatus_t NVGRAPH_API nvgraphGetVertexData_impl(nvgraphHandle_t handle,
                                                          nvgraphGraphDescr_t descrG,
                                                          void *vertexData,
                                                          size_t setnum) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle) || check_graph(descrG) || check_int_size(setnum)
                    || check_ptr(vertexData))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);

            if (descrG->graphStatus != HAS_VALUES) // need a MultiValuedCsrGraph
                FatalError("Graph should have values.", NVGRAPH_ERR_BAD_PARAMETERS);

            if (descrG->T == HIP_R_32F)
                    {
                nvgraph::MultiValuedCsrGraph<int, float> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, float>*>(descrG->graph_handle);
                if (setnum >= MCSRG->get_num_vertex_dim()) // base index is 0
                    return NVGRAPH_STATUS_INVALID_VALUE;
                hipMemcpy((float*) vertexData,
                                MCSRG->get_raw_vertex_dim(setnum),
                                (size_t) ((MCSRG->get_num_vertices()) * sizeof(float)),
                                hipMemcpyDefault);
            }
            else if (descrG->T == HIP_R_64F)
                    {
                nvgraph::MultiValuedCsrGraph<int, double> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, double>*>(descrG->graph_handle);
                if (setnum >= MCSRG->get_num_vertex_dim()) // base index is 0
                    return NVGRAPH_STATUS_INVALID_VALUE;
                hipMemcpy((double*) vertexData,
                                MCSRG->get_raw_vertex_dim(setnum),
                                (size_t) ((MCSRG->get_num_vertices()) * sizeof(double)),
                                hipMemcpyDefault);
            }
            else if (descrG->T == HIP_R_32I)
                    {
                nvgraph::MultiValuedCsrGraph<int, int> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, int>*>(descrG->graph_handle);
                if (setnum >= MCSRG->get_num_vertex_dim()) // base index is 0
                    return NVGRAPH_STATUS_INVALID_VALUE;
                hipMemcpy((int*) vertexData,
                                MCSRG->get_raw_vertex_dim(setnum),
                                (size_t) ((MCSRG->get_num_vertices()) * sizeof(int)),
                                hipMemcpyDefault);
            }
            else
                return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;

            cudaCheckError();
        }
        NVGRAPH_CATCHES(rc)

        return getCAPIStatusForError(rc);
    }

    nvgraphStatus_t NVGRAPH_API nvgraphSetEdgeData_impl(nvgraphHandle_t handle,
                                                        nvgraphGraphDescr_t descrG,
                                                        void *edgeData,
                                                        size_t setnum) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle) || check_graph(descrG) || check_int_size(setnum)
                    || check_ptr(edgeData))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);

            if (descrG->graphStatus != HAS_VALUES) // need a MultiValuedCsrGraph
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (descrG->T == HIP_R_32F)
                    {
                nvgraph::MultiValuedCsrGraph<int, float> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, float>*>(descrG->graph_handle);
                if (setnum >= MCSRG->get_num_edge_dim()) // base index is 0
                    return NVGRAPH_STATUS_INVALID_VALUE;
                hipMemcpy(MCSRG->get_raw_edge_dim(setnum),
                                (float*) edgeData,
                                (size_t) ((MCSRG->get_num_edges()) * sizeof(float)),
                                hipMemcpyDefault);
            }
            else if (descrG->T == HIP_R_64F)
                    {
                nvgraph::MultiValuedCsrGraph<int, double> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, double>*>(descrG->graph_handle);
                if (setnum >= MCSRG->get_num_edge_dim()) // base index is 0
                    return NVGRAPH_STATUS_INVALID_VALUE;
                hipMemcpy(MCSRG->get_raw_edge_dim(setnum),
                                (double*) edgeData,
                                (size_t) ((MCSRG->get_num_edges()) * sizeof(double)),
                                hipMemcpyDefault);
            }
            else if (descrG->T == HIP_R_32I)
                    {
                nvgraph::MultiValuedCsrGraph<int, int> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, int>*>(descrG->graph_handle);
                if (setnum >= MCSRG->get_num_edge_dim()) // base index is 0
                    return NVGRAPH_STATUS_INVALID_VALUE;
                hipMemcpy(MCSRG->get_raw_edge_dim(setnum),
                                (int*) edgeData,
                                (size_t) ((MCSRG->get_num_edges()) * sizeof(int)),
                                hipMemcpyDefault);
            }
            else
                return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;

            cudaCheckError();
        }
        NVGRAPH_CATCHES(rc)

        return getCAPIStatusForError(rc);
    }

    nvgraphStatus_t NVGRAPH_API nvgraphGetEdgeData_impl(nvgraphHandle_t handle,
                                                        nvgraphGraphDescr_t descrG,
                                                        void *edgeData,
                                                        size_t setnum) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle) || check_graph(descrG) || check_int_size(setnum)
                    || check_ptr(edgeData))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);

            if (descrG->graphStatus != HAS_VALUES) // need a MultiValuedCsrGraph
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (descrG->T == HIP_R_32F)
                    {
                nvgraph::MultiValuedCsrGraph<int, float> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, float>*>(descrG->graph_handle);
                if (setnum >= MCSRG->get_num_edge_dim()) // base index is 0
                    return NVGRAPH_STATUS_INVALID_VALUE;
                hipMemcpy((float*) edgeData,
                                MCSRG->get_raw_edge_dim(setnum),
                                (size_t) ((MCSRG->get_num_edges()) * sizeof(float)),
                                hipMemcpyDefault);
            }
            else if (descrG->T == HIP_R_64F)
                    {
                nvgraph::MultiValuedCsrGraph<int, double> *MCSRG =
                        static_cast<nvgraph::MultiValuedCsrGraph<int, double>*>(descrG->graph_handle);
                if (setnum >= MCSRG->get_num_edge_dim()) // base index is 0
                    return NVGRAPH_STATUS_INVALID_VALUE;
                hipMemcpy((double*) edgeData,
                                MCSRG->get_raw_edge_dim(setnum),
                                (size_t) ((MCSRG->get_num_edges()) * sizeof(double)),
                                hipMemcpyDefault);
            }
            else
                return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;

            cudaCheckError();
        }
        NVGRAPH_CATCHES(rc)

        return getCAPIStatusForError(rc);
    }

    nvgraphStatus_t NVGRAPH_API nvgraphBalancedCutClustering_impl(nvgraphHandle_t handle,
                                                                  const nvgraphGraphDescr_t descrG,
                                                                  const size_t weight_index,
                                                                  const int n_clusters,
                                                                  const int n_eig_vects,
                                                                  const int evs_type,
                                                                  const float evs_tolerance,
                                                                  const int evs_max_iter,
                                                                  const float kmean_tolerance,
                                                                  const int kmean_max_iter,
                                                                  int* clustering,
                                                                  void* eig_vals,
                                                                  void* eig_vects) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle) || check_graph(descrG) || check_int_size(weight_index))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);

            if (descrG->graphStatus != HAS_VALUES) // need a MultiValuedCsrGraph
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (descrG->TT != NVGRAPH_CSR_32) // supported topologies
                return NVGRAPH_STATUS_INVALID_VALUE;

            int evs_max_it, kmean_max_it;
            int iters_lanczos, iters_kmeans;
            float evs_tol, kmean_tol;

            if (evs_max_iter > 0)
                evs_max_it = evs_max_iter;
            else
                evs_max_it = 4000;

            if (evs_tolerance == 0.0f)
                evs_tol = 1.0E-3f;
            else if (evs_tolerance < 1.0f && evs_tolerance > 0.0f)
                evs_tol = evs_tolerance;
            else
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (kmean_max_iter > 0)
                kmean_max_it = kmean_max_iter;
            else
                kmean_max_it = 200;

            if (kmean_tolerance == 0.0f)
                kmean_tol = 1.0E-2f;
            else if (kmean_tolerance < 1.0f && kmean_tolerance > 0.0f)
                kmean_tol = kmean_tolerance;
            else
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (n_clusters < 2)
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (n_eig_vects > n_clusters)
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (!(evs_type == 0 || evs_type == 1))
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (clustering == NULL || eig_vals == NULL || eig_vects == NULL)
                return NVGRAPH_STATUS_INVALID_VALUE;

            switch (descrG->T)
            {
                case HIP_R_32F:
                    {
                    nvgraph::MultiValuedCsrGraph<int, float> *MCSRG =
                            static_cast<nvgraph::MultiValuedCsrGraph<int, float>*>(descrG->graph_handle);
                    if (weight_index >= MCSRG->get_num_edge_dim()
                            || n_clusters > static_cast<int>(MCSRG->get_num_vertices())) // base index is 0
                        return NVGRAPH_STATUS_INVALID_VALUE;
                    nvgraph::ValuedCsrGraph<int, float> network =
                            *MCSRG->get_valued_csr_graph(weight_index);
                    Vector<int> clust(MCSRG->get_num_vertices(), handle->stream);
                    Vector<float> eigVals(n_eig_vects, handle->stream);
                    Vector<float> eigVecs(MCSRG->get_num_vertices() * n_eig_vects, handle->stream);

                    if (evs_type == 0)
                            {
                        int restartIter_lanczos = 15 + n_eig_vects;
                        rc = partition<int, float>(network,
                                                   n_clusters,
                                                   n_eig_vects,
                                                   evs_max_it,
                                                   restartIter_lanczos,
                                                   evs_tol,
                                                   kmean_max_it,
                                                   kmean_tol,
                                                   clust.raw(),
                                                   eigVals,
                                                   eigVecs,
                                                   iters_lanczos,
                                                   iters_kmeans);
                    }
                    // give a copy of results to the user
                    if (rc == NVGRAPH_OK)
                            {
                        CHECK_CUDA(hipMemcpy((int* )clustering,
                                                        clust.raw(),
                                                        (size_t )(MCSRG->get_num_vertices() * sizeof(int)),
                                                        hipMemcpyDefault));
                        CHECK_CUDA(hipMemcpy((float* )eig_vals,
                                                        eigVals.raw(),
                                                        (size_t )(n_eig_vects * sizeof(float)),
                                                        hipMemcpyDefault));
                        CHECK_CUDA(hipMemcpy((float* )eig_vects,
                                                        eigVecs.raw(),
                                                        (size_t )(n_eig_vects * MCSRG->get_num_vertices()
                                                                * sizeof(float)),
                                                        hipMemcpyDefault));
                    }

                    break;
                }
                case HIP_R_64F:
                    {
                    nvgraph::MultiValuedCsrGraph<int, double> *MCSRG =
                            static_cast<nvgraph::MultiValuedCsrGraph<int, double>*>(descrG->graph_handle);
                    if (weight_index >= MCSRG->get_num_edge_dim()
                            || n_clusters > static_cast<int>(MCSRG->get_num_vertices())) // base index is 0
                        return NVGRAPH_STATUS_INVALID_VALUE;
                    nvgraph::ValuedCsrGraph<int, double> network =
                            *MCSRG->get_valued_csr_graph(weight_index);
                    Vector<int> clust(MCSRG->get_num_vertices(), handle->stream);
                    Vector<double> eigVals(n_eig_vects, handle->stream);
                    Vector<double> eigVecs(MCSRG->get_num_vertices() * n_eig_vects, handle->stream);
                    if (evs_type == 0)
                            {
                        int restartIter_lanczos = 15 + n_eig_vects;
                        rc = partition<int, double>(network,
                                                    n_clusters,
                                                    n_eig_vects,
                                                    evs_max_it,
                                                    restartIter_lanczos,
                                                    evs_tol,
                                                    kmean_max_it,
                                                    kmean_tol,
                                                    clust.raw(),
                                                    eigVals,
                                                    eigVecs,
                                                    iters_lanczos,
                                                    iters_kmeans);
                    }
                    // give a copy of results to the user
                    if (rc == NVGRAPH_OK)
                            {
                        CHECK_CUDA(hipMemcpy((int* )clustering,
                                                        clust.raw(),
                                                        (size_t )(MCSRG->get_num_vertices() * sizeof(int)),
                                                        hipMemcpyDefault));
                        CHECK_CUDA(hipMemcpy((double* )eig_vals,
                                                        eigVals.raw(),
                                                        (size_t )(n_eig_vects * sizeof(double)),
                                                        hipMemcpyDefault));
                        CHECK_CUDA(hipMemcpy((double* )eig_vects,
                                                        eigVecs.raw(),
                                                        (size_t )(n_eig_vects * MCSRG->get_num_vertices()
                                                                * sizeof(double)),
                                                        hipMemcpyDefault));
                    }
                    break;
                }
                default:
                    return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;
            }
        }
        NVGRAPH_CATCHES(rc)
        return getCAPIStatusForError(rc);
    }

    nvgraphStatus_t NVGRAPH_API nvgraphAnalyzeBalancedCut_impl(nvgraphHandle_t handle,
                                                               const nvgraphGraphDescr_t descrG,
                                                               const size_t weight_index,
                                                               const int n_clusters,
                                                               const int* clustering,
                                                               float * edgeCut,
                                                               float * ratioCut) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle) || check_graph(descrG) || check_int_size(weight_index))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);

            if (descrG->graphStatus != HAS_VALUES) // need a MultiValuedCsrGraph
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (descrG->TT != NVGRAPH_CSR_32) // supported topologies
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (n_clusters < 2)
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (clustering == NULL || edgeCut == NULL || ratioCut == NULL)
                return NVGRAPH_STATUS_INVALID_VALUE;

            switch (descrG->T)
            {
                case HIP_R_32F:
                    {
                    float edge_cut, ratio_cut;
                    nvgraph::MultiValuedCsrGraph<int, float> *MCSRG =
                            static_cast<nvgraph::MultiValuedCsrGraph<int, float>*>(descrG->graph_handle);
                    if (weight_index >= MCSRG->get_num_edge_dim()
                            || n_clusters > static_cast<int>(MCSRG->get_num_vertices()))
                        return NVGRAPH_STATUS_INVALID_VALUE;
                    nvgraph::ValuedCsrGraph<int, float> network =
                            *MCSRG->get_valued_csr_graph(weight_index);
                    Vector<int> clust(MCSRG->get_num_vertices(), handle->stream);
                    CHECK_CUDA(hipMemcpy(clust.raw(),
                                                    (int* )clustering,
                                                    (size_t )(MCSRG->get_num_vertices() * sizeof(int)),
                                                    hipMemcpyDefault));
                    rc = analyzePartition<int, float>(network,
                                                      n_clusters,
                                                      clust.raw(),
                                                      edge_cut,
                                                      ratio_cut);
                    *edgeCut = edge_cut;
                    *ratioCut = ratio_cut;
                    break;
                }
                case HIP_R_64F:
                    {
                    double edge_cut, ratio_cut;
                    nvgraph::MultiValuedCsrGraph<int, double> *MCSRG =
                            static_cast<nvgraph::MultiValuedCsrGraph<int, double>*>(descrG->graph_handle);
                    if (weight_index >= MCSRG->get_num_edge_dim()
                            || n_clusters > static_cast<int>(MCSRG->get_num_vertices())) // base index is 0
                        return NVGRAPH_STATUS_INVALID_VALUE;
                    nvgraph::ValuedCsrGraph<int, double> network =
                            *MCSRG->get_valued_csr_graph(weight_index);
                    Vector<int> clust(MCSRG->get_num_vertices(), handle->stream);
                    CHECK_CUDA(hipMemcpy(clust.raw(),
                                                    (int* )clustering,
                                                    (size_t )(MCSRG->get_num_vertices() * sizeof(int)),
                                                    hipMemcpyDefault));
                    rc = analyzePartition<int, double>(network,
                                                       n_clusters,
                                                       clust.raw(),
                                                       edge_cut,
                                                       ratio_cut);
                    *edgeCut = static_cast<float>(edge_cut);
                    *ratioCut = static_cast<float>(ratio_cut);
                    break;
                }

                default:
                    return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;
            }
        }
        NVGRAPH_CATCHES(rc)
        return getCAPIStatusForError(rc);

    }

    nvgraphStatus_t NVGRAPH_API nvgraphSpectralModularityMaximization_impl(nvgraphHandle_t handle,
                                                                           const nvgraphGraphDescr_t descrG,
                                                                           const size_t weight_index,
                                                                           const int n_clusters,
                                                                           const int n_eig_vects,
                                                                           const float evs_tolerance,
                                                                           const int evs_max_iter,
                                                                           const float kmean_tolerance,
                                                                           const int kmean_max_iter,
                                                                           int* clustering,
                                                                           void* eig_vals,
                                                                           void* eig_vects) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle) || check_graph(descrG) || check_int_size(weight_index))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);

            if (descrG->graphStatus != HAS_VALUES) // need a MultiValuedCsrGraph
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (descrG->TT != NVGRAPH_CSR_32) // supported topologies
                return NVGRAPH_STATUS_GRAPH_TYPE_NOT_SUPPORTED;

            int evs_max_it, kmean_max_it;
            int iters_lanczos, iters_kmeans;
            float evs_tol, kmean_tol;

            if (evs_max_iter > 0)
                evs_max_it = evs_max_iter;
            else
                evs_max_it = 4000;

            if (evs_tolerance == 0.0f)
                evs_tol = 1.0E-3f;
            else if (evs_tolerance < 1.0f && evs_tolerance > 0.0f)
                evs_tol = evs_tolerance;
            else
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (kmean_max_iter > 0)
                kmean_max_it = kmean_max_iter;
            else
                kmean_max_it = 200;

            if (kmean_tolerance == 0.0f)
                kmean_tol = 1.0E-2f;
            else if (kmean_tolerance < 1.0f && kmean_tolerance > 0.0f)
                kmean_tol = kmean_tolerance;
            else
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (n_clusters < 2)
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (n_eig_vects > n_clusters)
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (clustering == NULL || eig_vals == NULL || eig_vects == NULL)
                return NVGRAPH_STATUS_INVALID_VALUE;

            switch (descrG->T)
            {
                case HIP_R_32F:
                    {
                    nvgraph::MultiValuedCsrGraph<int, float> *MCSRG =
                            static_cast<nvgraph::MultiValuedCsrGraph<int, float>*>(descrG->graph_handle);
                    if (weight_index >= MCSRG->get_num_edge_dim()
                            || n_clusters > static_cast<int>(MCSRG->get_num_vertices())) // base index is 0
                        return NVGRAPH_STATUS_INVALID_VALUE;
                    nvgraph::ValuedCsrGraph<int, float> network =
                            *MCSRG->get_valued_csr_graph(weight_index);
                    Vector<int> clust(MCSRG->get_num_vertices(), handle->stream);
                    Vector<float> eigVals(n_eig_vects, handle->stream);
                    Vector<float> eigVecs(MCSRG->get_num_vertices() * n_eig_vects, handle->stream);
                    int restartIter_lanczos = 15 + n_eig_vects;
                    rc = modularity_maximization<int, float>(network,
                                                             n_clusters,
                                                             n_eig_vects,
                                                             evs_max_it,
                                                             restartIter_lanczos,
                                                             evs_tol,
                                                             kmean_max_it,
                                                             kmean_tol,
                                                             clust.raw(),
                                                             eigVals,
                                                             eigVecs,
                                                             iters_lanczos,
                                                             iters_kmeans);

                    // give a copy of results to the user
                    if (rc == NVGRAPH_OK)
                            {
                        CHECK_CUDA(hipMemcpy((int* )clustering,
                                                        clust.raw(),
                                                        (size_t )(MCSRG->get_num_vertices() * sizeof(int)),
                                                        hipMemcpyDefault));
                        CHECK_CUDA(hipMemcpy((float* )eig_vals,
                                                        eigVals.raw(),
                                                        (size_t )(n_eig_vects * sizeof(float)),
                                                        hipMemcpyDefault));
                        CHECK_CUDA(hipMemcpy((float* )eig_vects,
                                                        eigVecs.raw(),
                                                        (size_t )(n_eig_vects * MCSRG->get_num_vertices()
                                                                * sizeof(float)),
                                                        hipMemcpyDefault));
                    }

                    break;
                }
                case HIP_R_64F:
                    {
                    nvgraph::MultiValuedCsrGraph<int, double> *MCSRG =
                            static_cast<nvgraph::MultiValuedCsrGraph<int, double>*>(descrG->graph_handle);
                    if (weight_index >= MCSRG->get_num_edge_dim()
                            || n_clusters > static_cast<int>(MCSRG->get_num_vertices())) // base index is 0
                        return NVGRAPH_STATUS_INVALID_VALUE;
                    nvgraph::ValuedCsrGraph<int, double> network =
                            *MCSRG->get_valued_csr_graph(weight_index);
                    Vector<int> clust(MCSRG->get_num_vertices(), handle->stream);
                    Vector<double> eigVals(n_eig_vects, handle->stream);
                    Vector<double> eigVecs(MCSRG->get_num_vertices() * n_eig_vects, handle->stream);
                    int restartIter_lanczos = 15 + n_eig_vects;
                    rc = modularity_maximization<int, double>(network,
                                                              n_clusters,
                                                              n_eig_vects,
                                                              evs_max_it,
                                                              restartIter_lanczos,
                                                              evs_tol,
                                                              kmean_max_it,
                                                              kmean_tol,
                                                              clust.raw(),
                                                              eigVals,
                                                              eigVecs,
                                                              iters_lanczos,
                                                              iters_kmeans);
                    // give a copy of results to the user
                    if (rc == NVGRAPH_OK)
                            {
                        CHECK_CUDA(hipMemcpy((int* )clustering,
                                                        clust.raw(),
                                                        (size_t )(MCSRG->get_num_vertices() * sizeof(int)),
                                                        hipMemcpyDefault));
                        CHECK_CUDA(hipMemcpy((double* )eig_vals,
                                                        eigVals.raw(),
                                                        (size_t )(n_eig_vects * sizeof(double)),
                                                        hipMemcpyDefault));
                        CHECK_CUDA(hipMemcpy((double* )eig_vects,
                                                        eigVecs.raw(),
                                                        (size_t )(n_eig_vects * MCSRG->get_num_vertices()
                                                                * sizeof(double)),
                                                        hipMemcpyDefault));
                    }
                    break;
                }
                default:
                    return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;
            }
        }
        NVGRAPH_CATCHES(rc)
        return getCAPIStatusForError(rc);
    }

    nvgraphStatus_t NVGRAPH_API nvgraphAnalyzeModularityClustering_impl(nvgraphHandle_t handle,
                                                                        const nvgraphGraphDescr_t descrG,
                                                                        const size_t weight_index,
                                                                        const int n_clusters,
                                                                        const int* clustering,
                                                                        float * modularity) {
        NVGRAPH_ERROR rc = NVGRAPH_OK;
        try
        {
            if (check_context(handle) || check_graph(descrG) || check_int_size(weight_index))
                FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);

            if (descrG->graphStatus != HAS_VALUES) // need a MultiValuedCsrGraph
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (descrG->TT != NVGRAPH_CSR_32) // supported topologies
                return NVGRAPH_STATUS_GRAPH_TYPE_NOT_SUPPORTED;

            if (n_clusters < 2)
                return NVGRAPH_STATUS_INVALID_VALUE;

            if (clustering == NULL || modularity == NULL)
                return NVGRAPH_STATUS_INVALID_VALUE;

            switch (descrG->T)
            {
                case HIP_R_32F:
                    {
                    float mod;
                    nvgraph::MultiValuedCsrGraph<int, float> *MCSRG =
                            static_cast<nvgraph::MultiValuedCsrGraph<int, float>*>(descrG->graph_handle);
                    if (weight_index >= MCSRG->get_num_edge_dim()
                            || n_clusters > static_cast<int>(MCSRG->get_num_vertices()))
                        return NVGRAPH_STATUS_INVALID_VALUE;
                    nvgraph::ValuedCsrGraph<int, float> network =
                            *MCSRG->get_valued_csr_graph(weight_index);
                    Vector<int> clust(MCSRG->get_num_vertices(), handle->stream);
                    CHECK_CUDA(hipMemcpy(clust.raw(),
                                                    (int* )clustering,
                                                    (size_t )(MCSRG->get_num_vertices() * sizeof(int)),
                                                    hipMemcpyDefault));
                    rc = analyzeModularity<int, float>(network,
                                                       n_clusters,
                                                       clust.raw(),
                                                       mod);
                    *modularity = mod;
                    break;
                }
                case HIP_R_64F:
                    {
                    double mod;
                    nvgraph::MultiValuedCsrGraph<int, double> *MCSRG =
                            static_cast<nvgraph::MultiValuedCsrGraph<int, double>*>(descrG->graph_handle);
                    if (weight_index >= MCSRG->get_num_edge_dim()
                            || n_clusters > static_cast<int>(MCSRG->get_num_vertices())) // base index is 0
                        return NVGRAPH_STATUS_INVALID_VALUE;
                    Vector<int> clust(MCSRG->get_num_vertices(), handle->stream);
                    CHECK_CUDA(hipMemcpy(clust.raw(),
                                                    (int* )clustering,
                                                    (size_t )(MCSRG->get_num_vertices() * sizeof(int)),
                                                    hipMemcpyDefault));
                    nvgraph::ValuedCsrGraph<int, double> network =
                            *MCSRG->get_valued_csr_graph(weight_index);
                    rc = analyzeModularity<int, double>(network,
                                                        n_clusters,
                                                        clust.raw(),
                                                        mod);
                    *modularity = static_cast<float>(mod);
                    break;
                }

                default:
                    return NVGRAPH_STATUS_TYPE_NOT_SUPPORTED;
            }
        }
        NVGRAPH_CATCHES(rc)
        return getCAPIStatusForError(rc);
    }

    nvgraphStatus_t NVGRAPH_API nvgraphSpectralClustering_impl(nvgraphHandle_t handle, // nvGRAPH library handle.
                                                               const nvgraphGraphDescr_t descrG, // nvGRAPH graph descriptor, should contain the connectivity information in NVGRAPH_CSR_32 or NVGRAPH_CSR_32 at least 1 edge set (weights)
                                                               const size_t weight_index, // Index of the edge set for the weights.
                                                               const struct SpectralClusteringParameter *params, //parameters, see struct SpectralClusteringParameter
                                                               int* clustering, // (output) clustering
                                                               void* eig_vals, // (output) eigenvalues
                                                               void* eig_vects) {// (output) eigenvectors
        if (check_ptr(params) || check_ptr(clustering) || check_ptr(eig_vals) || check_ptr(eig_vects))
            FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);
        if (params->algorithm == NVGRAPH_MODULARITY_MAXIMIZATION)
            return nvgraph::nvgraphSpectralModularityMaximization_impl(handle,
                                                                       descrG,
                                                                       weight_index,
                                                                       params->n_clusters,
                                                                       params->n_eig_vects,
                                                                       params->evs_tolerance,
                                                                       params->evs_max_iter,
                                                                       params->kmean_tolerance,
                                                                       params->kmean_max_iter,
                                                                       clustering,
                                                                       eig_vals,
                                                                       eig_vects);
        else if (params->algorithm == NVGRAPH_BALANCED_CUT_LANCZOS)
            return nvgraph::nvgraphBalancedCutClustering_impl(handle,
                                                              descrG,
                                                              weight_index,
                                                              params->n_clusters,
                                                              params->n_eig_vects,
                                                              0,
                                                              params->evs_tolerance,
                                                              params->evs_max_iter,
                                                              params->kmean_tolerance,
                                                              params->kmean_max_iter,
                                                              clustering,
                                                              eig_vals,
                                                              eig_vects);
        else
            return NVGRAPH_STATUS_INVALID_VALUE;
    }

    nvgraphStatus_t NVGRAPH_API nvgraphAnalyzeClustering_impl(nvgraphHandle_t handle, // nvGRAPH library handle.
                                                              const nvgraphGraphDescr_t descrG, // nvGRAPH graph descriptor, should contain the connectivity information in NVGRAPH_CSR_32 at least 1 edge set (weights)
                                                              const size_t weight_index, // Index of the edge set for the weights.
                                                              const int n_clusters, //number of clusters
                                                              const int* clustering, // clustering to analyse
                                                              nvgraphClusteringMetric_t metric, // metric to compute to measure the clustering quality
                                                              float * score) {// (output) clustering score telling how good the clustering is for the selected metric.
        if (check_ptr(clustering) || check_ptr(score))
            FatalError("Incorrect parameters.", NVGRAPH_ERR_BAD_PARAMETERS);
        if (metric == NVGRAPH_MODULARITY)
            return nvgraphAnalyzeModularityClustering_impl(handle,
                                                           descrG,
                                                           weight_index,
                                                           n_clusters,
                                                           clustering,
                                                           score);
        else if (metric == NVGRAPH_EDGE_CUT) {
            float dummy = 0;
            return nvgraph::nvgraphAnalyzeBalancedCut_impl(handle,
                                                           descrG,
                                                           weight_index,
                                                           n_clusters,
                                                           clustering,
                                                           score,
                                                           &dummy);
        }
        else if (metric == NVGRAPH_RATIO_CUT) {
            float dummy = 0;
            return nvgraph::nvgraphAnalyzeBalancedCut_impl(handle,
                                                           descrG,
                                                           weight_index,
                                                           n_clusters,
                                                           clustering,
                                                           &dummy,
                                                           score);
        }
        else
            return NVGRAPH_STATUS_INVALID_VALUE;
    }
} /*namespace nvgraph*/

/*************************
 *        API
 *************************/

nvgraphStatus_t NVGRAPH_API nvgraphGetProperty(libraryPropertyType type, int *value) {
    switch (type) {
        case MAJOR_VERSION:
            *value = CUDART_VERSION / 1000;
            break;
        case MINOR_VERSION:
            *value = (CUDART_VERSION % 1000) / 10;
            break;
        case PATCH_LEVEL:
            *value = 0;
            break;
        default:
            return NVGRAPH_STATUS_INVALID_VALUE;
    }
    return NVGRAPH_STATUS_SUCCESS;
}

nvgraphStatus_t NVGRAPH_API nvgraphCreate(nvgraphHandle_t *handle) {
    return nvgraph::nvgraphCreate_impl(handle);
}

nvgraphStatus_t NVGRAPH_API nvgraphDestroy(nvgraphHandle_t handle) {
    return nvgraph::nvgraphDestroy_impl(handle);
}

nvgraphStatus_t NVGRAPH_API nvgraphCreateGraphDescr(nvgraphHandle_t handle,
                                                    nvgraphGraphDescr_t *descrG) {
    return nvgraph::nvgraphCreateGraphDescr_impl(handle, descrG);
}

nvgraphStatus_t NVGRAPH_API nvgraphDestroyGraphDescr(nvgraphHandle_t handle,
                                                     nvgraphGraphDescr_t descrG) {
    return nvgraph::nvgraphDestroyGraphDescr_impl(handle, descrG);
}

nvgraphStatus_t NVGRAPH_API nvgraphSetStream(nvgraphHandle_t handle, hipStream_t stream) {
    return nvgraph::nvgraphSetStream_impl(handle, stream);
}

nvgraphStatus_t NVGRAPH_API nvgraphSetGraphStructure(nvgraphHandle_t handle,
                                                     nvgraphGraphDescr_t descrG,
                                                     void* topologyData,
                                                     nvgraphTopologyType_t topologyType) {
    return nvgraph::nvgraphSetGraphStructure_impl(handle, descrG, topologyData, topologyType);
}

nvgraphStatus_t NVGRAPH_API nvgraphGetGraphStructure(nvgraphHandle_t handle,
                                                     nvgraphGraphDescr_t descrG,
                                                     void* topologyData,
                                                     nvgraphTopologyType_t* topologyType) {
    return nvgraph::nvgraphGetGraphStructure_impl(handle, descrG, topologyData, topologyType);
}
nvgraphStatus_t NVGRAPH_API nvgraphAllocateVertexData(nvgraphHandle_t handle,
                                                      nvgraphGraphDescr_t descrG,
                                                      size_t numsets,
                                                      hipDataType *settypes) {
    return nvgraph::nvgraphAllocateVertexData_impl(handle, descrG, numsets, settypes);
}

nvgraphStatus_t NVGRAPH_API nvgraphAllocateEdgeData(nvgraphHandle_t handle,
                                                    nvgraphGraphDescr_t descrG,
                                                    size_t numsets,
                                                    hipDataType *settypes) {
    return nvgraph::nvgraphAllocateEdgeData_impl(handle, descrG, numsets, settypes);
}

nvgraphStatus_t NVGRAPH_API nvgraphSetVertexData(nvgraphHandle_t handle,
                                                 nvgraphGraphDescr_t descrG,
                                                 void *vertexData,
                                                 size_t setnum) {
    return nvgraph::nvgraphSetVertexData_impl(handle, descrG, vertexData, setnum);
}

nvgraphStatus_t NVGRAPH_API nvgraphGetVertexData(nvgraphHandle_t handle,
                                                 nvgraphGraphDescr_t descrG,
                                                 void *vertexData,
                                                 size_t setnum) {
    return nvgraph::nvgraphGetVertexData_impl(handle, descrG, vertexData, setnum);
}

nvgraphStatus_t NVGRAPH_API nvgraphSetEdgeData(nvgraphHandle_t handle,
                                               nvgraphGraphDescr_t descrG,
                                               void *edgeData,
                                               size_t setnum) {
    return nvgraph::nvgraphSetEdgeData_impl(handle, descrG, edgeData, setnum);
}

nvgraphStatus_t NVGRAPH_API nvgraphGetEdgeData(nvgraphHandle_t handle,
                                               nvgraphGraphDescr_t descrG,
                                               void *edgeData,
                                               size_t setnum) {
    return nvgraph::nvgraphGetEdgeData_impl(handle, descrG, edgeData, setnum);
}

nvgraphStatus_t NVGRAPH_API nvgraphBalancedCutClustering(nvgraphHandle_t handle,
                                                         const nvgraphGraphDescr_t descrG,
                                                         const size_t weight_index,
                                                         const int n_clusters,
                                                         const int n_eig_vects,
                                                         const int evs_type,
                                                         const float evs_tolerance,
                                                         const int evs_max_iter,
                                                         const float kmean_tolerance,
                                                         const int kmean_max_iter,
                                                         int* clustering,
                                                         void* eig_vals,
                                                         void* eig_vects) {
    return nvgraph::nvgraphBalancedCutClustering_impl(handle,
                                                      descrG,
                                                      weight_index,
                                                      n_clusters,
                                                      n_eig_vects,
                                                      evs_type,
                                                      evs_tolerance,
                                                      evs_max_iter,
                                                      kmean_tolerance,
                                                      kmean_max_iter,
                                                      clustering,
                                                      eig_vals,
                                                      eig_vects);
}

nvgraphStatus_t NVGRAPH_API nvgraphAnalyzeBalancedCut(nvgraphHandle_t handle,
                                                      const nvgraphGraphDescr_t descrG,
                                                      const size_t weight_index,
                                                      const int n_clusters,
                                                      const int* clustering,
                                                      float * edgeCut,
                                                      float * ratioCut) {
    return nvgraph::nvgraphAnalyzeBalancedCut_impl(handle,
                                                   descrG,
                                                   weight_index,
                                                   n_clusters,
                                                   clustering,
                                                   edgeCut,
                                                   ratioCut);
}

nvgraphStatus_t NVGRAPH_API nvgraphSpectralModularityMaximization(nvgraphHandle_t handle,
                                                                  const nvgraphGraphDescr_t descrG,
                                                                  const size_t weight_index,
                                                                  const int n_clusters,
                                                                  const int n_eig_vects,
                                                                  const float evs_tolerance,
                                                                  const int evs_max_iter,
                                                                  const float kmean_tolerance,
                                                                  const int kmean_max_iter,
                                                                  int* clustering,
                                                                  void* eig_vals,
                                                                  void* eig_vects) {
    return nvgraph::nvgraphSpectralModularityMaximization_impl(handle,
                                                               descrG,
                                                               weight_index,
                                                               n_clusters,
                                                               n_eig_vects,
                                                               evs_tolerance,
                                                               evs_max_iter,
                                                               kmean_tolerance,
                                                               kmean_max_iter,
                                                               clustering,
                                                               eig_vals,
                                                               eig_vects);
}

nvgraphStatus_t NVGRAPH_API nvgraphAnalyzeModularityClustering(nvgraphHandle_t handle,
                                                               const nvgraphGraphDescr_t descrG,
                                                               const size_t weight_index,
                                                               const int n_clusters,
                                                               const int* clustering,
                                                               float * modularity) {
    return nvgraph::nvgraphAnalyzeModularityClustering_impl(handle,
                                                            descrG,
                                                            weight_index,
                                                            n_clusters,
                                                            clustering,
                                                            modularity);
}

nvgraphStatus_t NVGRAPH_API nvgraphSpectralClustering(nvgraphHandle_t handle, // nvGRAPH library handle.
                                                      const nvgraphGraphDescr_t descrG, // nvGRAPH graph descriptor, should contain the connectivity information in NVGRAPH_CSR_32 or NVGRAPH_CSR_32 at least 1 edge set (weights)
                                                      const size_t weight_index, // Index of the edge set for the weights.
                                                      const struct SpectralClusteringParameter *params, //parameters, see struct SpectralClusteringParameter
                                                      int* clustering, // (output) clustering
                                                      void* eig_vals,   // (output) eigenvalues
                                                      void* eig_vects)  // (output) eigenvectors
{
    return nvgraph::nvgraphSpectralClustering_impl(handle,
                                                   descrG,
                                                   weight_index,
                                                   params,
                                                   clustering,
                                                   eig_vals,
                                                   eig_vects);
}

nvgraphStatus_t NVGRAPH_API nvgraphAnalyzeClustering(nvgraphHandle_t handle, // nvGRAPH library handle.
                                                     const nvgraphGraphDescr_t descrG, // nvGRAPH graph descriptor, should contain the connectivity information in NVGRAPH_CSR_32 at least 1 edge set (weights)
                                                     const size_t weight_index, // Index of the edge set for the weights.
                                                     const int n_clusters, //number of clusters
                                                     const int* clustering, // clustering to analyse
                                                     nvgraphClusteringMetric_t metric, // metric to compute to measure the clustering quality
                                                     float * score) // (output) clustering score telling how good the clustering is for the selected metric.
{
    return nvgraph::nvgraphAnalyzeClustering_impl(handle,
                                                  descrG,
                                                  weight_index,
                                                  n_clusters,
                                                  clustering,
                                                  metric,
                                                  score);
}

nvgraphStatus_t NVGRAPH_API nvgraphLouvain (hipDataType index_type, hipDataType val_type, const size_t num_vertex, const size_t num_edges,
                            void* csr_ptr, void* csr_ind, void* csr_val, int weighted, int has_init_cluster, void* init_cluster,
                            void* final_modularity, void* best_cluster_vec, void* num_level, int max_iter)
{
    NVLOUVAIN_STATUS status = NVLOUVAIN_OK;
    if ((csr_ptr == NULL) || (csr_ind == NULL) || ((csr_val == NULL) && (weighted == 1)) ||
        ((init_cluster == NULL) && (has_init_cluster == 1)) || (final_modularity == NULL) || (best_cluster_vec == NULL) || (num_level == NULL))
       return NVGRAPH_STATUS_INVALID_VALUE;

    std::ostream log(0);
    bool weighted_b = weighted;
    bool has_init_cluster_b = has_init_cluster;
    if (val_type == HIP_R_32F)
        status = nvlouvain::louvain ((int*)csr_ptr, (int*)csr_ind, (float*)csr_val, num_vertex, num_edges,
               weighted_b, has_init_cluster_b, (int*)init_cluster, *((float*)final_modularity),
              (int*)best_cluster_vec,*((int*)num_level), max_iter, log);
    else
        status = nvlouvain::louvain ((int*)csr_ptr, (int*)csr_ind, (double*)csr_val, num_vertex, num_edges,
                weighted_b, has_init_cluster_b, (int*)init_cluster, *((double*)final_modularity),
                (int*)best_cluster_vec,*((int*)num_level), max_iter, log);

    if (status != NVLOUVAIN_OK)
        return NVGRAPH_STATUS_INTERNAL_ERROR;

    return NVGRAPH_STATUS_SUCCESS;
}

nvgraphStatus_t NVGRAPH_API nvgraphAttachGraphStructure(nvgraphHandle_t handle,
                                                        nvgraphGraphDescr_t descrG,
                                                        void* topologyData,
                                                        nvgraphTopologyType_t TT) {
    return nvgraph::nvgraphAttachGraphStructure_impl( handle, descrG, topologyData, TT);
}

nvgraphStatus_t NVGRAPH_API nvgraphAttachEdgeData(nvgraphHandle_t handle,
                                                  nvgraphGraphDescr_t descrG,
                                                  size_t setnum,
                                                  hipDataType settype,
                                                  void *edgeData) {
    return nvgraph::nvgraphAttachEdgeData_impl( handle, descrG, setnum, settype, edgeData);
}
