/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// snmg spmv
// Author: Alex Fender afender@nvidia.com

#include "spmv_1D.cuh"

namespace cugraph {
namespace opg {
template <typename VT, typename ET, typename WT>
OPGcsrmv<VT, ET, WT>::OPGcsrmv(const raft::comms::comms_t &comm_,
                               VT *local_vertices_,
                               VT *part_off_,
                               ET *off_,
                               VT *ind_,
                               WT *val_,
                               WT *x)
  : comm(comm_),
    local_vertices(local_vertices_),
    part_off(part_off_),
    off(off_),
    ind(ind_),
    val(val_)
{
  stream = nullptr;
  i      = comm.get_rank();
  p      = comm.get_size();
  v_glob = part_off[p];
  v_loc  = part_off[i + 1] - part_off[i];
  VT tmp;
  hipMemcpy(&tmp, &off[v_loc], sizeof(VT), hipMemcpyDeviceToHost);
  CUDA_CHECK_LAST();
  e_loc = tmp;
  y_loc.resize(v_loc);
  WT h_one  = 1.0;
  WT h_zero = 0.0;

  spmv.setup(v_loc, v_glob, e_loc, &h_one, val, off, ind, x, &h_zero, y_loc.data().get());
}

template <typename VT, typename ET, typename WT>
OPGcsrmv<VT, ET, WT>::~OPGcsrmv()
{
}

template <typename VT, typename ET, typename WT>
void OPGcsrmv<VT, ET, WT>::run(WT *x)
{
  WT h_one  = 1.0;
  WT h_zero = 0.0;
  spmv.run(v_loc, v_glob, e_loc, &h_one, val, off, ind, x, &h_zero, y_loc.data().get());
  // FIXME https://github.com/rapidsai/raft/issues/21
  size_t recvbuf[comm.get_size()];
  for (int i = 0; i < comm.get_size(); i++) recvbuf[i] = local_vertices[i];
  comm.allgatherv(y_loc.data().get(), x, recvbuf, part_off, stream);
}

template class OPGcsrmv<int, int, double>;
template class OPGcsrmv<int, int, float>;

}  // namespace opg
}  // namespace cugraph
