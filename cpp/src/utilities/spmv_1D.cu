/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <raft/spectral/matrix_wrappers.hpp>
#include "spmv_1D.cuh"

namespace cugraph {
namespace opg {
template <typename VT, typename ET, typename WT>
OPGcsrmv<VT, ET, WT>::OPGcsrmv(const raft::comms::comms_t &comm_,
                               VT *local_vertices_,
                               VT *part_off_,
                               ET *off_,
                               VT *ind_,
                               WT *val_,
                               WT *x)
  : comm(comm_),
    local_vertices(local_vertices_),
    part_off(part_off_),
    off(off_),
    ind(ind_),
    val(val_)
{
  stream = nullptr;
  i      = comm.get_rank();
  p      = comm.get_size();
  v_glob = part_off[p - 1] + local_vertices[p - 1];
  v_loc  = local_vertices[i];
  VT tmp;
  CUDA_TRY(hipMemcpy(&tmp, &off[v_loc], sizeof(VT), hipMemcpyDeviceToHost));
  e_loc = tmp;
  y_loc.resize(v_loc);
}

template <typename VT, typename ET, typename WT>
OPGcsrmv<VT, ET, WT>::~OPGcsrmv()
{
}

template <typename VT, typename ET, typename WT>
void OPGcsrmv<VT, ET, WT>::run(WT *x)
{
  using namespace raft::matrix;

  WT h_one  = 1.0;
  WT h_zero = 0.0;

  {
    raft::handle_t handle;

    sparse_matrix_t<VT, WT> mat{handle,
                                off,                      // CSR row_offsets
                                ind,                      // CSR col_indices
                                val,                      // CSR values
                                static_cast<VT>(v_loc),   // n_rows
                                static_cast<VT>(v_glob),  // n_cols
                                static_cast<VT>(e_loc)};  // nnz

    mat.mv(h_one,                             // alpha
           x,                                 // x
           h_zero,                            // beta
           y_loc.data().get(),                // y
           sparse_mv_alg_t::SPARSE_MV_ALG2);  // SpMV algorithm
  }

  std::vector<size_t> recvbuf(comm.get_size());
  std::copy(local_vertices, local_vertices + comm.get_size(), recvbuf.begin());
  comm.allgatherv(y_loc.data().get(), x, recvbuf.data(), part_off, stream);
}

template class OPGcsrmv<int, int, double>;
template class OPGcsrmv<int, int, float>;

}  // namespace opg
}  // namespace cugraph
