#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// opg 1D pagerank
// Author: Alex Fender afender@nvidia.com

#include <graph.hpp>
#include "pagerank_1D.cuh"

namespace cugraph {
namespace opg {

template <typename VT, typename ET, typename WT>
__global__ void __launch_bounds__(CUDA_MAX_KERNEL_THREADS)
  transition_kernel(const size_t e, const VT *ind, const VT *degree, WT *val)
{
  for (auto i = threadIdx.x + blockIdx.x * blockDim.x; i < e; i += gridDim.x * blockDim.x)
    val[i] = 1.0 / degree[ind[i]];
}

template <typename VT, typename ET, typename WT>
Pagerank<VT, WT>::Pagerank(
  const comms::comms_t &comm_, size_t *part_off_, ET *off_, VT *ind_, hipStream_t stream_)
  : comm(comm_), part_off(part_off_), off(off_), ind(ind_), stream(stream_)
{
  id     = comm->get_rank();
  nt     = comm->get_size();
  v_glob = part_off[nt];
  v_loc  = part_off[id + 1] - part_off[id];
  VT tmp_e;
  hipMemcpy(&tmp_e, &off[v_loc], sizeof(VT), hipMemcpyDeviceToHost);
  CUDA_CHECK_LAST();
  e_loc    = tmp_e;
  is_setup = false;
  bookmark.resize(v_glob);
  val.resize(e_loc);

  // intialize cusparse. This can take some time.
  cugraph::detail::Cusparse::get_handle();
}

template <typename VT, typename ET, typename WT>
Pagerank<VT, ET, WT>::~Pagerank()
{
  // cugraph::detail::Cusparse::destroy_handle();
}

template <typename VT, typename ET, typename WT>
void Pagerank<VT, ET, WT>::transition_vals(const VT *degree)
{
  int threads = min(static_cast<VT>(e_loc), 256);
  int blocks  = min(static_cast<VT>(32 * comm->get_sm_count()), CUDA_MAX_BLOCKS);
  transition_kernel<VT, WT><<<blocks, threads>>>(e_loc, ind, degree, val);
  CUDA_CHECK_LAST();
}

template <typename VT, typename ET, typename WT>
void Pagerank<VT, ET, WT>::flag_leafs(const VT *degree)
{
  int threads = min(static_cast<VT>(v_glob), 256);
  int blocks  = min(static_cast<VT>(32 * comm->get_sm_count()), CUDA_MAX_BLOCKS);
  cugraph::detail::flag_leafs_kernel<VT, WT><<<blocks, threads>>>(v_glob, degree, bookmark);
  CUDA_CHECK_LAST();
}

// Artificially create the google matrix by setting val and bookmark
template <typename VT, typename ET, typename WT>
void Pagerank<VT, ET, WT>::setup(WT _alpha, VT *degree)
{
  if (!is_setup) {
    alpha   = _alpha;
    WT zero = 0.0;

    // Update dangling node vector
    cugraph::detail::fill(v_glob, bookmark, zero);
    flag_leafs(degree);
    cugraph::detail::update_dangling_nodes(v_glob, bookmark, alpha);

    // Transition matrix
    transition_vals(degree);

    is_setup = true;
  } else
    CUGRAPH_FAIL("OPG PageRank : Setup can be called only once");
}

// run the power iteration on the google matrix
template <typename VT, typename ET, typename WT>
void Pagerank<VT, ET, WT>::solve(int max_iter, WT *pagerank)
{
  if (is_setup) {
    WT dot_res;
    WT one = 1.0;
    WT *pr = pagerank;
    cugraph::detail::fill(v_glob, pagerank, one / v_glob);
    // This cuda sync was added to fix #426
    // This should not be requiered in theory
    // This is not needed on one GPU at this time
    hipDeviceSynchronize();
    dot_res = cugraph::detail::dot(v_glob, bookmark, pr);
    OPGcsrmv<VT, ET, WT> spmv_solver(comm, part_off, off, ind, val, pagerank);
    for (auto i = 0; i < max_iter; ++i) {
      spmv_solver.run(pagerank);
      cugraph::detail::scal(v_glob, alpha, pr);
      cugraph::detail::addv(v_glob, dot_res * (one / v_glob), pr);
      dot_res = cugraph::detail::dot(v_glob, bookmark, pr);
      cugraph::detail::scal(v_glob, one / cugraph::detail::nrm2(v_glob, pr), pr);
    }
    cugraph::detail::scal(v_glob, one / cugraph::detail::nrm1(v_glob, pr), pr);
  } else {
    CUGRAPH_FAIL("OPG PageRank : Solve was called before setup");
  }
}

template class Pagerank<int, int, double>;
template class Pagerank<int, int, float>;

}  // namespace opg
}  // namespace cugraph
