 /*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

#include <cugraph.h>
#include "utilities/graph_utils.cuh"
#include "utilities/error_utils.h"
#include "hipcub/hipcub.hpp"
#include <rmm_utils.h>

namespace cugraph
{
  
#ifdef DEBUG
  #define PR_VERBOSE
#endif
template <typename IndexType, typename ValueType>
bool  pagerankIteration( IndexType n, IndexType e, IndexType *cscPtr, IndexType *cscInd,ValueType *cscVal,
                                     ValueType alpha, ValueType *a, ValueType *b, float tolerance, int iter, int max_iter, 
                                     ValueType * &tmp,  void* cub_d_temp_storage, size_t  cub_temp_storage_bytes, 
                                     ValueType * &pr, ValueType *residual) {
    
    ValueType  dot_res;
    hipcub::DeviceSpmv::CsrMV(cub_d_temp_storage, cub_temp_storage_bytes, cscVal,
        cscPtr, cscInd, tmp, pr,
        n, n, e);
   
    scal(n, alpha, pr);
    dot_res = dot( n, a, tmp);
    axpy(n, dot_res,  b,  pr);
    scal(n, (ValueType)1.0/nrm2(n, pr) , pr);
    axpy(n, (ValueType)-1.0,  pr,  tmp);
    *residual = nrm2(n, tmp);
    if (*residual < tolerance)
    {
        scal(n, (ValueType)1.0/nrm1(n,pr), pr);
        return true;
    }
    else
    {
        if (iter< max_iter)
        {
            std::swap(pr, tmp);
        }
        else
        {
           scal(n, (ValueType)1.0/nrm1(n,pr), pr);
        }
        return false;
    }
}

template <typename IndexType, typename ValueType>
int pagerank (  IndexType n, IndexType e, IndexType *cscPtr, IndexType *cscInd, ValueType *cscVal,
                       ValueType alpha, ValueType *a, bool has_guess, float tolerance, int max_iter, 
                       ValueType * &pagerank_vector, ValueType * &residual) {
  int max_it, i = 0 ;
  float tol;
  bool converged = false;
  ValueType randomProbability =  static_cast<ValueType>( 1.0/n);
  ValueType *b=0, *tmp=0;
  void*    cub_d_temp_storage = NULL;
  size_t   cub_temp_storage_bytes = 0;

  if (max_iter > 0 )
      max_it = max_iter;
  else
      max_it =  500;

  if (tolerance == 0.0f)
      tol =  1.0E-6f;
  else if (tolerance < 1.0f && tolerance > 0.0f)
      tol = tolerance;
  else
      return -1;

  if (alpha <= 0.0f || alpha >= 1.0f)
          return -1;

  hipStream_t stream{nullptr};
	
  ALLOC_MANAGED_TRY ((void**)&b,    sizeof(ValueType) * n, stream);
  ALLOC_MANAGED_TRY ((void**)&tmp,    sizeof(ValueType) * n, stream);
  cudaCheckError();

  if (!has_guess)  {
       fill(n, pagerank_vector, randomProbability);
       fill(n, tmp, randomProbability);
  }
  else {
    copy(n, pagerank_vector, tmp);
  }


  fill(n, b, randomProbability);
  update_dangling_nodes(n, a, alpha);

  hipcub::DeviceSpmv::CsrMV(cub_d_temp_storage, cub_temp_storage_bytes, cscVal,
                                             cscPtr, cscInd, tmp, pagerank_vector, n, n, e);
   // Allocate temporary storage
 ALLOC_MANAGED_TRY ((void**)&cub_d_temp_storage, cub_temp_storage_bytes, stream);
 cudaCheckError()
  #ifdef PR_VERBOSE
      std::stringstream ss;
      ss.str(std::string());
      ss <<" ------------------PageRank------------------"<< std::endl;
      ss <<" --------------------------------------------"<< std::endl;
      ss << std::setw(10) << "Iteration" << std::setw(15) << "Residual" << std::endl;
      ss <<" --------------------------------------------"<< std::endl;
      std::cout<<ss.str();
  #endif

  while (!converged && i < max_it)
  { 
      i++;
      converged = pagerankIteration(n, e, cscPtr, cscInd, cscVal,
                                           alpha, a, b, tol, i, max_it, tmp, 
                                           cub_d_temp_storage, cub_temp_storage_bytes, 
                                           pagerank_vector, residual);
       #ifdef PR_VERBOSE
          ss.str(std::string());
          ss << std::setw(10) << i ;
          ss.precision(3);
          ss << std::setw(15) << std::scientific << *residual  << std::endl;
          std::cout<<ss.str();
      #endif
  }
  #ifdef PR_VERBOSE
      std::cout <<" --------------------------------------------"<< std::endl;
  #endif
  //printv(n,pagerank_vector,0);

  ALLOC_FREE_TRY(b, stream);  
  ALLOC_FREE_TRY(tmp, stream);
  ALLOC_FREE_TRY(cub_d_temp_storage, stream);    
  
  return converged ? 0 : 1;
}

//template int pagerank<int, half> (  int n, int e, int *cscPtr, int *cscInd,half *cscVal, half alpha, half *a, bool has_guess, float tolerance, int max_iter, half * &pagerank_vector, half * &residual);
template int pagerank<int, float> (  int n, int e, int *cscPtr, int *cscInd,float *cscVal, float alpha, float *a, bool has_guess, float tolerance, int max_iter, float * &pagerank_vector, float * &residual);
template int pagerank<int, double> (  int n, int e, int *cscPtr, int *cscInd,double *cscVal, double alpha, double *a, bool has_guess, float tolerance, int max_iter, double * &pagerank_vector, double * &residual);

} //namespace cugraph

template <typename WT>
gdf_error gdf_pagerank_impl (gdf_graph *graph,
                      gdf_column *pagerank, float alpha = 0.85,
                      float tolerance = 1e-4, int max_iter = 200,
                      bool has_guess = false) {
  GDF_REQUIRE( graph->edgeList != nullptr, GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( graph->edgeList->src_indices->size == graph->edgeList->dest_indices->size, GDF_COLUMN_SIZE_MISMATCH );
  GDF_REQUIRE( graph->edgeList->src_indices->dtype == graph->edgeList->dest_indices->dtype, GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( graph->edgeList->src_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( graph->edgeList->dest_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( pagerank != nullptr , GDF_INVALID_API_CALL );
  GDF_REQUIRE( pagerank->data != nullptr , GDF_INVALID_API_CALL );
  GDF_REQUIRE( pagerank->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( pagerank->size > 0 , GDF_INVALID_API_CALL );

  int m=pagerank->size, nnz = graph->edgeList->src_indices->size, status = 0;
  WT *d_pr, *d_val = nullptr, *d_leaf_vector = nullptr;
  WT res = 1.0;
  WT *residual = &res;

  if (graph->transposedAdjList == nullptr) {
    gdf_add_transposed_adj_list(graph);
  }
  hipStream_t stream{nullptr};
  ALLOC_MANAGED_TRY((void**)&d_leaf_vector, sizeof(WT) * m, stream);
  ALLOC_MANAGED_TRY((void**)&d_val, sizeof(WT) * nnz , stream);
  ALLOC_MANAGED_TRY((void**)&d_pr,    sizeof(WT) * m, stream);

  //  The templating for HT_matrix_csc_coo assumes that m, nnz and data are all the same type
  cugraph::HT_matrix_csc_coo(m, nnz, (int *)graph->transposedAdjList->offsets->data, (int *)graph->transposedAdjList->indices->data, d_val, d_leaf_vector);

  if (has_guess)
  {
    GDF_REQUIRE( pagerank->data != nullptr, GDF_VALIDITY_UNSUPPORTED );
    cugraph::copy<WT>(m, (WT*)pagerank->data, d_pr);
  }

  status = cugraph::pagerank<int32_t,WT>( m,nnz, (int*)graph->transposedAdjList->offsets->data, (int*)graph->transposedAdjList->indices->data,
    d_val, alpha, d_leaf_vector, false, tolerance, max_iter, d_pr, residual);

  if (status !=0)
    switch ( status ) {
      case -1: std::cerr<< "Error : bad parameters in Pagerank"<<std::endl; return GDF_CUDA_ERROR;
      case 1: std::cerr<< "Warning : Pagerank did not reached the desired tolerance"<<std::endl;  return GDF_CUDA_ERROR;
      default:  std::cerr<< "Pagerank failed"<<std::endl;  return GDF_CUDA_ERROR;
    }

  cugraph::copy<WT>(m, d_pr, (WT*)pagerank->data);

  ALLOC_FREE_TRY(d_val, stream);
  ALLOC_FREE_TRY(d_pr, stream);
  ALLOC_FREE_TRY(d_leaf_vector, stream);

  return GDF_SUCCESS;
}

gdf_error gdf_pagerank(gdf_graph *graph, gdf_column *pagerank, float alpha, float tolerance, int max_iter, bool has_guess) {
  //
  //  page rank operates on CSR and can't currently support 64-bit integers.
  //
  //  If csr doesn't exist, create it.  Then check type to make sure it is 32-bit.
  //
  GDF_REQUIRE(graph->adjList != nullptr || graph->edgeList != nullptr, GDF_INVALID_API_CALL);
  gdf_error err = gdf_add_adj_list(graph);
  if (err != GDF_SUCCESS)
    return err;

  GDF_REQUIRE(graph->adjList->offsets->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(graph->adjList->indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);

  switch (pagerank->dtype) {
    case GDF_FLOAT32:   return gdf_pagerank_impl<float>(graph, pagerank, alpha, tolerance, max_iter, has_guess);
    case GDF_FLOAT64:   return gdf_pagerank_impl<double>(graph, pagerank, alpha, tolerance, max_iter, has_guess);
    default: return GDF_UNSUPPORTED_DTYPE;
  }
}
