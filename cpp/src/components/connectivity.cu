#include "hip/hip_runtime.h"
#include "weak_cc.cuh"
#include "scc_matrix.cuh"

#include <thrust/sequence.h>

#include "utilities/graph_utils.cuh"
#include "utilities/error_utils.h"
#include <cugraph.h>
#include <algo_types.h>

#include <iostream>
#include <type_traits>
#include <cstdint>

#include "topology/topology.cuh"

//#define _DEBUG_WEAK_CC

//
/**
 * @brief Compute connected components. 
 * The weak version (for undirected graphs, only) was imported from cuML.
 * This implementation comes from [1] and solves component labeling problem in
 * parallel on CSR-indexes based upon the vertex degree and adjacency graph.
 *
 * [1] Hawick, K.A et al, 2010. "Parallel graph component labelling with GPUs and CUDA"
 * 
 * The strong version (for directed or undirected graphs) is based on: 
 * [2] Gilbert, J. et al, 2011. "Graph Algorithms in the Language of Linear Algebra"
 *
 * C = I | A | A^2 |...| A^k
 * where matrix multiplication is via semi-ring: 
 * (combine, reduce) == (&, |) (bitwise ops)
 * Then: X = C & transpose(C); and finally, apply get_labels(X);
 *
 *
 * @tparam IndexT the numeric type of non-floating point elements
 * @tparam TPB_X the threads to use per block when configuring the kernel
 * @param graph input graph; assumed undirected for weakly CC [in]
 * @param table of 2 gdf_columns: output labels and vertex indices [out]
 * @param connectivity_type CUGRAPH_WEAK or CUGRAPH_STRONG [in]
 * @param stream the cuda stream [in]
 */
template<typename IndexT,
         int TPB_X = 32>
std::enable_if_t<std::is_signed<IndexT>::value,gdf_error>
gdf_connected_components_impl(gdf_graph *graph,
                              cudf::table *table,
                              cugraph_cc_t connectivity_type,
                              hipStream_t stream)
{
  using ByteT = unsigned char;//minimum addressable unit
  
  static auto row_offsets_ = [](const gdf_graph* G){
    return static_cast<const IndexT*>(G->adjList->offsets->data);
  };

  static auto col_indices_ = [](const gdf_graph* G){
    return static_cast<const IndexT*>(G->adjList->indices->data);
  };

  static auto nrows_ = [](const gdf_graph* G){
    return G->adjList->offsets->size - 1;
  };

  static auto nnz_ = [](const gdf_graph* G){
    return G->adjList->indices->size;
  };
  
  gdf_column* labels = table->get_column(0);
  gdf_column* verts = table->get_column(1);

  GDF_REQUIRE(graph != nullptr, GDF_INVALID_API_CALL);
    
  GDF_REQUIRE(graph->adjList != nullptr, GDF_INVALID_API_CALL);
    
  GDF_REQUIRE(row_offsets_(graph) != nullptr, GDF_INVALID_API_CALL);

  GDF_REQUIRE(col_indices_(graph) != nullptr, GDF_INVALID_API_CALL);
  
  GDF_REQUIRE(labels->data != nullptr, GDF_INVALID_API_CALL);

  GDF_REQUIRE(verts->data != nullptr, GDF_INVALID_API_CALL);
  
  auto type_id = graph->adjList->offsets->dtype;
  GDF_REQUIRE( type_id == GDF_INT32 || type_id == GDF_INT64, GDF_UNSUPPORTED_DTYPE);
  
  GDF_REQUIRE( type_id == graph->adjList->indices->dtype, GDF_UNSUPPORTED_DTYPE);
  
  //TODO: relax this requirement:
  //
  GDF_REQUIRE( type_id == labels->dtype, GDF_UNSUPPORTED_DTYPE);

  IndexT* p_d_labels = static_cast<IndexT*>(labels->data);
  IndexT* p_d_verts = static_cast<IndexT*>(verts->data);
  
  const IndexT* p_d_row_offsets = row_offsets_(graph);
  const IndexT* p_d_col_ind = col_indices_(graph);

  IndexT nnz = nnz_(graph);
  IndexT nrows = nrows_(graph);//static_cast<IndexT>(graph->adjList->offsets->size) - 1;
  
  if( connectivity_type == CUGRAPH_WEAK )
    {
      // using VectorT = thrust::device_vector<IndexT>;
      // VectorT d_ro(p_d_row_offsets, p_d_row_offsets + nrows + 1);
      // VectorT d_ci(p_d_col_ind, p_d_col_ind + nnz);

#ifdef _DEBUG_WEAK_CC
      IndexT last_elem{0};
      hipMemcpy((void*)(&last_elem), p_d_row_offsets+nrows, sizeof(IndexT), hipMemcpyDeviceToHost);
      std::cout<<"############## "
               <<"nrows = "<<nrows
               <<"; nnz = "<<nnz
               <<"; nnz_ro = "<<last_elem
               <<"; p_d_labels valid: "<<(p_d_labels != nullptr)
               <<"; p_d_row_offsets valid: "<<(p_d_row_offsets != nullptr)
               <<"; p_d_col_ind valid: " << (p_d_col_ind != nullptr)
               <<"\n";
      
      std::cout<<"############## d_ro:\n";
      print_v(d_ro, std::cout);

      std::cout<<"############## d_ci:\n";
      print_v(d_ci, std::cout);
#endif

      //check if graph is undirected; return w/ error, if not?
      //Yes, for now; in the future we may remove this constraint; 
      //
      bool is_symmetric = cugraph::detail::check_symmetry(nrows, p_d_row_offsets, nnz, p_d_col_ind);
#ifdef _DEBUG_WEAK_CC
      std::cout<<"############## "
               <<"; adj. matrix symmetric? " << is_symmetric
               <<"\n";
#endif
      
      GDF_REQUIRE( is_symmetric, GDF_INVALID_API_CALL);
      MLCommon::Sparse::weak_cc_entry<IndexT, TPB_X>(p_d_labels,
                                                     p_d_row_offsets,
                                                     p_d_col_ind,
                                                     nnz,
                                                     nrows,
                                                     stream);

    }
  else
    {
      
      //device memory requirements: 2n^2 + 2n x sizeof(IndexT) + 1 (for flag)
      //( n = |V|)
      //
      size_t n2 = 2*nrows;
      n2 = n2*(nrows*sizeof(ByteT) + sizeof(IndexT)) + 1;

      int device;
      hipDeviceProp_t prop;
      
      hipGetDevice(&device);
      hipGetDeviceProperties(&prop, device);

      if( n2 > prop.totalGlobalMem )
        {

          //not enough memory, dump error message and return unsupported:
          //
          std::cerr<<"ERROR: Insufficient device memory for SCC;"
                   <<" at: " << __FILE__ << ":" << __LINE__ << std::endl;
      
          return GDF_MEMORYMANAGER_ERROR;
        }
      SCC_Data<ByteT, IndexT> sccd(nrows, p_d_row_offsets, p_d_col_ind);
      sccd.run_scc(p_d_labels);
      
    }

  //fill the vertex indices column:
  //
  thrust::sequence(thrust::device, p_d_verts, p_d_verts + nrows);
  
  return GDF_SUCCESS;
}

/**
 * @brief Compute connected components. 
 * The weak version (for undirected graphs, only) was imported from cuML.
 * This implementation comes from [1] and solves component labeling problem in
 * parallel on CSR-indexes based upon the vertex degree and adjacency graph.
 *
 * [1] Hawick, K.A et al, 2010. "Parallel graph component labelling with GPUs and CUDA"
 * 
 * The strong version (for directed or undirected graphs) is based on: 
 * [2] Gilbert, J. et al, 2011. "Graph Algorithms in the Language of Linear Algebra"
 *
 * C = I | A | A^2 |...| A^k
 * where matrix multiplication is via semi-ring: 
 * (combine, reduce) == (&, |) (bitwise ops)
 * Then: X = C & transpose(C); and finally, apply get_labels(X);
 *
 *
 * @param graph input graph; assumed undirected for weakly CC [in]
 * @param connectivity_type CUGRAPH_WEAK or CUGRAPH_STRONG [in]
 * @param table of 2 gdf_columns: output labels and vertex indices [out]
 */
 gdf_error gdf_connected_components(gdf_graph *graph,
                                    cugraph_cc_t connectivity_type,
                                    cudf::table *table)  
{
  hipStream_t stream{nullptr};

  GDF_REQUIRE(table != nullptr, GDF_INVALID_API_CALL);
  GDF_REQUIRE(table->num_columns() > 1, GDF_INVALID_API_CALL);
  
  gdf_column* labels = table->get_column(0);
  gdf_column* verts = table->get_column(1);

  GDF_REQUIRE(labels != nullptr, GDF_INVALID_API_CALL);
  GDF_REQUIRE(verts != nullptr, GDF_INVALID_API_CALL);

  auto dtype = labels->dtype;
  GDF_REQUIRE( dtype == verts->dtype, GDF_INVALID_API_CALL);
  
  switch( dtype )//currently graph's row offsets, col_indices and labels are same type; that may change in the future
    {
    case GDF_INT32:
      return gdf_connected_components_impl<int32_t>(graph, table, connectivity_type, stream);
      //    case GDF_INT64:
      //return gdf_connected_components_impl<int64_t>(graph, labels, connectivity_type, stream);
      // PROBLEM: relies on atomicMin(), which won't work w/ int64_t
      // should work with `unsigned long long` but using signed `Type`'s
      //(initialized to `-1`)
    default:
      break;//warning eater
    }
  return GDF_UNSUPPORTED_DTYPE;
}
