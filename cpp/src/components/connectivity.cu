#include "hip/hip_runtime.h"
#include "weak_cc.cuh"

#include "utilities/graph_utils.cuh"
#include "utilities/error_utils.h"
#include <cugraph.h>
#include <algo_types.h>

#include <iostream>
#include <type_traits>
#include <cstdint>

#include "topology/topology.cuh"

//#define _DEBUG_WEAK_CC

//
/**
 * @brief Compute connected components. 
 * The weak version was imported from cuML.
 * This implementation comes from [1] and solves component labeling problem in
 * parallel on CSR-indexes based upon the vertex degree and adjacency graph.
 *
 * [1] Hawick, K.A et al, 2010. "Parallel graph component labelling with GPUs and CUDA"
 *
 * @tparam IndexT the numeric type of non-floating point elements
 * @tparam TPB_X the threads to use per block when configuring the kernel
 * @param graph input graph; assumed undirected for weakly CC [in]
 * @param labels gdf_column for the output labels [out]
 * @param connectivity_type CUGRAPH_WEAK or CUGRAPH_STRONG
 * @param stream the cuda stream
 */
template<typename IndexT,
         int TPB_X = 32>
std::enable_if_t<std::is_signed<IndexT>::value,gdf_error>
gdf_connected_components_impl(gdf_graph *graph,
                              gdf_column *labels,
                              cugraph_cc_t connectivity_type,
                              hipStream_t stream)
{
  static auto row_offsets_ = [](const gdf_graph* G){
    return static_cast<const IndexT*>(G->adjList->offsets->data);
  };

  static auto col_indices_ = [](const gdf_graph* G){
    return static_cast<const IndexT*>(G->adjList->indices->data);
  };

  static auto nrows_ = [](const gdf_graph* G){
    return G->adjList->offsets->size - 1;
  };

  static auto nnz_ = [](const gdf_graph* G){
    return G->adjList->indices->size;
  };


  GDF_REQUIRE(graph != nullptr, GDF_INVALID_API_CALL);
    
  GDF_REQUIRE(graph->adjList != nullptr, GDF_INVALID_API_CALL);
    
  GDF_REQUIRE(row_offsets_(graph) != nullptr, GDF_INVALID_API_CALL);

  GDF_REQUIRE(col_indices_(graph) != nullptr, GDF_INVALID_API_CALL);
  
  GDF_REQUIRE(labels != nullptr, GDF_INVALID_API_CALL);
  
  GDF_REQUIRE(labels->data != nullptr, GDF_INVALID_API_CALL);
  
  auto type_id = graph->adjList->offsets->dtype;
  GDF_REQUIRE( type_id == GDF_INT32 || type_id == GDF_INT64, GDF_UNSUPPORTED_DTYPE);
  
  GDF_REQUIRE( type_id == graph->adjList->indices->dtype, GDF_UNSUPPORTED_DTYPE);
  
  //TODO: relax this requirement:
  //
  GDF_REQUIRE( type_id == labels->dtype, GDF_UNSUPPORTED_DTYPE);

  //bool flag_dir = graph->prop->directed;//useless, for the time being...
  //TODO: direction_checker() to set this flag correctly; prop is not even allocated!

  IndexT* p_d_labels = static_cast<IndexT*>(labels->data);
  const IndexT* p_d_row_offsets = row_offsets_(graph);
  const IndexT* p_d_col_ind = col_indices_(graph);

  IndexT nnz = nnz_(graph);
  IndexT nrows = nrows_(graph);//static_cast<IndexT>(graph->adjList->offsets->size) - 1;
  
  if( connectivity_type == CUGRAPH_WEAK )
    {
      // using VectorT = thrust::device_vector<IndexT>;
      // VectorT d_ro(p_d_row_offsets, p_d_row_offsets + nrows + 1);
      // VectorT d_ci(p_d_col_ind, p_d_col_ind + nnz);

#ifdef _DEBUG_WEAK_CC
      IndexT last_elem{0};
      hipMemcpy((void*)(&last_elem), p_d_row_offsets+nrows, sizeof(IndexT), hipMemcpyDeviceToHost);
      std::cout<<"############## "
               <<"nrows = "<<nrows
               <<"; nnz = "<<nnz
               <<"; nnz_ro = "<<last_elem
               <<"; p_d_labels valid: "<<(p_d_labels != nullptr)
               <<"; p_d_row_offsets valid: "<<(p_d_row_offsets != nullptr)
               <<"; p_d_col_ind valid: " << (p_d_col_ind != nullptr)
               <<"\n";
      
      std::cout<<"############## d_ro:\n";
      print_v(d_ro, std::cout);

      std::cout<<"############## d_ci:\n";
      print_v(d_ci, std::cout);
#endif

      //check if graph is undirected; return w/ error, if not?
      //Yes, for now; in the future we may remove this constraint; 
      //
      bool is_symmetric = topology::check_symmetry(nrows, p_d_row_offsets, nnz, p_d_col_ind);
#ifdef _DEBUG_WEAK_CC
      std::cout<<"############## "
               <<"; adj. matrix symmetric? " << is_symmetric
               <<"\n";
#endif
      
      GDF_REQUIRE( is_symmetric, GDF_INVALID_API_CALL);
      MLCommon::Sparse::weak_cc_entry<IndexT, TPB_X>(p_d_labels,
                                                     p_d_row_offsets,
                                                     p_d_col_ind,
                                                     nnz,
                                                     nrows,
                                                     stream);

    }
  else
    {
      //dump error message and return unsupported, for now:
      //
      std::cerr<<"ERROR: Feature not supported, yet;"
               <<" at: " << __FILE__ << ":" << __LINE__ << std::endl;
      
      return GDF_INVALID_API_CALL;//for now...
    }
  return GDF_SUCCESS;
}

/**
 * @brief Compute connected components. 
 * The weak version was imported from cuML.
 * This implementation comes from [1] and solves component labeling problem in
 * parallel on CSR-indexes based upon the vertex degree and adjacency graph.
 *
 * [1] Hawick, K.A et al, 2010. "Parallel graph component labelling with GPUs and CUDA"
 * code is adapted / truncated from cuML: ml-prims/src/sparse/csr.h
 *
 
 * @param graph input graph; assumed undirected for weakly CC [in]
 * @param connectivity_type CUGRAPH_WEAK, CUGRAPH_STRONG  [in]
 * @param labels gdf_column for the output labels [out]
 */
 gdf_error gdf_connected_components(gdf_graph *graph,
                                    cugraph_cc_t connectivity_type,
                                    gdf_column *labels)  
{
  hipStream_t stream{nullptr};
  
  switch( labels->dtype )//currently graph's row offsets, col_indices and labels are same type; that may change in the future
    {
    case GDF_INT32:
      return gdf_connected_components_impl<int32_t>(graph, labels, connectivity_type, stream);
      //    case GDF_INT64:
      //return gdf_connected_components_impl<int64_t>(graph, labels, connectivity_type, stream);
      // PROBLEM: relies on atomicMin(), which won't work w/ int64_t
      // should work with `unsigned long long` but using signed `Type`'s
      //(initialized to `-1`)
    default:
      break;//warning eater
    }
  return GDF_UNSUPPORTED_DTYPE;
}
