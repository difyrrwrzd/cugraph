/*
 * Copyright (c) 2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

// Graph analytics features 
// Author: Alex Fender afender@nvidia.com

#include <cugraph.h>
#include "graph_utils.cuh"
#include "pagerank.cuh"
#include "COOtoCSR.cuh"
//#include <functions.h>

void gdf_col_delete(gdf_column* col) {
  if (col)
  {
    col->size = 0; 
    if(col->data)
      if (hipFree(col->data) != hipSuccess) 
        std::cerr << "CUDA ERROR : " << hipGetErrorString(hipGetLastError()) <<std::endl;    
    delete col;
    col->data = nullptr;
    col = nullptr;  
  }                                                       
}

void gdf_col_release(gdf_column* col) {
  delete col;                                                          
}

void cpy_column_view(const gdf_column *in, gdf_column *out) {
  if (in != nullptr && out !=nullptr) {
    gdf_column_view(out, in->data, in->valid, in->size, in->dtype);
  }
}

gdf_error gdf_adj_list_view(gdf_graph *graph, const gdf_column *offsets, 
                                 const gdf_column *indices, const gdf_column *edge_data) {
  GDF_REQUIRE( offsets->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );                    
  GDF_REQUIRE( indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( (offsets->dtype == indices->dtype), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( ((offsets->dtype == GDF_INT32) || (offsets->dtype == GDF_INT64)), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( (offsets->size > 0), GDF_DATASET_EMPTY ); 
  GDF_REQUIRE( (graph->adjList == nullptr) , GDF_INVALID_API_CALL);

  graph->adjList = new gdf_adj_list;
  graph->adjList->offsets = new gdf_column;
  graph->adjList->indices = new gdf_column;
  graph->adjList->ownership = 0;

  cpy_column_view(offsets, graph->adjList->offsets);
  cpy_column_view(indices, graph->adjList->indices);
  if (edge_data) {
      GDF_REQUIRE( indices->size == edge_data->size, GDF_COLUMN_SIZE_MISMATCH );
      graph->adjList->edge_data = new gdf_column;
      cpy_column_view(edge_data, graph->adjList->edge_data);
  }
  else {
    graph->adjList->edge_data = nullptr;
  }
  return GDF_SUCCESS;
}

gdf_error gdf_edge_list_view(gdf_graph *graph, const gdf_column *src_indices, 
                                 const gdf_column *dest_indices, const gdf_column *edge_data) {
  GDF_REQUIRE( src_indices->size == dest_indices->size, GDF_COLUMN_SIZE_MISMATCH );
  GDF_REQUIRE( src_indices->dtype == dest_indices->dtype, GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( ((src_indices->dtype == GDF_INT32) || (src_indices->dtype == GDF_INT64)), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( src_indices->size > 0, GDF_DATASET_EMPTY ); 
  GDF_REQUIRE( src_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );                    
  GDF_REQUIRE( dest_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( graph->edgeList == nullptr , GDF_INVALID_API_CALL);

  graph->edgeList = new gdf_edge_list;
  graph->edgeList->src_indices = new gdf_column;
  graph->edgeList->dest_indices = new gdf_column;
  graph->edgeList->ownership = 0;

  cpy_column_view(src_indices, graph->edgeList->src_indices);
  cpy_column_view(dest_indices, graph->edgeList->dest_indices);
  if (edge_data) {
      GDF_REQUIRE( src_indices->size == edge_data->size, GDF_COLUMN_SIZE_MISMATCH );
      graph->edgeList->edge_data = new gdf_column;
      cpy_column_view(edge_data, graph->edgeList->edge_data);
  }
  else {
    graph->edgeList->edge_data = nullptr;
  }
  return GDF_SUCCESS;
}

template <typename WT>
gdf_error gdf_add_adj_list_impl (gdf_graph *graph) {
    GDF_REQUIRE( graph->edgeList != nullptr , GDF_INVALID_API_CALL);
    GDF_REQUIRE( graph->adjList == nullptr , GDF_INVALID_API_CALL);
    
    int nnz = graph->edgeList->src_indices->size, status = 0;
    graph->adjList = new gdf_adj_list;
    graph->adjList->offsets = new gdf_column;
    graph->adjList->indices = new gdf_column;
    graph->adjList->ownership = 1;

  if (graph->edgeList->edge_data!= nullptr) {
    graph->adjList->edge_data = new gdf_column;

    CSR_Result_Weighted<int,WT> adj_list;
    status = ConvertCOOtoCSR_weighted((int*)graph->edgeList->src_indices->data, (int*)graph->edgeList->dest_indices->data, (WT*)graph->edgeList->edge_data->data, nnz, adj_list);
    
    gdf_column_view(graph->adjList->offsets, adj_list.rowOffsets, 
                          nullptr, adj_list.size, graph->edgeList->src_indices->dtype);
    gdf_column_view(graph->adjList->indices, adj_list.colIndices, 
                          nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
    gdf_column_view(graph->adjList->edge_data, adj_list.edgeWeights, 
                        nullptr, adj_list.nnz, graph->edgeList->edge_data->dtype);
  }
  else {
    CSR_Result<int> adj_list;
    status = ConvertCOOtoCSR((int*)graph->edgeList->src_indices->data,(int*)graph->edgeList->dest_indices->data, nnz, adj_list);      
    gdf_column_view(graph->adjList->offsets, adj_list.rowOffsets, 
                          nullptr, adj_list.size, graph->edgeList->src_indices->dtype);
    gdf_column_view(graph->adjList->indices, adj_list.colIndices, 
                          nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
  }
  if (status !=0) {
    std::cerr << "Could not generate the adj_list" << std::endl;
    return GDF_CUDA_ERROR;
  }
  return GDF_SUCCESS;
}


template <typename WT>
gdf_error gdf_add_transpose_impl (gdf_graph *graph) {
    GDF_REQUIRE( graph->edgeList != nullptr , GDF_INVALID_API_CALL);
    GDF_REQUIRE( graph->transposedAdjList == nullptr , GDF_INVALID_API_CALL);
    int nnz = graph->edgeList->src_indices->size, status = 0;
    graph->transposedAdjList = new gdf_adj_list;
    graph->transposedAdjList->offsets = new gdf_column;
    graph->transposedAdjList->indices = new gdf_column;
    graph->transposedAdjList->ownership = 1;
  
  if (graph->edgeList->edge_data) {
    graph->transposedAdjList->edge_data = new gdf_column;
    CSR_Result_Weighted<int,WT> adj_list;
    status = ConvertCOOtoCSR_weighted( (int*)graph->edgeList->dest_indices->data, (int*)graph->edgeList->src_indices->data, (WT*)graph->edgeList->edge_data->data, nnz, adj_list);
    gdf_column_view(graph->transposedAdjList->offsets, adj_list.rowOffsets, 
                          nullptr, adj_list.size, graph->edgeList->src_indices->dtype);
    gdf_column_view(graph->transposedAdjList->indices, adj_list.colIndices, 
                          nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
    gdf_column_view(graph->transposedAdjList->edge_data, adj_list.edgeWeights, 
                        nullptr, adj_list.nnz, graph->edgeList->edge_data->dtype);
  }
  else {

    CSR_Result<int> adj_list;
    status = ConvertCOOtoCSR((int*)graph->edgeList->dest_indices->data, (int*)graph->edgeList->src_indices->data, nnz, adj_list);      
    gdf_column_view(graph->transposedAdjList->offsets, adj_list.rowOffsets, 
                          nullptr, adj_list.size, graph->edgeList->src_indices->dtype);
    gdf_column_view(graph->transposedAdjList->indices, adj_list.colIndices, 
                          nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
  }
    if (status !=0) {
      std::cerr << "Could not generate the adj_list" << std::endl;
      return GDF_CUDA_ERROR;
    }
    return GDF_SUCCESS;
}

template <typename WT>
gdf_error gdf_pagerank_impl (gdf_graph *graph,
                      gdf_column *pagerank, float alpha = 0.85,
                      float tolerance = 1e-4, int max_iter = 200,
                      bool has_guess = false) {

  
  GDF_REQUIRE( graph->edgeList != nullptr, GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( graph->edgeList->src_indices->size == graph->edgeList->dest_indices->size, GDF_COLUMN_SIZE_MISMATCH ); 
  GDF_REQUIRE( graph->edgeList->src_indices->dtype == graph->edgeList->dest_indices->dtype, GDF_UNSUPPORTED_DTYPE );  
  GDF_REQUIRE( graph->edgeList->src_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );                 
  GDF_REQUIRE( graph->edgeList->dest_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );  
  GDF_REQUIRE( pagerank != nullptr , GDF_INVALID_API_CALL ); 
  GDF_REQUIRE( pagerank->data != nullptr , GDF_INVALID_API_CALL ); 
  GDF_REQUIRE( pagerank->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );          
  GDF_REQUIRE( pagerank->size > 0 , GDF_INVALID_API_CALL );         

  int m=pagerank->size, nnz = graph->edgeList->src_indices->size, status = 0;
  WT *d_pr, *d_val = nullptr, *d_leaf_vector = nullptr; 
  WT res = 1.0;
  WT *residual = &res;

  if (graph->transposedAdjList == nullptr) {
    gdf_add_transpose(graph);
  }

  CUDA_TRY(hipMallocManaged ((void**)&d_leaf_vector,    sizeof(WT) * m));
  CUDA_TRY(hipMallocManaged ((void**)&d_val, sizeof(WT) * nnz )); 
  CUDA_TRY(hipMallocManaged ((void**)&d_pr,    sizeof(WT) * m));

  cugraph::HT_matrix_csc_coo(m, nnz, (int*)graph->transposedAdjList->offsets->data, (int*)graph->transposedAdjList->indices->data, d_val, d_leaf_vector);

  if (has_guess)
  {
    GDF_REQUIRE( pagerank->data != nullptr, GDF_VALIDITY_UNSUPPORTED );
    cugraph::copy<WT>(m, (WT*)pagerank->data, d_pr);
  }

  status = cugraph::pagerank<int,WT>( m,nnz, (int*)graph->transposedAdjList->offsets->data, (int*)graph->transposedAdjList->indices->data, 
    d_val, alpha, d_leaf_vector, false, tolerance, max_iter, d_pr, residual);
 
  if (status !=0)
    switch ( status ) { 
      case -1: std::cerr<< "Error : bad parameters in Pagerank"<<std::endl; return GDF_CUDA_ERROR; 
      case 1: std::cerr<< "Warning : Pagerank did not reached the desired tolerance"<<std::endl;  return GDF_CUDA_ERROR; 
      default:  std::cerr<< "Pagerank failed"<<std::endl;  return GDF_CUDA_ERROR; 
    }   
 
  cugraph::copy<WT>(m, d_pr, (WT*)pagerank->data);
  hipFree(d_val);
  hipFree(d_pr);    
  hipFree(d_leaf_vector);  
  return GDF_SUCCESS;
}


gdf_error gdf_add_adj_list(gdf_graph *graph)
{
  if (graph->edgeList->edge_data != nullptr) {
    switch (graph->edgeList->edge_data->dtype) {
      case GDF_FLOAT32:   return gdf_add_adj_list_impl<float>(graph);
      case GDF_FLOAT64:   return gdf_add_adj_list_impl<double>(graph);
      default: return GDF_UNSUPPORTED_DTYPE;
    }
  }
  else {
    return gdf_add_adj_list_impl<float>(graph);
  }
}

gdf_error gdf_add_edge_list(gdf_graph *graph)
{
  return GDF_UNSUPPORTED_METHOD;
}

gdf_error gdf_add_transpose(gdf_graph *graph)
{
  if (graph->edgeList->edge_data != nullptr) {
    switch (graph->edgeList->edge_data->dtype) {
      case GDF_FLOAT32:   return gdf_add_transpose_impl<float>(graph);
      case GDF_FLOAT64:   return gdf_add_transpose_impl<double>(graph);
      default: return GDF_UNSUPPORTED_DTYPE;
    }
  }
  else {
    return gdf_add_transpose_impl<float>(graph);
  }
}

gdf_error gdf_delete_adj_list(gdf_graph *graph) {
  if (graph->adjList) {
    graph->adjList->ownership = 1;
    delete graph->adjList;
  }
  graph->adjList = nullptr;
  return GDF_SUCCESS;
}
gdf_error gdf_delete_edge_list(gdf_graph *graph) {
  if (graph->edgeList) {
    graph->edgeList->ownership = 1;
    delete graph->edgeList;
  }
  graph->edgeList = nullptr;
  return GDF_SUCCESS;
}
gdf_error gdf_delete_transpose(gdf_graph *graph) {
  if (graph->transposedAdjList) {
    graph->transposedAdjList->ownership = 1;
    delete graph->transposedAdjList;
  }
  graph->transposedAdjList = nullptr;
  return GDF_SUCCESS;
}


gdf_error gdf_pagerank(gdf_graph *graph, gdf_column *pagerank, float alpha, float tolerance, int max_iter, bool has_guess)
{ 
  switch (pagerank->dtype) {
    case GDF_FLOAT32:   return gdf_pagerank_impl<float>(graph, pagerank, alpha, tolerance, max_iter, has_guess);
    case GDF_FLOAT64:   return gdf_pagerank_impl<double>(graph, pagerank, alpha, tolerance, max_iter, has_guess);
    default: return GDF_UNSUPPORTED_DTYPE;
  }
}
